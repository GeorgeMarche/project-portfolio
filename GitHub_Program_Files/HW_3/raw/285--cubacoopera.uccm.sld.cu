#include "hip/hip_runtime.h"
<!DOCTYPE html>
<html lang="es">
<head>
<meta name="keywords"content="cubacoopera, colaboración, médicos, cubanos, Cuba, Salud">
<script async defer src="//telus.redcuba.cu/tracker/telus.min.js" type="text/javascript" id="telus-tracker" data-id="150"></script>
<meta charset="UTF-8" />
<meta name="msvalidate.01" content="ED4683FB2BD8CB7A8CCA343A9F69B048" />
<meta name="google-site-verification" content="6v_wNN3O0zMGR9kfa53yNlc-w73BvDq1drfb7_sIiDo" />

<link rel="profile" href="http://gmpg.org/xfn/11" />
<title>cubacoopera</title>
	<script type="text/javascript">function theChampLoadEvent(e){var t=window.onload;if(typeof window.onload!="function"){window.onload=e}else{window.onload=function(){t();e()}}}</script>
		<script type="text/javascript">var theChampDefaultLang = 'es_ES', theChampCloseIconPath = 'http://cubacoopera.uccm.sld.cu/wp-content/plugins/super-socializer/images/close.png';var heateorSsSDKBlockedMsg = `Your browser is blocking some features of this website. Please follow the instructions at <a href=\'http://support.heateor.com/browser-blocking-social-features/\' target=\'_blank\' style=\'color:#33a9d8\'>http://support.heateor.com/browser-blocking-social-features/</a> to unblock these.`</script>
		<script> var theChampSiteUrl = 'http://cubacoopera.uccm.sld.cu', theChampVerified = 0, theChampEmailPopup = 0; </script>
			<script> var theChampFBKey = '345268949277726', theChampFBLang = 'es_ES', theChampFbLikeMycred = 0, theChampSsga = 0, theChampCommentNotification = 0, theChampHeateorFcmRecentComments = 0, theChampFbIosLogin = 0; </script>
						<script type="text/javascript">var theChampFBCommentUrl = 'http://cubacoopera.uccm.sld.cu/salva-vidas-donando-organos/'; var theChampFBCommentColor = 'light'; var theChampFBCommentNumPosts = ''; var theChampFBCommentWidth = '100%'; var theChampFBCommentOrderby = 'social'; var theChampCommentingTabs = "wordpress,facebook,googleplus,disqus", theChampGpCommentsUrl = 'http://cubacoopera.uccm.sld.cu/salva-vidas-donando-organos/', theChampDisqusShortname = '', theChampScEnabledTabs = 'wordpress,fb,googleplus', theChampScLabel = 'Comentarios', theChampScTabLabels = {"wordpress":"Comentarios en Cubacoopera (0)","fb":"Comentarios en Facebook","googleplus":"Comentarios en G+","disqus":"Comentarios en Disqus"}, theChampGpCommentsWidth = 0, theChampCommentingId = 'respond'</script>
						<script> var theChampSharingAjaxUrl = 'http://cubacoopera.uccm.sld.cu/wp-admin/admin-ajax.php', heateorSsWhatsappShareAPI = 'web', heateorSsUrlCountFetched = [], heateorSsSharesText = 'Shares', heateorSsShareText = 'Share', theChampPluginIconPath = 'http://cubacoopera.uccm.sld.cu/wp-content/plugins/super-socializer/images/logo.png', theChampHorizontalSharingCountEnable = 1, theChampVerticalSharingCountEnable = 0, theChampSharingOffset = -10, theChampCounterOffset = -10, theChampMobileStickySharingEnabled = 0, heateorSsCopyLinkMessage = "Link copied.";
		var heateorSsHorSharingShortUrl = "http://cubacoopera.uccm.sld.cu/salva-vidas-donando-organos/";var theChampReduceHorizontalSvgWidth = true;var theChampReduceVerticalSvgWidth = true;		</script>
			<style type="text/css">.the_champ_horizontal_sharing .theChampSharing{
					color: #fff;
				border-width: 0px;
		border-style: solid;
		border-color: transparent;
	}
		.the_champ_horizontal_sharing .theChampTCBackground{
		color:#666;
	}
		.the_champ_horizontal_sharing .theChampSharing:hover{
				border-color: transparent;
	}
	.the_champ_vertical_sharing .theChampSharing{
					color: #fff;
				border-width: 0px;
		border-style: solid;
		border-color: transparent;
	}
		.the_champ_vertical_sharing .theChampTCBackground{
		color:#666;
	}
		.the_champ_vertical_sharing .theChampSharing:hover{
				border-color: transparent;
	}
				div.the_champ_horizontal_sharing .theChampSharingSvg{width:50%;margin:auto;}div.the_champ_horizontal_sharing .the_champ_square_count{float:left;width:50%;line-height:35px;}
						div.the_champ_vertical_sharing .theChampSharingSvg{width:50%;margin:auto;}div.the_champ_vertical_sharing .the_champ_square_count{float:left;width:50%;line-height:40px;}
			@media screen and (max-width:783px){.the_champ_vertical_sharing{display:none!important}}.theChampFacebookLogin, .theChampGoogleLogin, .theChampLinkedinLogin, .theChampVkontakteLogin {
    display: block !important;
}</style>
			 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 		 
<!-- This site is optimized with the Yoast SEO plugin v11.7 - https://yoast.com/wordpress/plugins/seo/ -->
<meta name="description" content="Portal web de la cooperación médica cubana, pretende mostrar los principales resultados de la colaboración cubana en el exterior"/>
<link rel="canonical" href="http://cubacoopera.uccm.sld.cu/" />
<link rel="next" href="http://cubacoopera.uccm.sld.cu/page/2/" />
<meta property="og:locale" content="es_ES" />
<meta property="og:type" content="website" />
<meta property="og:title" content="cubacoopera" />
<meta property="og:description" content="Portal web de la cooperación médica cubana, pretende mostrar los principales resultados de la colaboración cubana en el exterior" />
<meta property="og:url" content="http://cubacoopera.uccm.sld.cu/" />
<meta property="og:site_name" content="Cubacoopera" />
<meta property="fb:app_id" content="345268949277726" />
<meta property="og:image" content="http://cubacoopera.uccm.sld.cu/wp-content/uploads/2019/06/1-Logo-de-cubacoopera.jpg" />
<meta property="og:image:width" content="240" />
<meta property="og:image:height" content="240" />
<meta name="twitter:card" content="summary_large_image" />
<meta name="twitter:description" content="Portal web de la cooperación médica cubana, pretende mostrar los principales resultados de la colaboración cubana en el exterior" />
<meta name="twitter:title" content="cubacoopera" />
<meta name="twitter:site" content="@cuba_coopera" />
<meta name="twitter:image" content="http://cubacoopera.uccm.sld.cu/wp-content/uploads/2019/06/1-Logo-de-cubacoopera.jpg" />
<script type='application/ld+json' class='yoast-schema-graph yoast-schema-graph--main'>{"@context":"https://schema.org","@graph":[{"@type":"Organization","@id":"http://cubacoopera.uccm.sld.cu/#organization","name":"Unidad Central de Cooperaci\u00f3n M\u00e9dica","url":"http://cubacoopera.uccm.sld.cu/","sameAs":["https://www.facebook.com/CubacooperaCu/","https://twitter.com/cuba_coopera"],"logo":{"@type":"ImageObject","@id":"http://cubacoopera.uccm.sld.cu/#logo","url":"http://cubacoopera.uccm.sld.cu/wp-content/uploads/2019/06/logo-uccm-2.jpg","width":482,"height":322,"caption":"Unidad Central de Cooperaci\u00f3n M\u00e9dica"},"image":{"@id":"http://cubacoopera.uccm.sld.cu/#logo"}},{"@type":"WebSite","@id":"http://cubacoopera.uccm.sld.cu/#website","url":"http://cubacoopera.uccm.sld.cu/","name":"Cubacoopera","publisher":{"@id":"http://cubacoopera.uccm.sld.cu/#organization"},"potentialAction":{"@type":"SearchAction","target":"http://cubacoopera.uccm.sld.cu/?s={search_term_string}","query-input":"required name=search_term_string"}},{"@type":"CollectionPage","@id":"http://cubacoopera.uccm.sld.cu/#webpage","url":"http://cubacoopera.uccm.sld.cu/","inLanguage":"es","name":"cubacoopera","isPartOf":{"@id":"http://cubacoopera.uccm.sld.cu/#website"},"about":{"@id":"http://cubacoopera.uccm.sld.cu/#organization"},"description":"Portal web de la cooperaci\u00f3n m\u00e9dica cubana, pretende mostrar los principales resultados de la colaboraci\u00f3n cubana en el exterior"}]}</script>
<!-- / Yoast SEO plugin. -->

<link rel='dns-prefetch' href='//s.w.org' />
<link rel="alternate" type="application/rss+xml" title="Cubacoopera &raquo; Feed" href="http://cubacoopera.uccm.sld.cu/feed/" />
<link rel="alternate" type="application/rss+xml" title="Cubacoopera &raquo; RSS de los comentarios" href="http://cubacoopera.uccm.sld.cu/comments/feed/" />
		<script type="text/javascript">
			window._wpemojiSettings = {"baseUrl":"https:\/\/s.w.org\/images\/core\/emoji\/12.0.0-1\/72x72\/","ext":".png","svgUrl":"https:\/\/s.w.org\/images\/core\/emoji\/12.0.0-1\/svg\/","svgExt":".svg","source":{"concatemoji":"http:\/\/cubacoopera.uccm.sld.cu\/wp-includes\/js\/wp-emoji-release.min.js?ver=5.2.4"}};
			!function(a,b,c){function d(a,b){var c=String.fromCharCode;l.clearRect(0,0,k.width,k.height),l.fillText(c.apply(this,a),0,0);var d=k.toDataURL();l.clearRect(0,0,k.width,k.height),l.fillText(c.apply(this,b),0,0);var e=k.toDataURL();return d===e}function e(a){var b;if(!l||!l.fillText)return!1;switch(l.textBaseline="top",l.font="600 32px Arial",a){case"flag":return!(b=d([55356,56826,55356,56819],[55356,56826,8203,55356,56819]))&&(b=d([55356,57332,56128,56423,56128,56418,56128,56421,56128,56430,56128,56423,56128,56447],[55356,57332,8203,56128,56423,8203,56128,56418,8203,56128,56421,8203,56128,56430,8203,56128,56423,8203,56128,56447]),!b);case"emoji":return b=d([55357,56424,55356,57342,8205,55358,56605,8205,55357,56424,55356,57340],[55357,56424,55356,57342,8203,55358,56605,8203,55357,56424,55356,57340]),!b}return!1}function f(a){var c=b.createElement("script");c.src=a,c.defer=c.type="text/javascript",b.getElementsByTagName("head")[0].appendChild(c)}var g,h,i,j,k=b.createElement("canvas"),l=k.getContext&&k.getContext("2d");for(j=Array("flag","emoji"),c.supports={everything:!0,everythingExceptFlag:!0},i=0;i<j.length;i++)c.supports[j[i]]=e(j[i]),c.supports.everything=c.supports.everything&&c.supports[j[i]],"flag"!==j[i]&&(c.supports.everythingExceptFlag=c.supports.everythingExceptFlag&&c.supports[j[i]]);c.supports.everythingExceptFlag=c.supports.everythingExceptFlag&&!c.supports.flag,c.DOMReady=!1,c.readyCallback=function(){c.DOMReady=!0},c.supports.everything||(h=function(){c.readyCallback()},b.addEventListener?(b.addEventListener("DOMContentLoaded",h,!1),a.addEventListener("load",h,!1)):(a.attachEvent("onload",h),b.attachEvent("onreadystatechange",function(){"complete"===b.readyState&&c.readyCallback()})),g=c.source||{},g.concatemoji?f(g.concatemoji):g.wpemoji&&g.twemoji&&(f(g.twemoji),f(g.wpemoji)))}(window,document,window._wpemojiSettings);
		</script>
		<style type="text/css">
img.wp-smiley,
img.emoji {
	display: inline !important;
	border: none !important;
	box-shadow: none !important;
	height: 1em !important;
	width: 1em !important;
	margin: 0 .07em !important;
	vertical-align: -0.1em !important;
	background: none !important;
	padding: 0 !important;
}
</style>
	<link rel='stylesheet' id='ai1ec_style-css'  href='//cubacoopera.uccm.sld.cu/wp-content/plugins/all-in-one-event-calendar/public/themes-ai1ec/vortex/css/ai1ec_parsed_css.css?ver=2.5.36' type='text/css' media='all' />
<link rel='stylesheet' id='tp-sidebar-login-css'  href='http://cubacoopera.uccm.sld.cu/wp-content/plugins/tt-sidebar-login-widget.2.1/assets/css/tt-sidebar-login.css?ver=5.2.4' type='text/css' media='all' />
<link rel='stylesheet' id='wp-block-library-css'  href='http://cubacoopera.uccm.sld.cu/wp-includes/css/dist/block-library/style.min.css?ver=5.2.4' type='text/css' media='all' />
<link rel='stylesheet' id='gctl-timeline-styles-css-css'  href='http://cubacoopera.uccm.sld.cu/wp-content/plugins/cool-timeline/gutenberg-instant-builder/dist/blocks.style.build.css?ver=5.2.4' type='text/css' media='all' />
<link rel='stylesheet' id='contact-form-7-css'  href='http://cubacoopera.uccm.sld.cu/wp-content/plugins/contact-form-7/includes/css/styles.css?ver=5.1.1' type='text/css' media='all' />
<link rel='stylesheet' id='wpdm-font-awesome-css'  href='http://cubacoopera.uccm.sld.cu/wp-content/plugins/download-manager/assets/fontawesome/css/all.css?ver=5.2.4' type='text/css' media='all' />
<link rel='stylesheet' id='wpdm-bootstrap-css'  href='http://cubacoopera.uccm.sld.cu/wp-content/plugins/download-manager/assets/bootstrap/css/bootstrap.css?ver=5.2.4' type='text/css' media='all' />
<link rel='stylesheet' id='wpdm-front-css'  href='http://cubacoopera.uccm.sld.cu/wp-content/plugins/download-manager/assets/css/front.css?ver=5.2.4' type='text/css' media='all' />
<link rel='stylesheet' id='easingslider-css'  href='http://cubacoopera.uccm.sld.cu/wp-content/plugins/easing-slider/css/easingslider.min.css?ver=2.2.0.8' type='text/css' media='all' />
<link rel='stylesheet' id='style_login_widget-css'  href='http://cubacoopera.uccm.sld.cu/wp-content/plugins/login-sidebar-widget/css/style_login_widget.css?ver=5.2.4' type='text/css' media='all' />
<link rel='stylesheet' id='pdfprnt_frontend-css'  href='http://cubacoopera.uccm.sld.cu/wp-content/plugins/pdf-print/css/frontend.css?ver=2.1.9' type='text/css' media='all' />
<link rel='stylesheet' id='sb-styles-css'  href='http://cubacoopera.uccm.sld.cu/wp-content/plugins/sb-popular-posts-tabbed-widget/includes/css/style.css?ver=5.2.4' type='text/css' media='all' />
<link rel='stylesheet' id='social-widget-css'  href='http://cubacoopera.uccm.sld.cu/wp-content/plugins/social-media-widget/social_widget.css?ver=5.2.4' type='text/css' media='all' />
<link rel='stylesheet' id='cssnews-css'  href='http://cubacoopera.uccm.sld.cu/wp-content/plugins/sp-news-and-widget/css/stylenews.css?ver=3.2.8' type='text/css' media='all' />
<link rel='stylesheet' id='wonderplugin-gridgallery-engine-css-css'  href='http://cubacoopera.uccm.sld.cu/wp-content/plugins/wonderplugin-gridgallery/engine/wonderplugingridgalleryengine.css?ver=5.2.4' type='text/css' media='all' />
<link rel='stylesheet' id='wonderplugin-slider-css-css'  href='http://cubacoopera.uccm.sld.cu/wp-content/plugins/wonderplugin-slider/engine/wonderpluginsliderengine.css?ver=5.2.4' type='text/css' media='all' />
<link rel='stylesheet' id='font-awesome-css'  href='http://cubacoopera.uccm.sld.cu/wp-content/plugins/tm-timeline/css/font-awesome.min.css?ver=4.6.3' type='text/css' media='all' />
<link rel='stylesheet' id='tm-timeline-css-css'  href='http://cubacoopera.uccm.sld.cu/wp-content/plugins/tm-timeline/css/tm-timeline.css?ver=1.1.1' type='text/css' media='all' />
<link rel='stylesheet' id='magbook-style-css'  href='http://cubacoopera.uccm.sld.cu/wp-content/themes/magbook/style.css?ver=5.2.4' type='text/css' media='all' />
<style id='magbook-style-inline-css' type='text/css'>
/* Show Author */
			.entry-meta .author,
			.mb-entry-meta .author {
				float: left;
				display: block;
				visibility: visible;
			}
	/****************************************************************/
						/*.... Color Style ....*/
	/****************************************************************/
	/* Nav, links and hover */

a,
ul li a:hover,
ol li a:hover,
.top-bar .top-bar-menu a:hover,
.main-navigation a:hover, /* Navigation */
.main-navigation ul li.current-menu-item a,
.main-navigation ul li.current_page_ancestor a,
.main-navigation ul li.current-menu-ancestor a,
.main-navigation ul li.current_page_item a,
.main-navigation ul li:hover > a,
.main-navigation li.current-menu-ancestor.menu-item-has-children > a:after,
.main-navigation li.current-menu-item.menu-item-has-children > a:after,
.main-navigation ul li:hover > a:after,
.main-navigation li.menu-item-has-children > a:hover:after,
.main-navigation li.page_item_has_children > a:hover:after,
.main-navigation ul li ul li a:hover,
.main-navigation ul li ul li:hover > a,
.main-navigation ul li.current-menu-item ul li a:hover,
.side-menu-wrap .side-nav-wrap a:hover, /* Side Menu */
.entry-title a:hover, /* Post */
.entry-title a:focus,
.entry-title a:active,
.entry-meta a:hover,
.image-navigation .nav-links a,
a.more-link,
.widget ul li a:hover, /* Widgets */
.widget-title a:hover,
.widget_contact ul li a:hover,
.site-info .copyright a:hover, /* Footer */
#secondary .widget-title,
#colophon .widget ul li a:hover,
#footer-navigation a:hover {
	color: #bf1e2d;
}


.cat-tab-menu li:hover,
.cat-tab-menu li.active {
	color: #bf1e2d !important;
}

#sticky-header,
#secondary .widget-title,
.side-menu {
	border-top-color: #bf1e2d;
}

/* Webkit */
::selection {
	background: #bf1e2d;
	color: #fff;
}

/* Gecko/Mozilla */
::-moz-selection {
	background: #bf1e2d;
	color: #fff;
}

/* Accessibility
================================================== */
.screen-reader-text:hover,
.screen-reader-text:active,
.screen-reader-text:focus {
	background-color: #f1f1f1;
	color: #bf1e2d;
}

/* Default Buttons
================================================== */
input[type="reset"],/* Forms  */
input[type="button"],
input[type="submit"],
.main-slider .flex-control-nav a.flex-active,
.main-slider .flex-control-nav a:hover,
.go-to-top .icon-bg,
.search-submit,
.btn-default,
.widget_tag_cloud a {
	background-color: #3100c4;
}

/* Top Bar Background
================================================== */
.top-bar {
	background-color: #283891;
}

/* Breaking News
================================================== */
.breaking-news-header,
.news-header-title:after {
	background-color: #bf1e2d;
}

.breaking-news-slider .flex-direction-nav li a:hover,
.breaking-news-slider .flex-pauseplay a:hover {
	background-color: #bf1e2d;
	border-color: #bf1e2d;
}

/* Feature News
================================================== */
.feature-news-title {
	border-color: #bf1e2d;
}

.feature-news-slider .flex-direction-nav li a:hover {
	background-color: #bf1e2d;
	border-color: #bf1e2d;
}

/* Tab Widget
================================================== */
.tab-menu,
.mb-tag-cloud .mb-tags a {
	background-color: #bf1e2d;
}

/* Category Box Widgets
================================================== */
.widget-cat-box .widget-title {
	color: #bf1e2d;
}

.widget-cat-box .widget-title span {
	border-bottom: 1px solid #bf1e2d;
}

/* Category Box two Widgets
================================================== */
.widget-cat-box-2 .widget-title {
	color: #bf1e2d;
}

.widget-cat-box-2 .widget-title span {
	border-bottom: 1px solid #bf1e2d;
}

/* #bbpress
================================================== */
#bbpress-forums .bbp-topics a:hover {
	color: #bf1e2d;
}

.bbp-submit-wrapper button.submit {
	background-color: #bf1e2d;
	border: 1px solid #bf1e2d;
}

/* Woocommerce
================================================== */
.woocommerce #respond input#submit,
.woocommerce a.button, 
.woocommerce button.button, 
.woocommerce input.button,
.woocommerce #respond input#submit.alt, 
.woocommerce a.button.alt, 
.woocommerce button.button.alt, 
.woocommerce input.button.alt,
.woocommerce-demo-store p.demo_store,
.top-bar .cart-value {
	background-color: #bf1e2d;
}

.woocommerce .woocommerce-message:before {
	color: #bf1e2d;
}

/* Category Slider widget */
.widget-cat-slider .widget-title {
	color: #d4000e;
}

.widget-cat-slider .widget-title span {
	border-bottom: 1px solid #d4000e;
}

/* Category Grid widget */
.widget-cat-grid .widget-title {
	color: #d4000e;
}

.widget-cat-grid .widget-title span {
	border-bottom: 1px solid #d4000e;
}

</style>
<link rel='stylesheet' id='magbook-responsive-css'  href='http://cubacoopera.uccm.sld.cu/wp-content/themes/magbook/css/responsive.css?ver=5.2.4' type='text/css' media='all' />
<link rel='stylesheet' id='newsletter-css'  href='http://cubacoopera.uccm.sld.cu/wp-content/plugins/newsletter/style.css?ver=6.3.2' type='text/css' media='all' />
<link rel='stylesheet' id='the_champ_frontend_css-css'  href='http://cubacoopera.uccm.sld.cu/wp-content/plugins/super-socializer/css/front.css?ver=7.12.11' type='text/css' media='all' />
<link rel='stylesheet' id='the_champ_sharing_default_svg-css'  href='http://cubacoopera.uccm.sld.cu/wp-content/plugins/super-socializer/css/share-svg.css?ver=7.12.11' type='text/css' media='all' />
<link rel='stylesheet' id='tablepress-default-css'  href='http://cubacoopera.uccm.sld.cu/wp-content/tablepress-combined.min.css?ver=8' type='text/css' media='all' />
<link rel='stylesheet' id='wppb_stylesheet-css'  href='http://cubacoopera.uccm.sld.cu/wp-content/plugins/profile-builder/assets/css/style-front-end.css?ver=2.9.6' type='text/css' media='all' />
<link rel='stylesheet' id='responsive-posts-widget-css'  href='http://cubacoopera.uccm.sld.cu/wp-content/plugins/responsive-posts-widget/responsive-posts-widget.css?ver=1.0.3' type='text/css' media='screen' />
<script type='text/javascript'>
/* <![CDATA[ */
var wpdm_url = {"home":"http:\/\/cubacoopera.uccm.sld.cu\/","site":"http:\/\/cubacoopera.uccm.sld.cu\/","ajax":"http:\/\/cubacoopera.uccm.sld.cu\/wp-admin\/admin-ajax.php"};
/* ]]> */
</script>
<script type='text/javascript' src='http://cubacoopera.uccm.sld.cu/wp-includes/js/jquery/jquery.js?ver=1.12.4-wp'></script>
<script type='text/javascript' src='http://cubacoopera.uccm.sld.cu/wp-includes/js/jquery/jquery-migrate.min.js?ver=1.4.1'></script>
<script type='text/javascript' src='http://cubacoopera.uccm.sld.cu/wp-content/plugins/download-manager/assets/bootstrap/js/bootstrap.min.js?ver=5.2.4'></script>
<script type='text/javascript' src='http://cubacoopera.uccm.sld.cu/wp-content/plugins/download-manager/assets/js/front.js?ver=5.2.4'></script>
<script type='text/javascript' src='http://cubacoopera.uccm.sld.cu/wp-content/plugins/download-manager/assets/js/chosen.jquery.min.js?ver=5.2.4'></script>
<script type='text/javascript' src='http://cubacoopera.uccm.sld.cu/wp-content/plugins/easing-slider/js/jquery.easingslider.min.js?ver=2.2.0.8'></script>
<script type='text/javascript' src='http://cubacoopera.uccm.sld.cu/wp-content/plugins/login-sidebar-widget/js/jquery.validate.min.js?ver=5.2.4'></script>
<script type='text/javascript' src='http://cubacoopera.uccm.sld.cu/wp-content/plugins/login-sidebar-widget/js/additional-methods.js?ver=5.2.4'></script>
<script type='text/javascript' src='http://cubacoopera.uccm.sld.cu/wp-content/plugins/pdf-print/js/html2canvas.js?ver=5.2.4'></script>
<script type='text/javascript' src='http://cubacoopera.uccm.sld.cu/wp-content/plugins/pdf-print/js/jspdf.js?ver=5.2.4'></script>
<script type='text/javascript'>
/* <![CDATA[ */
var pdfprnt_file_settings = {"margin_left":"15","margin_right":"15","margin_top":"16","margin_bottom":"16","page_size":"A4","file_action":"open","file_name":"Salva vidas donando \u00f3rganos"};
/* ]]> */
</script>
<script type='text/javascript' src='http://cubacoopera.uccm.sld.cu/wp-content/plugins/pdf-print/js/front-script.js?ver=5.2.4'></script>
<script type='text/javascript' src='http://cubacoopera.uccm.sld.cu/wp-content/plugins/sp-news-and-widget/js/jquery.newstape.js?ver=3.2.8'></script>
<script type='text/javascript' src='http://cubacoopera.uccm.sld.cu/wp-content/plugins/wonderplugin-audio/engine/wonderpluginaudioskins.js?ver=6.5'></script>
<script type='text/javascript'>
/* <![CDATA[ */
var wonderplugin_audio_ajaxobject = {"ajaxurl":"http:\/\/cubacoopera.uccm.sld.cu\/wp-admin\/admin-ajax.php","nonce":"e71034c034"};
/* ]]> */
</script>
<script type='text/javascript' src='http://cubacoopera.uccm.sld.cu/wp-content/plugins/wonderplugin-audio/engine/wonderpluginaudio.js?ver=6.5'></script>
<script type='text/javascript' src='http://cubacoopera.uccm.sld.cu/wp-content/plugins/wonderplugin-gridgallery/engine/wonderplugingridlightbox.js?ver=8.4'></script>
<script type='text/javascript' src='http://cubacoopera.uccm.sld.cu/wp-content/plugins/wonderplugin-gridgallery/engine/wonderplugingridgallery.js?ver=8.4'></script>
<script type='text/javascript' src='http://cubacoopera.uccm.sld.cu/wp-content/plugins/wonderplugin-slider/engine/wonderpluginsliderskins.js?ver=8.5'></script>
<script type='text/javascript' src='http://cubacoopera.uccm.sld.cu/wp-content/plugins/wonderplugin-slider/engine/wonderpluginslider.js?ver=8.5'></script>
<script type='text/javascript' src='http://cubacoopera.uccm.sld.cu/wp-content/plugins/wonderplugin-video-embed/engine/wonderpluginvideoembed.js?ver=1.3'></script>
<!--[if lt IE 9]>
<script type='text/javascript' src='http://cubacoopera.uccm.sld.cu/wp-content/themes/magbook/js/html5.js?ver=3.7.3'></script>
<![endif]-->
<link rel='https://api.w.org/' href='http://cubacoopera.uccm.sld.cu/wp-json/' />
<link rel="EditURI" type="application/rsd+xml" title="RSD" href="http://cubacoopera.uccm.sld.cu/xmlrpc.php?rsd" />
<link rel="wlwmanifest" type="application/wlwmanifest+xml" href="http://cubacoopera.uccm.sld.cu/wp-includes/wlwmanifest.xml" /> 
<meta name="generator" content="WordPress 5.2.4" />
<meta name="generator" content="Custom Login v3.2.9" />

        <script>
            var wpdm_site_url = 'http://cubacoopera.uccm.sld.cu/';
            var wpdm_home_url = 'http://cubacoopera.uccm.sld.cu/';
            var ajax_url = 'http://cubacoopera.uccm.sld.cu/wp-admin/admin-ajax.php';
            var wpdm_ajax_url = 'http://cubacoopera.uccm.sld.cu/wp-admin/admin-ajax.php';
            var wpdm_ajax_popup = '0';
        </script>


        
		<script>
			(function(i,s,o,g,r,a,m){i['GoogleAnalyticsObject']=r;i[r]=i[r]||function(){
			(i[r].q=i[r].q||[]).push(arguments)},i[r].l=1*new Date();a=s.createElement(o),
			m=s.getElementsByTagName(o)[0];a.async=1;a.src=g;m.parentNode.insertBefore(a,m)
			})(window,document,'script','https://www.google-analytics.com/analytics.js','ga');
			ga('create', 'UA-144027962-1', 'auto');
			ga('send', 'pageview');
		</script>

	<style>.log_forms{ width:98%; padding:5px; border:0px solid #CCC; margin:2px; } .log_forms input[type=text], input[type=password] { margin: 10px 0 20px; width: 99%; padding: 7px 0 7px 4px; border: 1px solid #E3E3E3; } .log_forms input[type=submit] { margin: 10px 0 20px; width: 100%; padding: 7px; border: 1px solid #7ac9b7; } .log_forms input[type=text]:focus, input[type=password]:focus { border-color: #4697e4; }</style><!-- Analytics by WP-Statistics v12.6 - https://wp-statistics.com/ -->
	<meta name="viewport" content="width=device-width" />
			<style type="text/css">.recentcomments a{display:inline !important;padding:0 !important;margin:0 !important;}</style>
		<style type="text/css" id="custom-background-css">
body.custom-background { background-color: #c6c6c6; }
</style>
	<link rel="icon" href="http://cubacoopera.uccm.sld.cu/wp-content/uploads/2019/07/icono.png" sizes="32x32" />
<link rel="icon" href="http://cubacoopera.uccm.sld.cu/wp-content/uploads/2019/07/icono.png" sizes="192x192" />
<link rel="apple-touch-icon-precomposed" href="http://cubacoopera.uccm.sld.cu/wp-content/uploads/2019/07/icono.png" />
<meta name="msapplication-TileImage" content="http://cubacoopera.uccm.sld.cu/wp-content/uploads/2019/07/icono.png" />
<meta name="generator" content="WordPress Download Manager 2.9.90" />
            <style>
                


                .w3eden .fetfont,
                .w3eden .btn,
                .w3eden .btn.wpdm-front h3.title,
                .w3eden .wpdm-social-lock-box .IN-widget a span:last-child,
                .w3eden #xfilelist .panel-heading,
                .w3eden .wpdm-frontend-tabs a,
                .w3eden .alert:before,
                .w3eden .panel .panel-heading,
                .w3eden .discount-msg,
                .w3eden .panel.dashboard-panel h3,
                .w3eden #wpdm-dashboard-sidebar .list-group-item,
                .w3eden #package-description .wp-switch-editor,
                .w3eden .w3eden.author-dashbboard .nav.nav-tabs li a,
                .w3eden .wpdm_cart thead th,
                .w3eden #csp .list-group-item,
                .w3eden .modal-title {
                    font-family: Cantarell, -apple-system, BlinkMacSystemFont, "Segoe UI", Roboto, Helvetica, Arial, sans-serif, "Apple Color Emoji", "Segoe UI Emoji", "Segoe UI Symbol";
                    text-transform: uppercase;
                    font-weight: 700;
                }
                .w3eden #csp .list-group-item{
                    text-transform: unset;
                }
            </style>
                    <style>

            :root{
                --color-primary: #4a8eff;
                --color-primary-hover: #4a8eff;
                --color-primary-active: #4a8eff;
                --color-success: #18ce0f;
                --color-success-hover: #18ce0f;
                --color-success-active: #18ce0f;
                --color-info: #2CA8FF;
                --color-info-hover: #2CA8FF;
                --color-info-active: #2CA8FF;
                --color-warning: #FFB236;
                --color-warning-hover: #FFB236;
                --color-warning-active: #FFB236;
                --color-danger: #ff5062;
                --color-danger-hover: #ff5062;
                --color-danger-active: #ff5062;
                --color-green: #30b570;
                --color-blue: #0073ff;
                --color-purple: #8557D3;
                --color-red: #ff5062;
                --color-muted: rgba(69, 89, 122, 0.6);
                --wpdm-font: Cantarell, -apple-system, BlinkMacSystemFont, "Segoe UI", Roboto, Helvetica, Arial, sans-serif, "Apple Color Emoji", "Segoe UI Emoji", "Segoe UI Symbol";
            }


        </style>
        </head>
<body class="home blog custom-background wp-custom-logo cooltimeline-body boxed-layout two-column-blog n-sld top-logo-title">
<div id="page" class="site">
	<!-- Masthead ============================================= -->
	<header id="masthead" class="site-header">
		<div class="header-wrap">
						<!-- Top Header============================================= -->
			<div class="top-header">

									<div class="top-bar">
						<div class="wrap">
															<div class="top-bar-date">
									<span>miércoles, octubre 14, 2020</span>
								</div>
							<div class='the_champ_sharing_container the_champ_horizontal_sharing' super-socializer-data-href="http://cubacoopera.uccm.sld.cu"><aside id="thechamphorizontalsharing-4" class="widget widget_contact"><h3 class="widget-title">Share the joy</h3><ul class="the_champ_sharing_ul"><li class="theChampSharingRound"><i style="width:35px;height:35px;border-radius:999px;" alt="Facebook" Title="Facebook" class="theChampSharing theChampFacebookBackground" onclick='theChampPopup("https://www.facebook.com/sharer/sharer.php?u=http%3A%2F%2Fcubacoopera.uccm.sld.cu")'><ss style="display:block;border-radius:999px;" class="theChampSharingSvg theChampFacebookSvg"></ss></i></li><li class="theChampSharingRound"><i style="width:35px;height:35px;border-radius:999px;" alt="Twitter" Title="Twitter" class="theChampSharing theChampTwitterBackground" onclick='theChampPopup("http://twitter.com/intent/tweet?text=Cubacoopera%20-%20Portal%20web%20de%20la%20cooperaci%C3%B3n%20m%C3%A9dica%20cubana%2C%20pretende%20mostrar%20los%20principales%20resultados%20de%20la%20colaboraci%C3%B3n%20cubana%20en%20el%20exterior&url=http%3A%2F%2Fcubacoopera.uccm.sld.cu")'><ss style="display:block;border-radius:999px;" class="theChampSharingSvg theChampTwitterSvg"></ss></i></li><li class="theChampSharingRound"><i style="width:35px;height:35px;border-radius:999px;" alt="Linkedin" Title="Linkedin" class="theChampSharing theChampLinkedinBackground" onclick='theChampPopup("http://www.linkedin.com/shareArticle?mini=true&url=http%3A%2F%2Fcubacoopera.uccm.sld.cu&title=Cubacoopera%20-%20Portal%20web%20de%20la%20cooperaci%C3%B3n%20m%C3%A9dica%20cubana%2C%20pretende%20mostrar%20los%20principales%20resultados%20de%20la%20colaboraci%C3%B3n%20cubana%20en%20el%20exterior")'><ss style="display:block;border-radius:999px;" class="theChampSharingSvg theChampLinkedinSvg"></ss></i></li><li class="theChampSharingRound"><i style="width:35px;height:35px;border-radius:999px;" alt="Whatsapp" Title="Whatsapp" class="theChampSharing theChampWhatsappBackground"><a href="https://web.whatsapp.com/send?text=Cubacoopera%20-%20Portal%20web%20de%20la%20cooperaci%C3%B3n%20m%C3%A9dica%20cubana%2C%20pretende%20mostrar%20los%20principales%20resultados%20de%20la%20colaboraci%C3%B3n%20cubana%20en%20el%20exterior http%3A%2F%2Fcubacoopera.uccm.sld.cu" rel="nofollow noopener" target="_blank"><ss style="display:block" class="theChampSharingSvg theChampWhatsappSvg"></ss></a></i></li><li class="theChampSharingRound"><i style="width:35px;height:35px;border-radius:999px;" alt="Telegram" Title="Telegram" class="theChampSharing theChampTelegramBackground" onclick='theChampPopup("https://telegram.me/share/url?url=http%3A%2F%2Fcubacoopera.uccm.sld.cu&text=Cubacoopera%20-%20Portal%20web%20de%20la%20cooperaci%C3%B3n%20m%C3%A9dica%20cubana%2C%20pretende%20mostrar%20los%20principales%20resultados%20de%20la%20colaboraci%C3%B3n%20cubana%20en%20el%20exterior")'><ss style="display:block;border-radius:999px;" class="theChampSharingSvg theChampTelegramSvg"></ss></i></li><li class="theChampSharingRound"><i style="width:35px;height:35px;border-radius:999px;" alt="Google plus" Title="Google plus" class="theChampSharing theChampGoogleplusBackground" onclick='theChampPopup("https://plus.google.com/share?url=http%3A%2F%2Fcubacoopera.uccm.sld.cu")'><ss style="display:block;border-radius:999px;" class="theChampSharingSvg theChampGoogleplusSvg"></ss></i></li><li class="theChampSharingRound"><i style="width:35px;height:35px;border-radius:999px;" alt="Skype" Title="Skype" class="theChampSharing theChampSkypeBackground" onclick='theChampPopup("https://web.skype.com/share?url=http%3A%2F%2Fcubacoopera.uccm.sld.cu")'><ss style="display:block;border-radius:999px;" class="theChampSharingSvg theChampSkypeSvg"></ss></i></li><li class="theChampSharingRound"><i style="width:35px;height:35px;border-radius:999px;" title="More" alt="More" class="theChampSharing theChampMoreBackground" onclick="theChampMoreSharingPopup(this, 'http://cubacoopera.uccm.sld.cu', 'Cubacoopera%20-%20Portal%20web%20de%20la%20cooperaci%C3%B3n%20m%C3%A9dica%20cubana%2C%20pretende%20mostrar%20los%20principales%20resultados%20de%20la%20colaboraci%C3%B3n%20cubana%20en%20el%20exterior', '')" ><ss style="display:block" class="theChampSharingSvg theChampMoreSvg"></ss></i></li></ul><div style="clear:both"></div></div></aside>
						</div> <!-- end .wrap -->
					</div> <!-- end .top-bar -->
				
				<!-- Main Header============================================= -->
				<div class="logo-bar"> <div class="wrap"> <div id="site-branding"><a href="http://cubacoopera.uccm.sld.cu/" class="custom-logo-link" rel="home"><img width="907" height="194" src="http://cubacoopera.uccm.sld.cu/wp-content/uploads/2020/04/cropped-proyecto-2-Banner-Cubacoopera-covid-19.png" class="custom-logo" alt="Cubacoopera" srcset="http://cubacoopera.uccm.sld.cu/wp-content/uploads/2020/04/cropped-proyecto-2-Banner-Cubacoopera-covid-19.png 907w, http://cubacoopera.uccm.sld.cu/wp-content/uploads/2020/04/cropped-proyecto-2-Banner-Cubacoopera-covid-19-250x53.png 250w, http://cubacoopera.uccm.sld.cu/wp-content/uploads/2020/04/cropped-proyecto-2-Banner-Cubacoopera-covid-19-768x164.png 768w, http://cubacoopera.uccm.sld.cu/wp-content/uploads/2020/04/cropped-proyecto-2-Banner-Cubacoopera-covid-19-834x178.png 834w" sizes="(max-width: 907px) 100vw, 907px" /></a></div>		</div><!-- end .wrap -->
	</div><!-- end .logo-bar -->


				<div id="sticky-header" class="clearfix">
					<div class="wrap">
						<div class="main-header clearfix">

							<!-- Main Nav ============================================= -->
									<div id="site-branding">
			<a href="http://cubacoopera.uccm.sld.cu/" class="custom-logo-link" rel="home"><img width="907" height="194" src="http://cubacoopera.uccm.sld.cu/wp-content/uploads/2020/04/cropped-proyecto-2-Banner-Cubacoopera-covid-19.png" class="custom-logo" alt="Cubacoopera" srcset="http://cubacoopera.uccm.sld.cu/wp-content/uploads/2020/04/cropped-proyecto-2-Banner-Cubacoopera-covid-19.png 907w, http://cubacoopera.uccm.sld.cu/wp-content/uploads/2020/04/cropped-proyecto-2-Banner-Cubacoopera-covid-19-250x53.png 250w, http://cubacoopera.uccm.sld.cu/wp-content/uploads/2020/04/cropped-proyecto-2-Banner-Cubacoopera-covid-19-768x164.png 768w, http://cubacoopera.uccm.sld.cu/wp-content/uploads/2020/04/cropped-proyecto-2-Banner-Cubacoopera-covid-19-834x178.png 834w" sizes="(max-width: 907px) 100vw, 907px" /></a>		</div> <!-- end #site-branding -->
									<nav id="site-navigation" class="main-navigation clearfix" role="navigation">
																
									<button class="menu-toggle" aria-controls="primary-menu" aria-expanded="false">
										<span class="line-bar"></span>
									</button><!-- end .menu-toggle -->
									<ul id="primary-menu" class="menu nav-menu"><li id="menu-item-31475" class="menu-item menu-item-type-custom menu-item-object-custom current-menu-item current_page_item menu-item-home menu-item-31475"><a href="http://cubacoopera.uccm.sld.cu/" aria-current="page">Inicio</a></li>
<li id="menu-item-31476" class="menu-item menu-item-type-post_type menu-item-object-page menu-item-31476"><a href="http://cubacoopera.uccm.sld.cu/quienes-somos/">UCCM</a></li>
<li id="menu-item-157629" class="menu-item menu-item-type-post_type menu-item-object-page menu-item-has-children menu-item-157629"><a href="http://cubacoopera.uccm.sld.cu/red-social/brigadas-medicas-cubanas/">Brigadas Médicas</a>
<ul class="sub-menu">
	<li id="menu-item-31484" class="menu-item menu-item-type-post_type menu-item-object-page menu-item-31484"><a href="http://cubacoopera.uccm.sld.cu/datos-y-estadisticas/indicadores-de-servicio/">Principales Resultados</a></li>
</ul>
</li>
<li id="menu-item-163797" class="menu-item menu-item-type-post_type menu-item-object-page menu-item-has-children menu-item-163797"><a href="http://cubacoopera.uccm.sld.cu/documentos/">Documentos</a>
<ul class="sub-menu">
	<li id="menu-item-165818" class="menu-item menu-item-type-post_type menu-item-object-page menu-item-165818"><a href="http://cubacoopera.uccm.sld.cu/requisitos-de-publicacion-en-el-portal-cubacoopera/">Instructivo para publicaciones</a></li>
	<li id="menu-item-157935" class="menu-item menu-item-type-post_type menu-item-object-page menu-item-157935"><a href="http://cubacoopera.uccm.sld.cu/esinformacion-importante-para-el-colaborador/157920-2/">Boletín informativo sobre aspectos aduanales</a></li>
</ul>
</li>
<li id="menu-item-165022" class="menu-item menu-item-type-post_type menu-item-object-page menu-item-165022"><a href="http://cubacoopera.uccm.sld.cu/equipo-de-desarrollo/">Desarrolladores</a></li>
<li id="menu-item-159716" class="menu-item menu-item-type-custom menu-item-object-custom menu-item-159716"><a href="http://cubacoopera.uccm.sld.cu/news/mapa-del-sitio/">Mapa Sitio</a></li>
<li id="menu-item-161802" class="menu-item menu-item-type-custom menu-item-object-custom menu-item-161802"><a href="http://cubacoopera.uccm.sld.cu/161803-2/">Contacto</a></li>
</ul>								</nav> <!-- end #site-navigation -->
															<div id="search-toggle" class="header-search"></div>
								<div id="search-box" class="clearfix">
									<form class="search-form" action="http://cubacoopera.uccm.sld.cu/" method="get">
			<input type="search" name="s" class="search-field" placeholder="Buscar ..." autocomplete="off" />
		<button type="submit" class="search-submit"><i class="fa fa-search"></i></button>
</form> <!-- end .search-form -->								</div>  <!-- end #search-box -->
							
						</div><!-- end .main-header -->
					</div> <!-- end .wrap -->
				</div><!-- end #sticky-header -->

							</div><!-- end .top-header -->
					</div><!-- end .header-wrap -->

		<!-- Breaking News ============================================= -->
		
		<!-- Main Slider ============================================= -->
			</header> <!-- end #masthead -->

	<!-- Main Page Start ============================================= -->
	<div class="site-content-contain">
		<div id="content" class="site-content">
				<div class="wrap">
	<div id="primary" class="content-area">
		<main id="main" class="site-main">
			<header class="page-header">
				<h2 class="page-title"></h2>
				<!-- .page-title -->
				<!-- .breadcrumb -->
			</header><!-- .page-header -->
				<article id="post-191490" class="post-191490 post type-post status-publish format-standard has-post-thumbnail hentry category-contenidos category-uccm tag-cubacoopera-cubaporlasalud-cuba">
					<div class="post-image-content">
				<figure class="post-featured-image">
					<a href="http://cubacoopera.uccm.sld.cu/salva-vidas-donando-organos/" title="Salva vidas donando órganos">
					<img width="479" height="492" src="http://cubacoopera.uccm.sld.cu/wp-content/uploads/2020/10/Salva-vidas-donando-órganos.jpg" class="attachment-magbook-featured-blog size-magbook-featured-blog wp-post-image" alt="" srcset="http://cubacoopera.uccm.sld.cu/wp-content/uploads/2020/10/Salva-vidas-donando-órganos.jpg 479w, http://cubacoopera.uccm.sld.cu/wp-content/uploads/2020/10/Salva-vidas-donando-órganos-243x250.jpg 243w, http://cubacoopera.uccm.sld.cu/wp-content/uploads/2020/10/Salva-vidas-donando-órganos-429x441.jpg 429w" sizes="(max-width: 479px) 100vw, 479px" />					</a>
				</figure><!-- end.post-featured-image  -->
			</div><!-- end.post-image-content -->
					<header class="entry-header">
				 
					<div class="entry-meta">
							<span class="cats-links">
								<a class="cl-1731" href="http://cubacoopera.uccm.sld.cu/UCCM/contenidos/">Contenidos</a>
									<a class="cl-107" href="http://cubacoopera.uccm.sld.cu/UCCM/uccm/">UCCM</a>
				</span><!-- end .cat-links -->
					</div>
								<h2 class="entry-title"> <a href="http://cubacoopera.uccm.sld.cu/salva-vidas-donando-organos/" title="Salva vidas donando órganos"> Salva vidas donando órganos </a> </h2> <!-- end.entry-title -->

				<div class="entry-meta"><span class="author vcard"><a href="http://cubacoopera.uccm.sld.cu/author/cubacooperauccm/" title="Salva vidas donando órganos"><i class="fa fa-user-o"></i> Yanet Herrera Sierra</a></span><span class="posted-on"><a href="http://cubacoopera.uccm.sld.cu/salva-vidas-donando-organos/" title="14 octubre, 2020"><i class="fa fa-calendar-check-o"></i>14 octubre, 2020</a></span>							<span class="comments">
							<a href="http://cubacoopera.uccm.sld.cu/salva-vidas-donando-organos/#respond"><i class="fa fa-comments-o"></i> No Comments</a> </span>
												<span class="tag-links">
								<a href="http://cubacoopera.uccm.sld.cu/CubaCoopera/cubacoopera-cubaporlasalud-cuba/" rel="tag">CubaCoopera CubaPorLaSalud Cuba</a>							</span> <!-- end .tag-links -->
						</div> <!-- end .entry-meta -->			</header><!-- end .entry-header -->
			<div class="entry-content">
				<p>Desde el 2005 el 14 de octubre de cada año se conmemora el Día Mundial de la Donación de  Órganos, Tejidos y Trasplantes, ser donante salva vidas.</p>
<div style='clear: both'></div><div  class='the_champ_sharing_container the_champ_horizontal_sharing' super-socializer-data-href="http://cubacoopera.uccm.sld.cu/salva-vidas-donando-organos/"><div class='the_champ_sharing_title' style="font-weight:bold" >Compartir</div><ul class="the_champ_sharing_ul"><li class="theChampSharingRound"><i style="width:35px;height:35px;border-radius:999px;" alt="Facebook" Title="Facebook" class="theChampSharing theChampFacebookBackground" onclick='theChampPopup("https://www.facebook.com/sharer/sharer.php?u=http%3A%2F%2Fcubacoopera.uccm.sld.cu%2Fsalva-vidas-donando-organos%2F")'><ss style="display:block;border-radius:999px;" class="theChampSharingSvg theChampFacebookSvg"></ss><ss class="the_champ_square_count the_champ_facebook_count">&nbsp;</ss></i></li><li class="theChampSharingRound"><i style="width:35px;height:35px;border-radius:999px;" alt="Twitter" Title="Twitter" class="theChampSharing theChampTwitterBackground" onclick='theChampPopup("http://twitter.com/intent/tweet?text=Salva%20vidas%20donando%20%C3%B3rganos&url=http%3A%2F%2Fcubacoopera.uccm.sld.cu%2Fsalva-vidas-donando-organos%2F")'><ss style="display:block;border-radius:999px;" class="theChampSharingSvg theChampTwitterSvg"></ss><ss class="the_champ_square_count the_champ_twitter_count">&nbsp;</ss></i></li><li class="theChampSharingRound"><i style="width:35px;height:35px;border-radius:999px;" alt="Linkedin" Title="Linkedin" class="theChampSharing theChampLinkedinBackground" onclick='theChampPopup("http://www.linkedin.com/shareArticle?mini=true&url=http%3A%2F%2Fcubacoopera.uccm.sld.cu%2Fsalva-vidas-donando-organos%2F&title=Salva%20vidas%20donando%20%C3%B3rganos")'><ss style="display:block;border-radius:999px;" class="theChampSharingSvg theChampLinkedinSvg"></ss><ss class="the_champ_square_count the_champ_linkedin_count">&nbsp;</ss></i></li><li class="theChampSharingRound"><i style="width:35px;height:35px;border-radius:999px;" alt="Whatsapp" Title="Whatsapp" class="theChampSharing theChampWhatsappBackground"><a href="https://web.whatsapp.com/send?text=Salva%20vidas%20donando%20%C3%B3rganos http%3A%2F%2Fcubacoopera.uccm.sld.cu%2Fsalva-vidas-donando-organos%2F" rel="nofollow noopener" target="_blank"><ss style="display:block" class="theChampSharingSvg theChampWhatsappSvg"></ss><ss class="the_champ_square_count the_champ_whatsapp_count">&nbsp;</ss></a></i></li><li class="theChampSharingRound"><i style="width:35px;height:35px;border-radius:999px;" alt="Telegram" Title="Telegram" class="theChampSharing theChampTelegramBackground" onclick='theChampPopup("https://telegram.me/share/url?url=http%3A%2F%2Fcubacoopera.uccm.sld.cu%2Fsalva-vidas-donando-organos%2F&text=Salva%20vidas%20donando%20%C3%B3rganos")'><ss style="display:block;border-radius:999px;" class="theChampSharingSvg theChampTelegramSvg"></ss><ss class="the_champ_square_count the_champ_Telegram_count">&nbsp;</ss></i></li><li class="theChampSharingRound"><i style="width:35px;height:35px;border-radius:999px;" alt="Google plus" Title="Google plus" class="theChampSharing theChampGoogleplusBackground" onclick='theChampPopup("https://plus.google.com/share?url=http%3A%2F%2Fcubacoopera.uccm.sld.cu%2Fsalva-vidas-donando-organos%2F")'><ss style="display:block;border-radius:999px;" class="theChampSharingSvg theChampGoogleplusSvg"></ss><ss class="the_champ_square_count the_champ_google_plus_count">&nbsp;</ss></i></li><li class="theChampSharingRound"><i style="width:35px;height:35px;border-radius:999px;" alt="Skype" Title="Skype" class="theChampSharing theChampSkypeBackground" onclick='theChampPopup("https://web.skype.com/share?url=http%3A%2F%2Fcubacoopera.uccm.sld.cu%2Fsalva-vidas-donando-organos%2F")'><ss style="display:block;border-radius:999px;" class="theChampSharingSvg theChampSkypeSvg"></ss><ss class="the_champ_square_count the_champ_Skype_count">&nbsp;</ss></i></li><li class="theChampSharingRound"><ss class="the_champ_square_count">&nbsp;</ss><i style="width:35px;height:35px;border-radius:999px;" title="More" alt="More" class="theChampSharing theChampMoreBackground" onclick="theChampMoreSharingPopup(this, 'http://cubacoopera.uccm.sld.cu/salva-vidas-donando-organos/', 'Salva%20vidas%20donando%20%C3%B3rganos', '')" ><ss style="display:block" class="theChampSharingSvg theChampMoreSvg"></ss></i></li><li class="theChampSharingRound"><ss class="the_champ_square_count">&nbsp;</ss><i style="width:35px;height:35px;border-radius:999px;margin-left:9px !important;" title="Total Shares" alt="Total Shares" class="theChampSharing theChampTCBackground"></i></li></ul><div style="clear:both"></div></div><div style='clear: both'></div>					<a href="http://cubacoopera.uccm.sld.cu/salva-vidas-donando-organos/" class="more-link">Leer más...<span class="screen-reader-text"> Salva vidas donando órganos</span></a><!-- wp-default -->
								</div> <!-- end .entry-content -->
					</article><!-- end .post -->	<article id="post-191469" class="post-191469 post type-post status-publish format-standard has-post-thumbnail hentry category-bmc category-guinea_conakry tag-cuba tag-cubacoopera tag-cubaporlasalud tag-cubaporlavida">
					<div class="post-image-content">
				<figure class="post-featured-image">
					<a href="http://cubacoopera.uccm.sld.cu/logros-alcanzados-en-temas-de-salud/" title="Logros alcanzados en temas de salud">
					<img width="220" height="310" src="http://cubacoopera.uccm.sld.cu/wp-content/uploads/2020/10/121449456_352777602827750_5980798037519744570_n-1.jpg" class="attachment-magbook-featured-blog size-magbook-featured-blog wp-post-image" alt="" srcset="http://cubacoopera.uccm.sld.cu/wp-content/uploads/2020/10/121449456_352777602827750_5980798037519744570_n-1.jpg 220w, http://cubacoopera.uccm.sld.cu/wp-content/uploads/2020/10/121449456_352777602827750_5980798037519744570_n-1-177x250.jpg 177w" sizes="(max-width: 220px) 100vw, 220px" />					</a>
				</figure><!-- end.post-featured-image  -->
			</div><!-- end.post-image-content -->
					<header class="entry-header">
				 
					<div class="entry-meta">
							<span class="cats-links">
								<a class="cl-1620" href="http://cubacoopera.uccm.sld.cu/UCCM/bmc/">Brigadas Médicas Cubanas</a>
									<a class="cl-1635" href="http://cubacoopera.uccm.sld.cu/UCCM/bmc/guinea_conakry/">Guinea Conakry</a>
				</span><!-- end .cat-links -->
					</div>
								<h2 class="entry-title"> <a href="http://cubacoopera.uccm.sld.cu/logros-alcanzados-en-temas-de-salud/" title="Logros alcanzados en temas de salud"> Logros alcanzados en temas de salud </a> </h2> <!-- end.entry-title -->

				<div class="entry-meta"><span class="author vcard"><a href="http://cubacoopera.uccm.sld.cu/author/raisarp/" title="Logros alcanzados en temas de salud"><i class="fa fa-user-o"></i> Raisa Beatriz Rodríguez Pedro</a></span><span class="posted-on"><a href="http://cubacoopera.uccm.sld.cu/logros-alcanzados-en-temas-de-salud/" title="14 octubre, 2020"><i class="fa fa-calendar-check-o"></i>14 octubre, 2020</a></span>							<span class="comments">
							<a href="http://cubacoopera.uccm.sld.cu/logros-alcanzados-en-temas-de-salud/#respond"><i class="fa fa-comments-o"></i> No Comments</a> </span>
												<span class="tag-links">
								<a href="http://cubacoopera.uccm.sld.cu/CubaCoopera/cuba/" rel="tag">Cuba</a><a href="http://cubacoopera.uccm.sld.cu/CubaCoopera/cubacoopera/" rel="tag">Cubacoopera</a><a href="http://cubacoopera.uccm.sld.cu/CubaCoopera/cubaporlasalud/" rel="tag">CubaPorLaSalud</a><a href="http://cubacoopera.uccm.sld.cu/CubaCoopera/cubaporlavida/" rel="tag">CubaPorLaVida</a>							</span> <!-- end .tag-links -->
						</div> <!-- end .entry-meta -->			</header><!-- end .entry-header -->
			<div class="entry-content">
				<p>Nos sentimos realmente satisfechos con el trabajo realizado, sabiendo que estamos cumpliendo con la hermosa tarea de salvar vidas y dejando nuestra huella humilde en este hermano país, realmente no hay mayor satisfacción que la del deber cumplido.</p>
<div style='clear: both'></div><div  class='the_champ_sharing_container the_champ_horizontal_sharing' super-socializer-data-href="http://cubacoopera.uccm.sld.cu/logros-alcanzados-en-temas-de-salud/"><div class='the_champ_sharing_title' style="font-weight:bold" >Compartir</div><ul class="the_champ_sharing_ul"><li class="theChampSharingRound"><i style="width:35px;height:35px;border-radius:999px;" alt="Facebook" Title="Facebook" class="theChampSharing theChampFacebookBackground" onclick='theChampPopup("https://www.facebook.com/sharer/sharer.php?u=http%3A%2F%2Fcubacoopera.uccm.sld.cu%2Flogros-alcanzados-en-temas-de-salud%2F")'><ss style="display:block;border-radius:999px;" class="theChampSharingSvg theChampFacebookSvg"></ss><ss class="the_champ_square_count the_champ_facebook_count">&nbsp;</ss></i></li><li class="theChampSharingRound"><i style="width:35px;height:35px;border-radius:999px;" alt="Twitter" Title="Twitter" class="theChampSharing theChampTwitterBackground" onclick='theChampPopup("http://twitter.com/intent/tweet?text=Logros%20alcanzados%20en%20temas%20de%20salud&url=http%3A%2F%2Fcubacoopera.uccm.sld.cu%2Flogros-alcanzados-en-temas-de-salud%2F")'><ss style="display:block;border-radius:999px;" class="theChampSharingSvg theChampTwitterSvg"></ss><ss class="the_champ_square_count the_champ_twitter_count">&nbsp;</ss></i></li><li class="theChampSharingRound"><i style="width:35px;height:35px;border-radius:999px;" alt="Linkedin" Title="Linkedin" class="theChampSharing theChampLinkedinBackground" onclick='theChampPopup("http://www.linkedin.com/shareArticle?mini=true&url=http%3A%2F%2Fcubacoopera.uccm.sld.cu%2Flogros-alcanzados-en-temas-de-salud%2F&title=Logros%20alcanzados%20en%20temas%20de%20salud")'><ss style="display:block;border-radius:999px;" class="theChampSharingSvg theChampLinkedinSvg"></ss><ss class="the_champ_square_count the_champ_linkedin_count">&nbsp;</ss></i></li><li class="theChampSharingRound"><i style="width:35px;height:35px;border-radius:999px;" alt="Whatsapp" Title="Whatsapp" class="theChampSharing theChampWhatsappBackground"><a href="https://web.whatsapp.com/send?text=Logros%20alcanzados%20en%20temas%20de%20salud http%3A%2F%2Fcubacoopera.uccm.sld.cu%2Flogros-alcanzados-en-temas-de-salud%2F" rel="nofollow noopener" target="_blank"><ss style="display:block" class="theChampSharingSvg theChampWhatsappSvg"></ss><ss class="the_champ_square_count the_champ_whatsapp_count">&nbsp;</ss></a></i></li><li class="theChampSharingRound"><i style="width:35px;height:35px;border-radius:999px;" alt="Telegram" Title="Telegram" class="theChampSharing theChampTelegramBackground" onclick='theChampPopup("https://telegram.me/share/url?url=http%3A%2F%2Fcubacoopera.uccm.sld.cu%2Flogros-alcanzados-en-temas-de-salud%2F&text=Logros%20alcanzados%20en%20temas%20de%20salud")'><ss style="display:block;border-radius:999px;" class="theChampSharingSvg theChampTelegramSvg"></ss><ss class="the_champ_square_count the_champ_Telegram_count">&nbsp;</ss></i></li><li class="theChampSharingRound"><i style="width:35px;height:35px;border-radius:999px;" alt="Google plus" Title="Google plus" class="theChampSharing theChampGoogleplusBackground" onclick='theChampPopup("https://plus.google.com/share?url=http%3A%2F%2Fcubacoopera.uccm.sld.cu%2Flogros-alcanzados-en-temas-de-salud%2F")'><ss style="display:block;border-radius:999px;" class="theChampSharingSvg theChampGoogleplusSvg"></ss><ss class="the_champ_square_count the_champ_google_plus_count">&nbsp;</ss></i></li><li class="theChampSharingRound"><i style="width:35px;height:35px;border-radius:999px;" alt="Skype" Title="Skype" class="theChampSharing theChampSkypeBackground" onclick='theChampPopup("https://web.skype.com/share?url=http%3A%2F%2Fcubacoopera.uccm.sld.cu%2Flogros-alcanzados-en-temas-de-salud%2F")'><ss style="display:block;border-radius:999px;" class="theChampSharingSvg theChampSkypeSvg"></ss><ss class="the_champ_square_count the_champ_Skype_count">&nbsp;</ss></i></li><li class="theChampSharingRound"><ss class="the_champ_square_count">&nbsp;</ss><i style="width:35px;height:35px;border-radius:999px;" title="More" alt="More" class="theChampSharing theChampMoreBackground" onclick="theChampMoreSharingPopup(this, 'http://cubacoopera.uccm.sld.cu/logros-alcanzados-en-temas-de-salud/', 'Logros%20alcanzados%20en%20temas%20de%20salud', '')" ><ss style="display:block" class="theChampSharingSvg theChampMoreSvg"></ss></i></li><li class="theChampSharingRound"><ss class="the_champ_square_count">&nbsp;</ss><i style="width:35px;height:35px;border-radius:999px;margin-left:9px !important;" title="Total Shares" alt="Total Shares" class="theChampSharing theChampTCBackground"></i></li></ul><div style="clear:both"></div></div><div style='clear: both'></div>					<a href="http://cubacoopera.uccm.sld.cu/logros-alcanzados-en-temas-de-salud/" class="more-link">Leer más...<span class="screen-reader-text"> Logros alcanzados en temas de salud</span></a><!-- wp-default -->
								</div> <!-- end .entry-content -->
					</article><!-- end .post -->	<article id="post-191460" class="post-191460 post type-post status-publish format-standard has-post-thumbnail hentry category-bmc category-timor_leste tag-cubacoperacubaporlasaludcuba">
					<div class="post-image-content">
				<figure class="post-featured-image">
					<a href="http://cubacoopera.uccm.sld.cu/verdaderos-protagonistas-de-la-ensenanza-en-medicina/" title="Verdaderos protagonistas de la enseñanza en Medicina">
					<img width="960" height="512" src="http://cubacoopera.uccm.sld.cu/wp-content/uploads/2020/10/121094879_642217573149828_2537175291302288808_n-960x512.jpg" class="attachment-magbook-featured-blog size-magbook-featured-blog wp-post-image" alt="" />					</a>
				</figure><!-- end.post-featured-image  -->
			</div><!-- end.post-image-content -->
					<header class="entry-header">
				 
					<div class="entry-meta">
							<span class="cats-links">
								<a class="cl-1620" href="http://cubacoopera.uccm.sld.cu/UCCM/bmc/">Brigadas Médicas Cubanas</a>
									<a class="cl-1681" href="http://cubacoopera.uccm.sld.cu/UCCM/bmc/timor_leste/">Timor Leste</a>
				</span><!-- end .cat-links -->
					</div>
								<h2 class="entry-title"> <a href="http://cubacoopera.uccm.sld.cu/verdaderos-protagonistas-de-la-ensenanza-en-medicina/" title="Verdaderos protagonistas de la enseñanza en Medicina"> Verdaderos protagonistas de la enseñanza en Medicina </a> </h2> <!-- end.entry-title -->

				<div class="entry-meta"><span class="author vcard"><a href="http://cubacoopera.uccm.sld.cu/author/tsalgado/" title="Verdaderos protagonistas de la enseñanza en Medicina"><i class="fa fa-user-o"></i> Lic. Tania Salgado Grimón</a></span><span class="posted-on"><a href="http://cubacoopera.uccm.sld.cu/verdaderos-protagonistas-de-la-ensenanza-en-medicina/" title="14 octubre, 2020"><i class="fa fa-calendar-check-o"></i>14 octubre, 2020</a></span>							<span class="comments">
							<a href="http://cubacoopera.uccm.sld.cu/verdaderos-protagonistas-de-la-ensenanza-en-medicina/#respond"><i class="fa fa-comments-o"></i> No Comments</a> </span>
												<span class="tag-links">
								<a href="http://cubacoopera.uccm.sld.cu/CubaCoopera/cubacoperacubaporlasaludcuba/" rel="tag">CubaCoperaCubaPorLaSaludCuba</a>							</span> <!-- end .tag-links -->
						</div> <!-- end .entry-meta -->			</header><!-- end .entry-header -->
			<div class="entry-content">
				<p>Médicos y especialistas cubanos de la Salud curan y salvan a sus pacientes; también imparten clases a los estudiantes, verdaderos protagonistas de la enseñanza en Medicina. Como parte del proceso docente educativo, el pasado lunes se efectuó el II Encuentro de la Junta Nacional de Estudiantes, encaminado a la búsqueda de soluciones para continuar perfeccionando la calidad de la formación médica de los jóvenes en este país. </p>
<div style='clear: both'></div><div  class='the_champ_sharing_container the_champ_horizontal_sharing' super-socializer-data-href="http://cubacoopera.uccm.sld.cu/verdaderos-protagonistas-de-la-ensenanza-en-medicina/"><div class='the_champ_sharing_title' style="font-weight:bold" >Compartir</div><ul class="the_champ_sharing_ul"><li class="theChampSharingRound"><i style="width:35px;height:35px;border-radius:999px;" alt="Facebook" Title="Facebook" class="theChampSharing theChampFacebookBackground" onclick='theChampPopup("https://www.facebook.com/sharer/sharer.php?u=http%3A%2F%2Fcubacoopera.uccm.sld.cu%2Fverdaderos-protagonistas-de-la-ensenanza-en-medicina%2F")'><ss style="display:block;border-radius:999px;" class="theChampSharingSvg theChampFacebookSvg"></ss><ss class="the_champ_square_count the_champ_facebook_count">&nbsp;</ss></i></li><li class="theChampSharingRound"><i style="width:35px;height:35px;border-radius:999px;" alt="Twitter" Title="Twitter" class="theChampSharing theChampTwitterBackground" onclick='theChampPopup("http://twitter.com/intent/tweet?text=Verdaderos%20protagonistas%20de%20la%20ense%C3%B1anza%20en%20Medicina&url=http%3A%2F%2Fcubacoopera.uccm.sld.cu%2Fverdaderos-protagonistas-de-la-ensenanza-en-medicina%2F")'><ss style="display:block;border-radius:999px;" class="theChampSharingSvg theChampTwitterSvg"></ss><ss class="the_champ_square_count the_champ_twitter_count">&nbsp;</ss></i></li><li class="theChampSharingRound"><i style="width:35px;height:35px;border-radius:999px;" alt="Linkedin" Title="Linkedin" class="theChampSharing theChampLinkedinBackground" onclick='theChampPopup("http://www.linkedin.com/shareArticle?mini=true&url=http%3A%2F%2Fcubacoopera.uccm.sld.cu%2Fverdaderos-protagonistas-de-la-ensenanza-en-medicina%2F&title=Verdaderos%20protagonistas%20de%20la%20ense%C3%B1anza%20en%20Medicina")'><ss style="display:block;border-radius:999px;" class="theChampSharingSvg theChampLinkedinSvg"></ss><ss class="the_champ_square_count the_champ_linkedin_count">&nbsp;</ss></i></li><li class="theChampSharingRound"><i style="width:35px;height:35px;border-radius:999px;" alt="Whatsapp" Title="Whatsapp" class="theChampSharing theChampWhatsappBackground"><a href="https://web.whatsapp.com/send?text=Verdaderos%20protagonistas%20de%20la%20ense%C3%B1anza%20en%20Medicina http%3A%2F%2Fcubacoopera.uccm.sld.cu%2Fverdaderos-protagonistas-de-la-ensenanza-en-medicina%2F" rel="nofollow noopener" target="_blank"><ss style="display:block" class="theChampSharingSvg theChampWhatsappSvg"></ss><ss class="the_champ_square_count the_champ_whatsapp_count">&nbsp;</ss></a></i></li><li class="theChampSharingRound"><i style="width:35px;height:35px;border-radius:999px;" alt="Telegram" Title="Telegram" class="theChampSharing theChampTelegramBackground" onclick='theChampPopup("https://telegram.me/share/url?url=http%3A%2F%2Fcubacoopera.uccm.sld.cu%2Fverdaderos-protagonistas-de-la-ensenanza-en-medicina%2F&text=Verdaderos%20protagonistas%20de%20la%20ense%C3%B1anza%20en%20Medicina")'><ss style="display:block;border-radius:999px;" class="theChampSharingSvg theChampTelegramSvg"></ss><ss class="the_champ_square_count the_champ_Telegram_count">&nbsp;</ss></i></li><li class="theChampSharingRound"><i style="width:35px;height:35px;border-radius:999px;" alt="Google plus" Title="Google plus" class="theChampSharing theChampGoogleplusBackground" onclick='theChampPopup("https://plus.google.com/share?url=http%3A%2F%2Fcubacoopera.uccm.sld.cu%2Fverdaderos-protagonistas-de-la-ensenanza-en-medicina%2F")'><ss style="display:block;border-radius:999px;" class="theChampSharingSvg theChampGoogleplusSvg"></ss><ss class="the_champ_square_count the_champ_google_plus_count">&nbsp;</ss></i></li><li class="theChampSharingRound"><i style="width:35px;height:35px;border-radius:999px;" alt="Skype" Title="Skype" class="theChampSharing theChampSkypeBackground" onclick='theChampPopup("https://web.skype.com/share?url=http%3A%2F%2Fcubacoopera.uccm.sld.cu%2Fverdaderos-protagonistas-de-la-ensenanza-en-medicina%2F")'><ss style="display:block;border-radius:999px;" class="theChampSharingSvg theChampSkypeSvg"></ss><ss class="the_champ_square_count the_champ_Skype_count">&nbsp;</ss></i></li><li class="theChampSharingRound"><ss class="the_champ_square_count">&nbsp;</ss><i style="width:35px;height:35px;border-radius:999px;" title="More" alt="More" class="theChampSharing theChampMoreBackground" onclick="theChampMoreSharingPopup(this, 'http://cubacoopera.uccm.sld.cu/verdaderos-protagonistas-de-la-ensenanza-en-medicina/', 'Verdaderos%20protagonistas%20de%20la%20ense%C3%B1anza%20en%20Medicina', '')" ><ss style="display:block" class="theChampSharingSvg theChampMoreSvg"></ss></i></li><li class="theChampSharingRound"><ss class="the_champ_square_count">&nbsp;</ss><i style="width:35px;height:35px;border-radius:999px;margin-left:9px !important;" title="Total Shares" alt="Total Shares" class="theChampSharing theChampTCBackground"></i></li></ul><div style="clear:both"></div></div><div style='clear: both'></div>					<a href="http://cubacoopera.uccm.sld.cu/verdaderos-protagonistas-de-la-ensenanza-en-medicina/" class="more-link">Leer más...<span class="screen-reader-text"> Verdaderos protagonistas de la enseñanza en Medicina</span></a><!-- wp-default -->
								</div> <!-- end .entry-content -->
					</article><!-- end .post -->	<article id="post-191459" class="post-191459 post type-post status-publish format-standard has-post-thumbnail hentry category-brigadashr category-bmc category-guinea_bissau tag-cubacoopera-cubaporlasalud-cuba">
					<div class="post-image-content">
				<figure class="post-featured-image">
					<a href="http://cubacoopera.uccm.sld.cu/nuestro-homenaje-para-todos-los-medicos-cubanos/" title="Nuestro homenaje para todos los médicos cubanos">
					<img width="766" height="421" src="http://cubacoopera.uccm.sld.cu/wp-content/uploads/2020/10/medicosolid.jpg" class="attachment-magbook-featured-blog size-magbook-featured-blog wp-post-image" alt="" srcset="http://cubacoopera.uccm.sld.cu/wp-content/uploads/2020/10/medicosolid.jpg 766w, http://cubacoopera.uccm.sld.cu/wp-content/uploads/2020/10/medicosolid-250x137.jpg 250w" sizes="(max-width: 766px) 100vw, 766px" />					</a>
				</figure><!-- end.post-featured-image  -->
			</div><!-- end.post-image-content -->
					<header class="entry-header">
				 
					<div class="entry-meta">
							<span class="cats-links">
								<a class="cl-2028" href="http://cubacoopera.uccm.sld.cu/UCCM/brigadashr/">Brigadas Henry Reeve</a>
									<a class="cl-1620" href="http://cubacoopera.uccm.sld.cu/UCCM/bmc/">Brigadas Médicas Cubanas</a>
									<a class="cl-1634" href="http://cubacoopera.uccm.sld.cu/UCCM/bmc/guinea_bissau/">Guinea Bissau</a>
				</span><!-- end .cat-links -->
					</div>
								<h2 class="entry-title"> <a href="http://cubacoopera.uccm.sld.cu/nuestro-homenaje-para-todos-los-medicos-cubanos/" title="Nuestro homenaje para todos los médicos cubanos"> Nuestro homenaje para todos los médicos cubanos </a> </h2> <!-- end.entry-title -->

				<div class="entry-meta"><span class="author vcard"><a href="http://cubacoopera.uccm.sld.cu/author/mario/" title="Nuestro homenaje para todos los médicos cubanos"><i class="fa fa-user-o"></i> Mario Barrientos Rodriguez</a></span><span class="posted-on"><a href="http://cubacoopera.uccm.sld.cu/nuestro-homenaje-para-todos-los-medicos-cubanos/" title="14 octubre, 2020"><i class="fa fa-calendar-check-o"></i>14 octubre, 2020</a></span>							<span class="comments">
							<a href="http://cubacoopera.uccm.sld.cu/nuestro-homenaje-para-todos-los-medicos-cubanos/#respond"><i class="fa fa-comments-o"></i> No Comments</a> </span>
												<span class="tag-links">
								<a href="http://cubacoopera.uccm.sld.cu/CubaCoopera/cubacoopera-cubaporlasalud-cuba/" rel="tag">CubaCoopera CubaPorLaSalud Cuba</a>							</span> <!-- end .tag-links -->
						</div> <!-- end .entry-meta -->			</header><!-- end .entry-header -->
			<div class="entry-content">
				<p>El ejército de batas blancas continuará firme al legado de nuestro Comandante en Jefe, teniendo como máximas aquellas palabras suyas cuando dijo: «Basta ya de palabras, hacen falta hechos, y la diferencia entre los  pueblos debe cesar. Un mundo mejor es posible&#8230;», y eso es lo que  intentamos hacer todos los días: salvar y traer vidas a un mundo por el  que también luchamos para que sea un poco mejor. </p>
<div style='clear: both'></div><div  class='the_champ_sharing_container the_champ_horizontal_sharing' super-socializer-data-href="http://cubacoopera.uccm.sld.cu/nuestro-homenaje-para-todos-los-medicos-cubanos/"><div class='the_champ_sharing_title' style="font-weight:bold" >Compartir</div><ul class="the_champ_sharing_ul"><li class="theChampSharingRound"><i style="width:35px;height:35px;border-radius:999px;" alt="Facebook" Title="Facebook" class="theChampSharing theChampFacebookBackground" onclick='theChampPopup("https://www.facebook.com/sharer/sharer.php?u=http%3A%2F%2Fcubacoopera.uccm.sld.cu%2Fnuestro-homenaje-para-todos-los-medicos-cubanos%2F")'><ss style="display:block;border-radius:999px;" class="theChampSharingSvg theChampFacebookSvg"></ss><ss class="the_champ_square_count the_champ_facebook_count">&nbsp;</ss></i></li><li class="theChampSharingRound"><i style="width:35px;height:35px;border-radius:999px;" alt="Twitter" Title="Twitter" class="theChampSharing theChampTwitterBackground" onclick='theChampPopup("http://twitter.com/intent/tweet?text=Nuestro%20homenaje%20para%20todos%20los%20m%C3%A9dicos%20cubanos&url=http%3A%2F%2Fcubacoopera.uccm.sld.cu%2Fnuestro-homenaje-para-todos-los-medicos-cubanos%2F")'><ss style="display:block;border-radius:999px;" class="theChampSharingSvg theChampTwitterSvg"></ss><ss class="the_champ_square_count the_champ_twitter_count">&nbsp;</ss></i></li><li class="theChampSharingRound"><i style="width:35px;height:35px;border-radius:999px;" alt="Linkedin" Title="Linkedin" class="theChampSharing theChampLinkedinBackground" onclick='theChampPopup("http://www.linkedin.com/shareArticle?mini=true&url=http%3A%2F%2Fcubacoopera.uccm.sld.cu%2Fnuestro-homenaje-para-todos-los-medicos-cubanos%2F&title=Nuestro%20homenaje%20para%20todos%20los%20m%C3%A9dicos%20cubanos")'><ss style="display:block;border-radius:999px;" class="theChampSharingSvg theChampLinkedinSvg"></ss><ss class="the_champ_square_count the_champ_linkedin_count">&nbsp;</ss></i></li><li class="theChampSharingRound"><i style="width:35px;height:35px;border-radius:999px;" alt="Whatsapp" Title="Whatsapp" class="theChampSharing theChampWhatsappBackground"><a href="https://web.whatsapp.com/send?text=Nuestro%20homenaje%20para%20todos%20los%20m%C3%A9dicos%20cubanos http%3A%2F%2Fcubacoopera.uccm.sld.cu%2Fnuestro-homenaje-para-todos-los-medicos-cubanos%2F" rel="nofollow noopener" target="_blank"><ss style="display:block" class="theChampSharingSvg theChampWhatsappSvg"></ss><ss class="the_champ_square_count the_champ_whatsapp_count">&nbsp;</ss></a></i></li><li class="theChampSharingRound"><i style="width:35px;height:35px;border-radius:999px;" alt="Telegram" Title="Telegram" class="theChampSharing theChampTelegramBackground" onclick='theChampPopup("https://telegram.me/share/url?url=http%3A%2F%2Fcubacoopera.uccm.sld.cu%2Fnuestro-homenaje-para-todos-los-medicos-cubanos%2F&text=Nuestro%20homenaje%20para%20todos%20los%20m%C3%A9dicos%20cubanos")'><ss style="display:block;border-radius:999px;" class="theChampSharingSvg theChampTelegramSvg"></ss><ss class="the_champ_square_count the_champ_Telegram_count">&nbsp;</ss></i></li><li class="theChampSharingRound"><i style="width:35px;height:35px;border-radius:999px;" alt="Google plus" Title="Google plus" class="theChampSharing theChampGoogleplusBackground" onclick='theChampPopup("https://plus.google.com/share?url=http%3A%2F%2Fcubacoopera.uccm.sld.cu%2Fnuestro-homenaje-para-todos-los-medicos-cubanos%2F")'><ss style="display:block;border-radius:999px;" class="theChampSharingSvg theChampGoogleplusSvg"></ss><ss class="the_champ_square_count the_champ_google_plus_count">&nbsp;</ss></i></li><li class="theChampSharingRound"><i style="width:35px;height:35px;border-radius:999px;" alt="Skype" Title="Skype" class="theChampSharing theChampSkypeBackground" onclick='theChampPopup("https://web.skype.com/share?url=http%3A%2F%2Fcubacoopera.uccm.sld.cu%2Fnuestro-homenaje-para-todos-los-medicos-cubanos%2F")'><ss style="display:block;border-radius:999px;" class="theChampSharingSvg theChampSkypeSvg"></ss><ss class="the_champ_square_count the_champ_Skype_count">&nbsp;</ss></i></li><li class="theChampSharingRound"><ss class="the_champ_square_count">&nbsp;</ss><i style="width:35px;height:35px;border-radius:999px;" title="More" alt="More" class="theChampSharing theChampMoreBackground" onclick="theChampMoreSharingPopup(this, 'http://cubacoopera.uccm.sld.cu/nuestro-homenaje-para-todos-los-medicos-cubanos/', 'Nuestro%20homenaje%20para%20todos%20los%20m%C3%A9dicos%20cubanos', '')" ><ss style="display:block" class="theChampSharingSvg theChampMoreSvg"></ss></i></li><li class="theChampSharingRound"><ss class="the_champ_square_count">&nbsp;</ss><i style="width:35px;height:35px;border-radius:999px;margin-left:9px !important;" title="Total Shares" alt="Total Shares" class="theChampSharing theChampTCBackground"></i></li></ul><div style="clear:both"></div></div><div style='clear: both'></div>					<a href="http://cubacoopera.uccm.sld.cu/nuestro-homenaje-para-todos-los-medicos-cubanos/" class="more-link">Leer más...<span class="screen-reader-text"> Nuestro homenaje para todos los médicos cubanos</span></a><!-- wp-default -->
								</div> <!-- end .entry-content -->
					</article><!-- end .post -->	<article id="post-191434" class="post-191434 post type-post status-publish format-standard has-post-thumbnail hentry category-brigadashr category-coronavirus category-uccm tag-cubacoopera-cubaporlasalud-cuba">
					<div class="post-image-content">
				<figure class="post-featured-image">
					<a href="http://cubacoopera.uccm.sld.cu/premio-nobel-de-la-paz-para-henry-reeve/" title="Premio Nobel de la Paz para las Brigadas Henry Reeve">
					<img width="425" height="456" src="http://cubacoopera.uccm.sld.cu/wp-content/uploads/2020/10/Premio-Nobel-de-la-Paz-para-las-Brigadas-Henry-Reeve-1.jpg" class="attachment-magbook-featured-blog size-magbook-featured-blog wp-post-image" alt="" srcset="http://cubacoopera.uccm.sld.cu/wp-content/uploads/2020/10/Premio-Nobel-de-la-Paz-para-las-Brigadas-Henry-Reeve-1.jpg 425w, http://cubacoopera.uccm.sld.cu/wp-content/uploads/2020/10/Premio-Nobel-de-la-Paz-para-las-Brigadas-Henry-Reeve-1-233x250.jpg 233w, http://cubacoopera.uccm.sld.cu/wp-content/uploads/2020/10/Premio-Nobel-de-la-Paz-para-las-Brigadas-Henry-Reeve-1-411x441.jpg 411w" sizes="(max-width: 425px) 100vw, 425px" />					</a>
				</figure><!-- end.post-featured-image  -->
			</div><!-- end.post-image-content -->
					<header class="entry-header">
				 
					<div class="entry-meta">
							<span class="cats-links">
								<a class="cl-2028" href="http://cubacoopera.uccm.sld.cu/UCCM/brigadashr/">Brigadas Henry Reeve</a>
									<a class="cl-1802" href="http://cubacoopera.uccm.sld.cu/UCCM/coronavirus/">Coronavirus</a>
									<a class="cl-107" href="http://cubacoopera.uccm.sld.cu/UCCM/uccm/">UCCM</a>
				</span><!-- end .cat-links -->
					</div>
								<h2 class="entry-title"> <a href="http://cubacoopera.uccm.sld.cu/premio-nobel-de-la-paz-para-henry-reeve/" title="Premio Nobel de la Paz para las Brigadas Henry Reeve"> Premio Nobel de la Paz para las Brigadas Henry Reeve </a> </h2> <!-- end.entry-title -->

				<div class="entry-meta"><span class="author vcard"><a href="http://cubacoopera.uccm.sld.cu/author/cubacooperauccm/" title="Premio Nobel de la Paz para las Brigadas Henry Reeve"><i class="fa fa-user-o"></i> Yanet Herrera Sierra</a></span><span class="posted-on"><a href="http://cubacoopera.uccm.sld.cu/premio-nobel-de-la-paz-para-henry-reeve/" title="13 octubre, 2020"><i class="fa fa-calendar-check-o"></i>13 octubre, 2020</a></span>							<span class="comments">
							<a href="http://cubacoopera.uccm.sld.cu/premio-nobel-de-la-paz-para-henry-reeve/#respond"><i class="fa fa-comments-o"></i> No Comments</a> </span>
												<span class="tag-links">
								<a href="http://cubacoopera.uccm.sld.cu/CubaCoopera/cubacoopera-cubaporlasalud-cuba/" rel="tag">CubaCoopera CubaPorLaSalud Cuba</a>							</span> <!-- end .tag-links -->
						</div> <!-- end .entry-meta -->			</header><!-- end .entry-header -->
			<div class="entry-content">
				<p>Presenta oficialmente solicitud de Premio Nobel 2021 para las Brigadas Médicas Cubanas Henry Reeve el presidente del grupo parlamentario de amistad Francia-Cuba</p>
<div style='clear: both'></div><div  class='the_champ_sharing_container the_champ_horizontal_sharing' super-socializer-data-href="http://cubacoopera.uccm.sld.cu/premio-nobel-de-la-paz-para-henry-reeve/"><div class='the_champ_sharing_title' style="font-weight:bold" >Compartir</div><ul class="the_champ_sharing_ul"><li class="theChampSharingRound"><i style="width:35px;height:35px;border-radius:999px;" alt="Facebook" Title="Facebook" class="theChampSharing theChampFacebookBackground" onclick='theChampPopup("https://www.facebook.com/sharer/sharer.php?u=http%3A%2F%2Fcubacoopera.uccm.sld.cu%2Fpremio-nobel-de-la-paz-para-henry-reeve%2F")'><ss style="display:block;border-radius:999px;" class="theChampSharingSvg theChampFacebookSvg"></ss><ss class="the_champ_square_count the_champ_facebook_count">&nbsp;</ss></i></li><li class="theChampSharingRound"><i style="width:35px;height:35px;border-radius:999px;" alt="Twitter" Title="Twitter" class="theChampSharing theChampTwitterBackground" onclick='theChampPopup("http://twitter.com/intent/tweet?text=Premio%20Nobel%20de%20la%20Paz%20para%20las%20Brigadas%20Henry%20Reeve&url=http%3A%2F%2Fcubacoopera.uccm.sld.cu%2Fpremio-nobel-de-la-paz-para-henry-reeve%2F")'><ss style="display:block;border-radius:999px;" class="theChampSharingSvg theChampTwitterSvg"></ss><ss class="the_champ_square_count the_champ_twitter_count">&nbsp;</ss></i></li><li class="theChampSharingRound"><i style="width:35px;height:35px;border-radius:999px;" alt="Linkedin" Title="Linkedin" class="theChampSharing theChampLinkedinBackground" onclick='theChampPopup("http://www.linkedin.com/shareArticle?mini=true&url=http%3A%2F%2Fcubacoopera.uccm.sld.cu%2Fpremio-nobel-de-la-paz-para-henry-reeve%2F&title=Premio%20Nobel%20de%20la%20Paz%20para%20las%20Brigadas%20Henry%20Reeve")'><ss style="display:block;border-radius:999px;" class="theChampSharingSvg theChampLinkedinSvg"></ss><ss class="the_champ_square_count the_champ_linkedin_count">&nbsp;</ss></i></li><li class="theChampSharingRound"><i style="width:35px;height:35px;border-radius:999px;" alt="Whatsapp" Title="Whatsapp" class="theChampSharing theChampWhatsappBackground"><a href="https://web.whatsapp.com/send?text=Premio%20Nobel%20de%20la%20Paz%20para%20las%20Brigadas%20Henry%20Reeve http%3A%2F%2Fcubacoopera.uccm.sld.cu%2Fpremio-nobel-de-la-paz-para-henry-reeve%2F" rel="nofollow noopener" target="_blank"><ss style="display:block" class="theChampSharingSvg theChampWhatsappSvg"></ss><ss class="the_champ_square_count the_champ_whatsapp_count">&nbsp;</ss></a></i></li><li class="theChampSharingRound"><i style="width:35px;height:35px;border-radius:999px;" alt="Telegram" Title="Telegram" class="theChampSharing theChampTelegramBackground" onclick='theChampPopup("https://telegram.me/share/url?url=http%3A%2F%2Fcubacoopera.uccm.sld.cu%2Fpremio-nobel-de-la-paz-para-henry-reeve%2F&text=Premio%20Nobel%20de%20la%20Paz%20para%20las%20Brigadas%20Henry%20Reeve")'><ss style="display:block;border-radius:999px;" class="theChampSharingSvg theChampTelegramSvg"></ss><ss class="the_champ_square_count the_champ_Telegram_count">&nbsp;</ss></i></li><li class="theChampSharingRound"><i style="width:35px;height:35px;border-radius:999px;" alt="Google plus" Title="Google plus" class="theChampSharing theChampGoogleplusBackground" onclick='theChampPopup("https://plus.google.com/share?url=http%3A%2F%2Fcubacoopera.uccm.sld.cu%2Fpremio-nobel-de-la-paz-para-henry-reeve%2F")'><ss style="display:block;border-radius:999px;" class="theChampSharingSvg theChampGoogleplusSvg"></ss><ss class="the_champ_square_count the_champ_google_plus_count">&nbsp;</ss></i></li><li class="theChampSharingRound"><i style="width:35px;height:35px;border-radius:999px;" alt="Skype" Title="Skype" class="theChampSharing theChampSkypeBackground" onclick='theChampPopup("https://web.skype.com/share?url=http%3A%2F%2Fcubacoopera.uccm.sld.cu%2Fpremio-nobel-de-la-paz-para-henry-reeve%2F")'><ss style="display:block;border-radius:999px;" class="theChampSharingSvg theChampSkypeSvg"></ss><ss class="the_champ_square_count the_champ_Skype_count">&nbsp;</ss></i></li><li class="theChampSharingRound"><ss class="the_champ_square_count">&nbsp;</ss><i style="width:35px;height:35px;border-radius:999px;" title="More" alt="More" class="theChampSharing theChampMoreBackground" onclick="theChampMoreSharingPopup(this, 'http://cubacoopera.uccm.sld.cu/premio-nobel-de-la-paz-para-henry-reeve/', 'Premio%20Nobel%20de%20la%20Paz%20para%20las%20Brigadas%20Henry%20Reeve', '')" ><ss style="display:block" class="theChampSharingSvg theChampMoreSvg"></ss></i></li><li class="theChampSharingRound"><ss class="the_champ_square_count">&nbsp;</ss><i style="width:35px;height:35px;border-radius:999px;margin-left:9px !important;" title="Total Shares" alt="Total Shares" class="theChampSharing theChampTCBackground"></i></li></ul><div style="clear:both"></div></div><div style='clear: both'></div>					<a href="http://cubacoopera.uccm.sld.cu/premio-nobel-de-la-paz-para-henry-reeve/" class="more-link">Leer más...<span class="screen-reader-text"> Premio Nobel de la Paz para las Brigadas Henry Reeve</span></a><!-- wp-default -->
								</div> <!-- end .entry-content -->
					</article><!-- end .post -->	<article id="post-191433" class="post-191433 post type-post status-publish format-standard has-post-thumbnail hentry category-bmc category-timor_leste category-uccm tag-cubacoopera-cubaporlasalud-cuba">
					<div class="post-image-content">
				<figure class="post-featured-image">
					<a href="http://cubacoopera.uccm.sld.cu/prevenir-trombosis-salvar-vidas/" title="Prevenir Trombosis es salvar vidas">
					<img width="860" height="412" src="http://cubacoopera.uccm.sld.cu/wp-content/uploads/2020/10/dia-mundial-de-la-trombosis1.jpg" class="attachment-magbook-featured-blog size-magbook-featured-blog wp-post-image" alt="" srcset="http://cubacoopera.uccm.sld.cu/wp-content/uploads/2020/10/dia-mundial-de-la-trombosis1.jpg 860w, http://cubacoopera.uccm.sld.cu/wp-content/uploads/2020/10/dia-mundial-de-la-trombosis1-250x120.jpg 250w, http://cubacoopera.uccm.sld.cu/wp-content/uploads/2020/10/dia-mundial-de-la-trombosis1-768x368.jpg 768w, http://cubacoopera.uccm.sld.cu/wp-content/uploads/2020/10/dia-mundial-de-la-trombosis1-834x400.jpg 834w" sizes="(max-width: 860px) 100vw, 860px" />					</a>
				</figure><!-- end.post-featured-image  -->
			</div><!-- end.post-image-content -->
					<header class="entry-header">
				 
					<div class="entry-meta">
							<span class="cats-links">
								<a class="cl-1620" href="http://cubacoopera.uccm.sld.cu/UCCM/bmc/">Brigadas Médicas Cubanas</a>
									<a class="cl-1681" href="http://cubacoopera.uccm.sld.cu/UCCM/bmc/timor_leste/">Timor Leste</a>
									<a class="cl-107" href="http://cubacoopera.uccm.sld.cu/UCCM/uccm/">UCCM</a>
				</span><!-- end .cat-links -->
					</div>
								<h2 class="entry-title"> <a href="http://cubacoopera.uccm.sld.cu/prevenir-trombosis-salvar-vidas/" title="Prevenir Trombosis es salvar vidas"> Prevenir Trombosis es salvar vidas </a> </h2> <!-- end.entry-title -->

				<div class="entry-meta"><span class="author vcard"><a href="http://cubacoopera.uccm.sld.cu/author/yoyi/" title="Prevenir Trombosis es salvar vidas"><i class="fa fa-user-o"></i> Lic. Ognys González Hechavarría</a></span><span class="posted-on"><a href="http://cubacoopera.uccm.sld.cu/prevenir-trombosis-salvar-vidas/" title="13 octubre, 2020"><i class="fa fa-calendar-check-o"></i>13 octubre, 2020</a></span>							<span class="comments">
							<a href="http://cubacoopera.uccm.sld.cu/prevenir-trombosis-salvar-vidas/#respond"><i class="fa fa-comments-o"></i> No Comments</a> </span>
												<span class="tag-links">
								<a href="http://cubacoopera.uccm.sld.cu/CubaCoopera/cubacoopera-cubaporlasalud-cuba/" rel="tag">CubaCoopera CubaPorLaSalud Cuba</a>							</span> <!-- end .tag-links -->
						</div> <!-- end .entry-meta -->			</header><!-- end .entry-header -->
			<div class="entry-content">
				<p>13 de octubre, Día Mundial de la Trombosis, enfermedad que acarrea graves consecuenciasy altos niveles de morbilidad y mortalidad. Aumentar el conocimiento de la población mundial para prevenir la enfermedad y salvar vidas.</p>
<div style='clear: both'></div><div  class='the_champ_sharing_container the_champ_horizontal_sharing' super-socializer-data-href="http://cubacoopera.uccm.sld.cu/prevenir-trombosis-salvar-vidas/"><div class='the_champ_sharing_title' style="font-weight:bold" >Compartir</div><ul class="the_champ_sharing_ul"><li class="theChampSharingRound"><i style="width:35px;height:35px;border-radius:999px;" alt="Facebook" Title="Facebook" class="theChampSharing theChampFacebookBackground" onclick='theChampPopup("https://www.facebook.com/sharer/sharer.php?u=http%3A%2F%2Fcubacoopera.uccm.sld.cu%2Fprevenir-trombosis-salvar-vidas%2F")'><ss style="display:block;border-radius:999px;" class="theChampSharingSvg theChampFacebookSvg"></ss><ss class="the_champ_square_count the_champ_facebook_count">&nbsp;</ss></i></li><li class="theChampSharingRound"><i style="width:35px;height:35px;border-radius:999px;" alt="Twitter" Title="Twitter" class="theChampSharing theChampTwitterBackground" onclick='theChampPopup("http://twitter.com/intent/tweet?text=Prevenir%20Trombosis%20es%20salvar%20vidas&url=http%3A%2F%2Fcubacoopera.uccm.sld.cu%2Fprevenir-trombosis-salvar-vidas%2F")'><ss style="display:block;border-radius:999px;" class="theChampSharingSvg theChampTwitterSvg"></ss><ss class="the_champ_square_count the_champ_twitter_count">&nbsp;</ss></i></li><li class="theChampSharingRound"><i style="width:35px;height:35px;border-radius:999px;" alt="Linkedin" Title="Linkedin" class="theChampSharing theChampLinkedinBackground" onclick='theChampPopup("http://www.linkedin.com/shareArticle?mini=true&url=http%3A%2F%2Fcubacoopera.uccm.sld.cu%2Fprevenir-trombosis-salvar-vidas%2F&title=Prevenir%20Trombosis%20es%20salvar%20vidas")'><ss style="display:block;border-radius:999px;" class="theChampSharingSvg theChampLinkedinSvg"></ss><ss class="the_champ_square_count the_champ_linkedin_count">&nbsp;</ss></i></li><li class="theChampSharingRound"><i style="width:35px;height:35px;border-radius:999px;" alt="Whatsapp" Title="Whatsapp" class="theChampSharing theChampWhatsappBackground"><a href="https://web.whatsapp.com/send?text=Prevenir%20Trombosis%20es%20salvar%20vidas http%3A%2F%2Fcubacoopera.uccm.sld.cu%2Fprevenir-trombosis-salvar-vidas%2F" rel="nofollow noopener" target="_blank"><ss style="display:block" class="theChampSharingSvg theChampWhatsappSvg"></ss><ss class="the_champ_square_count the_champ_whatsapp_count">&nbsp;</ss></a></i></li><li class="theChampSharingRound"><i style="width:35px;height:35px;border-radius:999px;" alt="Telegram" Title="Telegram" class="theChampSharing theChampTelegramBackground" onclick='theChampPopup("https://telegram.me/share/url?url=http%3A%2F%2Fcubacoopera.uccm.sld.cu%2Fprevenir-trombosis-salvar-vidas%2F&text=Prevenir%20Trombosis%20es%20salvar%20vidas")'><ss style="display:block;border-radius:999px;" class="theChampSharingSvg theChampTelegramSvg"></ss><ss class="the_champ_square_count the_champ_Telegram_count">&nbsp;</ss></i></li><li class="theChampSharingRound"><i style="width:35px;height:35px;border-radius:999px;" alt="Google plus" Title="Google plus" class="theChampSharing theChampGoogleplusBackground" onclick='theChampPopup("https://plus.google.com/share?url=http%3A%2F%2Fcubacoopera.uccm.sld.cu%2Fprevenir-trombosis-salvar-vidas%2F")'><ss style="display:block;border-radius:999px;" class="theChampSharingSvg theChampGoogleplusSvg"></ss><ss class="the_champ_square_count the_champ_google_plus_count">&nbsp;</ss></i></li><li class="theChampSharingRound"><i style="width:35px;height:35px;border-radius:999px;" alt="Skype" Title="Skype" class="theChampSharing theChampSkypeBackground" onclick='theChampPopup("https://web.skype.com/share?url=http%3A%2F%2Fcubacoopera.uccm.sld.cu%2Fprevenir-trombosis-salvar-vidas%2F")'><ss style="display:block;border-radius:999px;" class="theChampSharingSvg theChampSkypeSvg"></ss><ss class="the_champ_square_count the_champ_Skype_count">&nbsp;</ss></i></li><li class="theChampSharingRound"><ss class="the_champ_square_count">&nbsp;</ss><i style="width:35px;height:35px;border-radius:999px;" title="More" alt="More" class="theChampSharing theChampMoreBackground" onclick="theChampMoreSharingPopup(this, 'http://cubacoopera.uccm.sld.cu/prevenir-trombosis-salvar-vidas/', 'Prevenir%20Trombosis%20es%20salvar%20vidas', '')" ><ss style="display:block" class="theChampSharingSvg theChampMoreSvg"></ss></i></li><li class="theChampSharingRound"><ss class="the_champ_square_count">&nbsp;</ss><i style="width:35px;height:35px;border-radius:999px;margin-left:9px !important;" title="Total Shares" alt="Total Shares" class="theChampSharing theChampTCBackground"></i></li></ul><div style="clear:both"></div></div><div style='clear: both'></div>					<a href="http://cubacoopera.uccm.sld.cu/prevenir-trombosis-salvar-vidas/" class="more-link">Leer más...<span class="screen-reader-text"> Prevenir Trombosis es salvar vidas</span></a><!-- wp-default -->
								</div> <!-- end .entry-content -->
					</article><!-- end .post -->	<article id="post-191392" class="post-191392 post type-post status-publish format-standard has-post-thumbnail hentry category-uccm tag-cubacoopera-cubaporlasalud-cuba">
					<div class="post-image-content">
				<figure class="post-featured-image">
					<a href="http://cubacoopera.uccm.sld.cu/artritis-reumatoide-es-una-enfermedad-de-padecimiento-cronico/" title="Artritis Reumatoide">
					<img width="1024" height="474" src="http://cubacoopera.uccm.sld.cu/wp-content/uploads/2020/10/dia-1024x474.jpg" class="attachment-magbook-featured-blog size-magbook-featured-blog wp-post-image" alt="" />					</a>
				</figure><!-- end.post-featured-image  -->
			</div><!-- end.post-image-content -->
					<header class="entry-header">
				 
					<div class="entry-meta">
							<span class="cats-links">
								<a class="cl-107" href="http://cubacoopera.uccm.sld.cu/UCCM/uccm/">UCCM</a>
				</span><!-- end .cat-links -->
					</div>
								<h2 class="entry-title"> <a href="http://cubacoopera.uccm.sld.cu/artritis-reumatoide-es-una-enfermedad-de-padecimiento-cronico/" title="Artritis Reumatoide"> Artritis Reumatoide </a> </h2> <!-- end.entry-title -->

				<div class="entry-meta"><span class="author vcard"><a href="http://cubacoopera.uccm.sld.cu/author/cubacooperauccm/" title="Artritis Reumatoide"><i class="fa fa-user-o"></i> Yanet Herrera Sierra</a></span><span class="posted-on"><a href="http://cubacoopera.uccm.sld.cu/artritis-reumatoide-es-una-enfermedad-de-padecimiento-cronico/" title="12 octubre, 2020"><i class="fa fa-calendar-check-o"></i>12 octubre, 2020</a></span>							<span class="comments">
							<a href="http://cubacoopera.uccm.sld.cu/artritis-reumatoide-es-una-enfermedad-de-padecimiento-cronico/#respond"><i class="fa fa-comments-o"></i> No Comments</a> </span>
												<span class="tag-links">
								<a href="http://cubacoopera.uccm.sld.cu/CubaCoopera/cubacoopera-cubaporlasalud-cuba/" rel="tag">CubaCoopera CubaPorLaSalud Cuba</a>							</span> <!-- end .tag-links -->
						</div> <!-- end .entry-meta -->			</header><!-- end .entry-header -->
			<div class="entry-content">
				<p>La  artritis reumatoide es una enfermedad que se caracteriza por la infamación de las articulaciones, es de padecimiento crónico, es decir que va empeorando con el tiempo y acarrea grandes dolores y la  perdida de movilidad del paciente.</p>
<div style='clear: both'></div><div  class='the_champ_sharing_container the_champ_horizontal_sharing' super-socializer-data-href="http://cubacoopera.uccm.sld.cu/artritis-reumatoide-es-una-enfermedad-de-padecimiento-cronico/"><div class='the_champ_sharing_title' style="font-weight:bold" >Compartir</div><ul class="the_champ_sharing_ul"><li class="theChampSharingRound"><i style="width:35px;height:35px;border-radius:999px;" alt="Facebook" Title="Facebook" class="theChampSharing theChampFacebookBackground" onclick='theChampPopup("https://www.facebook.com/sharer/sharer.php?u=http%3A%2F%2Fcubacoopera.uccm.sld.cu%2Fartritis-reumatoide-es-una-enfermedad-de-padecimiento-cronico%2F")'><ss style="display:block;border-radius:999px;" class="theChampSharingSvg theChampFacebookSvg"></ss><ss class="the_champ_square_count the_champ_facebook_count">&nbsp;</ss></i></li><li class="theChampSharingRound"><i style="width:35px;height:35px;border-radius:999px;" alt="Twitter" Title="Twitter" class="theChampSharing theChampTwitterBackground" onclick='theChampPopup("http://twitter.com/intent/tweet?text=Artritis%20Reumatoide&url=http%3A%2F%2Fcubacoopera.uccm.sld.cu%2Fartritis-reumatoide-es-una-enfermedad-de-padecimiento-cronico%2F")'><ss style="display:block;border-radius:999px;" class="theChampSharingSvg theChampTwitterSvg"></ss><ss class="the_champ_square_count the_champ_twitter_count">&nbsp;</ss></i></li><li class="theChampSharingRound"><i style="width:35px;height:35px;border-radius:999px;" alt="Linkedin" Title="Linkedin" class="theChampSharing theChampLinkedinBackground" onclick='theChampPopup("http://www.linkedin.com/shareArticle?mini=true&url=http%3A%2F%2Fcubacoopera.uccm.sld.cu%2Fartritis-reumatoide-es-una-enfermedad-de-padecimiento-cronico%2F&title=Artritis%20Reumatoide")'><ss style="display:block;border-radius:999px;" class="theChampSharingSvg theChampLinkedinSvg"></ss><ss class="the_champ_square_count the_champ_linkedin_count">&nbsp;</ss></i></li><li class="theChampSharingRound"><i style="width:35px;height:35px;border-radius:999px;" alt="Whatsapp" Title="Whatsapp" class="theChampSharing theChampWhatsappBackground"><a href="https://web.whatsapp.com/send?text=Artritis%20Reumatoide http%3A%2F%2Fcubacoopera.uccm.sld.cu%2Fartritis-reumatoide-es-una-enfermedad-de-padecimiento-cronico%2F" rel="nofollow noopener" target="_blank"><ss style="display:block" class="theChampSharingSvg theChampWhatsappSvg"></ss><ss class="the_champ_square_count the_champ_whatsapp_count">&nbsp;</ss></a></i></li><li class="theChampSharingRound"><i style="width:35px;height:35px;border-radius:999px;" alt="Telegram" Title="Telegram" class="theChampSharing theChampTelegramBackground" onclick='theChampPopup("https://telegram.me/share/url?url=http%3A%2F%2Fcubacoopera.uccm.sld.cu%2Fartritis-reumatoide-es-una-enfermedad-de-padecimiento-cronico%2F&text=Artritis%20Reumatoide")'><ss style="display:block;border-radius:999px;" class="theChampSharingSvg theChampTelegramSvg"></ss><ss class="the_champ_square_count the_champ_Telegram_count">&nbsp;</ss></i></li><li class="theChampSharingRound"><i style="width:35px;height:35px;border-radius:999px;" alt="Google plus" Title="Google plus" class="theChampSharing theChampGoogleplusBackground" onclick='theChampPopup("https://plus.google.com/share?url=http%3A%2F%2Fcubacoopera.uccm.sld.cu%2Fartritis-reumatoide-es-una-enfermedad-de-padecimiento-cronico%2F")'><ss style="display:block;border-radius:999px;" class="theChampSharingSvg theChampGoogleplusSvg"></ss><ss class="the_champ_square_count the_champ_google_plus_count">&nbsp;</ss></i></li><li class="theChampSharingRound"><i style="width:35px;height:35px;border-radius:999px;" alt="Skype" Title="Skype" class="theChampSharing theChampSkypeBackground" onclick='theChampPopup("https://web.skype.com/share?url=http%3A%2F%2Fcubacoopera.uccm.sld.cu%2Fartritis-reumatoide-es-una-enfermedad-de-padecimiento-cronico%2F")'><ss style="display:block;border-radius:999px;" class="theChampSharingSvg theChampSkypeSvg"></ss><ss class="the_champ_square_count the_champ_Skype_count">&nbsp;</ss></i></li><li class="theChampSharingRound"><ss class="the_champ_square_count">&nbsp;</ss><i style="width:35px;height:35px;border-radius:999px;" title="More" alt="More" class="theChampSharing theChampMoreBackground" onclick="theChampMoreSharingPopup(this, 'http://cubacoopera.uccm.sld.cu/artritis-reumatoide-es-una-enfermedad-de-padecimiento-cronico/', 'Artritis%20Reumatoide', '')" ><ss style="display:block" class="theChampSharingSvg theChampMoreSvg"></ss></i></li><li class="theChampSharingRound"><ss class="the_champ_square_count">&nbsp;</ss><i style="width:35px;height:35px;border-radius:999px;margin-left:9px !important;" title="Total Shares" alt="Total Shares" class="theChampSharing theChampTCBackground"></i></li></ul><div style="clear:both"></div></div><div style='clear: both'></div>					<a href="http://cubacoopera.uccm.sld.cu/artritis-reumatoide-es-una-enfermedad-de-padecimiento-cronico/" class="more-link">Leer más...<span class="screen-reader-text"> Artritis Reumatoide</span></a><!-- wp-default -->
								</div> <!-- end .entry-content -->
					</article><!-- end .post -->	<article id="post-191423" class="post-191423 post type-post status-publish format-standard has-post-thumbnail hentry category-bmc category-guinea_bissau category-historias_vida tag-cuba tag-cubacoopera tag-cubaporlasalud tag-cubaporlavida">
					<div class="post-image-content">
				<figure class="post-featured-image">
					<a href="http://cubacoopera.uccm.sld.cu/la-asignatura-medicina-legal-y-etica-medica/" title="La asignatura Medicina Legal y Ética Médica">
					<img width="270" height="138" src="http://cubacoopera.uccm.sld.cu/wp-content/uploads/2020/10/121318984_715621955710426_8112399297971867532_n.jpg" class="attachment-magbook-featured-blog size-magbook-featured-blog wp-post-image" alt="" srcset="http://cubacoopera.uccm.sld.cu/wp-content/uploads/2020/10/121318984_715621955710426_8112399297971867532_n.jpg 270w, http://cubacoopera.uccm.sld.cu/wp-content/uploads/2020/10/121318984_715621955710426_8112399297971867532_n-250x128.jpg 250w" sizes="(max-width: 270px) 100vw, 270px" />					</a>
				</figure><!-- end.post-featured-image  -->
			</div><!-- end.post-image-content -->
					<header class="entry-header">
				 
					<div class="entry-meta">
							<span class="cats-links">
								<a class="cl-1620" href="http://cubacoopera.uccm.sld.cu/UCCM/bmc/">Brigadas Médicas Cubanas</a>
									<a class="cl-1634" href="http://cubacoopera.uccm.sld.cu/UCCM/bmc/guinea_bissau/">Guinea Bissau</a>
									<a class="cl-1732" href="http://cubacoopera.uccm.sld.cu/UCCM/contenidos/historias_vida/">Historias de vida</a>
				</span><!-- end .cat-links -->
					</div>
								<h2 class="entry-title"> <a href="http://cubacoopera.uccm.sld.cu/la-asignatura-medicina-legal-y-etica-medica/" title="La asignatura Medicina Legal y Ética Médica"> La asignatura Medicina Legal y Ética Médica </a> </h2> <!-- end.entry-title -->

				<div class="entry-meta"><span class="author vcard"><a href="http://cubacoopera.uccm.sld.cu/author/raisarp/" title="La asignatura Medicina Legal y Ética Médica"><i class="fa fa-user-o"></i> Raisa Beatriz Rodríguez Pedro</a></span><span class="posted-on"><a href="http://cubacoopera.uccm.sld.cu/la-asignatura-medicina-legal-y-etica-medica/" title="12 octubre, 2020"><i class="fa fa-calendar-check-o"></i>12 octubre, 2020</a></span>							<span class="comments">
							<a href="http://cubacoopera.uccm.sld.cu/la-asignatura-medicina-legal-y-etica-medica/#respond"><i class="fa fa-comments-o"></i> No Comments</a> </span>
												<span class="tag-links">
								<a href="http://cubacoopera.uccm.sld.cu/CubaCoopera/cuba/" rel="tag">Cuba</a><a href="http://cubacoopera.uccm.sld.cu/CubaCoopera/cubacoopera/" rel="tag">Cubacoopera</a><a href="http://cubacoopera.uccm.sld.cu/CubaCoopera/cubaporlasalud/" rel="tag">CubaPorLaSalud</a><a href="http://cubacoopera.uccm.sld.cu/CubaCoopera/cubaporlavida/" rel="tag">CubaPorLaVida</a>							</span> <!-- end .tag-links -->
						</div> <!-- end .entry-meta -->			</header><!-- end .entry-header -->
			<div class="entry-content">
				<p>La práctica  médica se fundamenta en normas morales y legales que el Médico General  debe dominar para alcanzar un desempeño en su vida profesional sin  inconvenientes que resulten de infringirlas por ignorancia. Preparación  que se obtiene en la asignatura Medicina Legal y Ética Médica.</p>
<div style='clear: both'></div><div  class='the_champ_sharing_container the_champ_horizontal_sharing' super-socializer-data-href="http://cubacoopera.uccm.sld.cu/la-asignatura-medicina-legal-y-etica-medica/"><div class='the_champ_sharing_title' style="font-weight:bold" >Compartir</div><ul class="the_champ_sharing_ul"><li class="theChampSharingRound"><i style="width:35px;height:35px;border-radius:999px;" alt="Facebook" Title="Facebook" class="theChampSharing theChampFacebookBackground" onclick='theChampPopup("https://www.facebook.com/sharer/sharer.php?u=http%3A%2F%2Fcubacoopera.uccm.sld.cu%2Fla-asignatura-medicina-legal-y-etica-medica%2F")'><ss style="display:block;border-radius:999px;" class="theChampSharingSvg theChampFacebookSvg"></ss><ss class="the_champ_square_count the_champ_facebook_count">&nbsp;</ss></i></li><li class="theChampSharingRound"><i style="width:35px;height:35px;border-radius:999px;" alt="Twitter" Title="Twitter" class="theChampSharing theChampTwitterBackground" onclick='theChampPopup("http://twitter.com/intent/tweet?text=La%20asignatura%20Medicina%20Legal%20y%20%C3%89tica%20M%C3%A9dica&url=http%3A%2F%2Fcubacoopera.uccm.sld.cu%2Fla-asignatura-medicina-legal-y-etica-medica%2F")'><ss style="display:block;border-radius:999px;" class="theChampSharingSvg theChampTwitterSvg"></ss><ss class="the_champ_square_count the_champ_twitter_count">&nbsp;</ss></i></li><li class="theChampSharingRound"><i style="width:35px;height:35px;border-radius:999px;" alt="Linkedin" Title="Linkedin" class="theChampSharing theChampLinkedinBackground" onclick='theChampPopup("http://www.linkedin.com/shareArticle?mini=true&url=http%3A%2F%2Fcubacoopera.uccm.sld.cu%2Fla-asignatura-medicina-legal-y-etica-medica%2F&title=La%20asignatura%20Medicina%20Legal%20y%20%C3%89tica%20M%C3%A9dica")'><ss style="display:block;border-radius:999px;" class="theChampSharingSvg theChampLinkedinSvg"></ss><ss class="the_champ_square_count the_champ_linkedin_count">&nbsp;</ss></i></li><li class="theChampSharingRound"><i style="width:35px;height:35px;border-radius:999px;" alt="Whatsapp" Title="Whatsapp" class="theChampSharing theChampWhatsappBackground"><a href="https://web.whatsapp.com/send?text=La%20asignatura%20Medicina%20Legal%20y%20%C3%89tica%20M%C3%A9dica http%3A%2F%2Fcubacoopera.uccm.sld.cu%2Fla-asignatura-medicina-legal-y-etica-medica%2F" rel="nofollow noopener" target="_blank"><ss style="display:block" class="theChampSharingSvg theChampWhatsappSvg"></ss><ss class="the_champ_square_count the_champ_whatsapp_count">&nbsp;</ss></a></i></li><li class="theChampSharingRound"><i style="width:35px;height:35px;border-radius:999px;" alt="Telegram" Title="Telegram" class="theChampSharing theChampTelegramBackground" onclick='theChampPopup("https://telegram.me/share/url?url=http%3A%2F%2Fcubacoopera.uccm.sld.cu%2Fla-asignatura-medicina-legal-y-etica-medica%2F&text=La%20asignatura%20Medicina%20Legal%20y%20%C3%89tica%20M%C3%A9dica")'><ss style="display:block;border-radius:999px;" class="theChampSharingSvg theChampTelegramSvg"></ss><ss class="the_champ_square_count the_champ_Telegram_count">&nbsp;</ss></i></li><li class="theChampSharingRound"><i style="width:35px;height:35px;border-radius:999px;" alt="Google plus" Title="Google plus" class="theChampSharing theChampGoogleplusBackground" onclick='theChampPopup("https://plus.google.com/share?url=http%3A%2F%2Fcubacoopera.uccm.sld.cu%2Fla-asignatura-medicina-legal-y-etica-medica%2F")'><ss style="display:block;border-radius:999px;" class="theChampSharingSvg theChampGoogleplusSvg"></ss><ss class="the_champ_square_count the_champ_google_plus_count">&nbsp;</ss></i></li><li class="theChampSharingRound"><i style="width:35px;height:35px;border-radius:999px;" alt="Skype" Title="Skype" class="theChampSharing theChampSkypeBackground" onclick='theChampPopup("https://web.skype.com/share?url=http%3A%2F%2Fcubacoopera.uccm.sld.cu%2Fla-asignatura-medicina-legal-y-etica-medica%2F")'><ss style="display:block;border-radius:999px;" class="theChampSharingSvg theChampSkypeSvg"></ss><ss class="the_champ_square_count the_champ_Skype_count">&nbsp;</ss></i></li><li class="theChampSharingRound"><ss class="the_champ_square_count">&nbsp;</ss><i style="width:35px;height:35px;border-radius:999px;" title="More" alt="More" class="theChampSharing theChampMoreBackground" onclick="theChampMoreSharingPopup(this, 'http://cubacoopera.uccm.sld.cu/la-asignatura-medicina-legal-y-etica-medica/', 'La%20asignatura%20Medicina%20Legal%20y%20%C3%89tica%20M%C3%A9dica', '')" ><ss style="display:block" class="theChampSharingSvg theChampMoreSvg"></ss></i></li><li class="theChampSharingRound"><ss class="the_champ_square_count">&nbsp;</ss><i style="width:35px;height:35px;border-radius:999px;margin-left:9px !important;" title="Total Shares" alt="Total Shares" class="theChampSharing theChampTCBackground"></i></li></ul><div style="clear:both"></div></div><div style='clear: both'></div>					<a href="http://cubacoopera.uccm.sld.cu/la-asignatura-medicina-legal-y-etica-medica/" class="more-link">Leer más...<span class="screen-reader-text"> La asignatura Medicina Legal y Ética Médica</span></a><!-- wp-default -->
								</div> <!-- end .entry-content -->
					</article><!-- end .post -->	<article id="post-191386" class="post-191386 post type-post status-publish format-standard has-post-thumbnail hentry category-bmc category-caboverde tag-cubacoopera-cubaporlasalud-cuba">
					<div class="post-image-content">
				<figure class="post-featured-image">
					<a href="http://cubacoopera.uccm.sld.cu/dia-mundial-de-la-artritis-reumatoide/" title="Día Mundial de la Artritis Reumatoide">
					<img width="510" height="375" src="http://cubacoopera.uccm.sld.cu/wp-content/uploads/2020/10/3-1.jpg" class="attachment-magbook-featured-blog size-magbook-featured-blog wp-post-image" alt="" srcset="http://cubacoopera.uccm.sld.cu/wp-content/uploads/2020/10/3-1.jpg 510w, http://cubacoopera.uccm.sld.cu/wp-content/uploads/2020/10/3-1-250x184.jpg 250w" sizes="(max-width: 510px) 100vw, 510px" />					</a>
				</figure><!-- end.post-featured-image  -->
			</div><!-- end.post-image-content -->
					<header class="entry-header">
				 
					<div class="entry-meta">
							<span class="cats-links">
								<a class="cl-1620" href="http://cubacoopera.uccm.sld.cu/UCCM/bmc/">Brigadas Médicas Cubanas</a>
									<a class="cl-1625" href="http://cubacoopera.uccm.sld.cu/UCCM/bmc/caboverde/">Cabo Verde</a>
				</span><!-- end .cat-links -->
					</div>
								<h2 class="entry-title"> <a href="http://cubacoopera.uccm.sld.cu/dia-mundial-de-la-artritis-reumatoide/" title="Día Mundial de la Artritis Reumatoide"> Día Mundial de la Artritis Reumatoide </a> </h2> <!-- end.entry-title -->

				<div class="entry-meta"><span class="author vcard"><a href="http://cubacoopera.uccm.sld.cu/author/raisarp/" title="Día Mundial de la Artritis Reumatoide"><i class="fa fa-user-o"></i> Raisa Beatriz Rodríguez Pedro</a></span><span class="posted-on"><a href="http://cubacoopera.uccm.sld.cu/dia-mundial-de-la-artritis-reumatoide/" title="12 octubre, 2020"><i class="fa fa-calendar-check-o"></i>12 octubre, 2020</a></span>							<span class="comments">
							<a href="http://cubacoopera.uccm.sld.cu/dia-mundial-de-la-artritis-reumatoide/#respond"><i class="fa fa-comments-o"></i> No Comments</a> </span>
												<span class="tag-links">
								<a href="http://cubacoopera.uccm.sld.cu/CubaCoopera/cubacoopera-cubaporlasalud-cuba/" rel="tag">CubaCoopera CubaPorLaSalud Cuba</a>							</span> <!-- end .tag-links -->
						</div> <!-- end .entry-meta -->			</header><!-- end .entry-header -->
			<div class="entry-content">
				<p>El Día Mundial de la Artritis Reumatoide se institucionalizó  por la organización Arthritis Rheumatism International desde el año 1996, con el objetivo de educar a las personas sobre las enfermedades reumáticas y musculoesqueléticas, además de promover una mejor  calidad de vida y tratamientos para aquellos que las padecen.</p>
<div style='clear: both'></div><div  class='the_champ_sharing_container the_champ_horizontal_sharing' super-socializer-data-href="http://cubacoopera.uccm.sld.cu/dia-mundial-de-la-artritis-reumatoide/"><div class='the_champ_sharing_title' style="font-weight:bold" >Compartir</div><ul class="the_champ_sharing_ul"><li class="theChampSharingRound"><i style="width:35px;height:35px;border-radius:999px;" alt="Facebook" Title="Facebook" class="theChampSharing theChampFacebookBackground" onclick='theChampPopup("https://www.facebook.com/sharer/sharer.php?u=http%3A%2F%2Fcubacoopera.uccm.sld.cu%2Fdia-mundial-de-la-artritis-reumatoide%2F")'><ss style="display:block;border-radius:999px;" class="theChampSharingSvg theChampFacebookSvg"></ss><ss class="the_champ_square_count the_champ_facebook_count">&nbsp;</ss></i></li><li class="theChampSharingRound"><i style="width:35px;height:35px;border-radius:999px;" alt="Twitter" Title="Twitter" class="theChampSharing theChampTwitterBackground" onclick='theChampPopup("http://twitter.com/intent/tweet?text=D%C3%ADa%20Mundial%20de%20la%20Artritis%20Reumatoide&url=http%3A%2F%2Fcubacoopera.uccm.sld.cu%2Fdia-mundial-de-la-artritis-reumatoide%2F")'><ss style="display:block;border-radius:999px;" class="theChampSharingSvg theChampTwitterSvg"></ss><ss class="the_champ_square_count the_champ_twitter_count">&nbsp;</ss></i></li><li class="theChampSharingRound"><i style="width:35px;height:35px;border-radius:999px;" alt="Linkedin" Title="Linkedin" class="theChampSharing theChampLinkedinBackground" onclick='theChampPopup("http://www.linkedin.com/shareArticle?mini=true&url=http%3A%2F%2Fcubacoopera.uccm.sld.cu%2Fdia-mundial-de-la-artritis-reumatoide%2F&title=D%C3%ADa%20Mundial%20de%20la%20Artritis%20Reumatoide")'><ss style="display:block;border-radius:999px;" class="theChampSharingSvg theChampLinkedinSvg"></ss><ss class="the_champ_square_count the_champ_linkedin_count">&nbsp;</ss></i></li><li class="theChampSharingRound"><i style="width:35px;height:35px;border-radius:999px;" alt="Whatsapp" Title="Whatsapp" class="theChampSharing theChampWhatsappBackground"><a href="https://web.whatsapp.com/send?text=D%C3%ADa%20Mundial%20de%20la%20Artritis%20Reumatoide http%3A%2F%2Fcubacoopera.uccm.sld.cu%2Fdia-mundial-de-la-artritis-reumatoide%2F" rel="nofollow noopener" target="_blank"><ss style="display:block" class="theChampSharingSvg theChampWhatsappSvg"></ss><ss class="the_champ_square_count the_champ_whatsapp_count">&nbsp;</ss></a></i></li><li class="theChampSharingRound"><i style="width:35px;height:35px;border-radius:999px;" alt="Telegram" Title="Telegram" class="theChampSharing theChampTelegramBackground" onclick='theChampPopup("https://telegram.me/share/url?url=http%3A%2F%2Fcubacoopera.uccm.sld.cu%2Fdia-mundial-de-la-artritis-reumatoide%2F&text=D%C3%ADa%20Mundial%20de%20la%20Artritis%20Reumatoide")'><ss style="display:block;border-radius:999px;" class="theChampSharingSvg theChampTelegramSvg"></ss><ss class="the_champ_square_count the_champ_Telegram_count">&nbsp;</ss></i></li><li class="theChampSharingRound"><i style="width:35px;height:35px;border-radius:999px;" alt="Google plus" Title="Google plus" class="theChampSharing theChampGoogleplusBackground" onclick='theChampPopup("https://plus.google.com/share?url=http%3A%2F%2Fcubacoopera.uccm.sld.cu%2Fdia-mundial-de-la-artritis-reumatoide%2F")'><ss style="display:block;border-radius:999px;" class="theChampSharingSvg theChampGoogleplusSvg"></ss><ss class="the_champ_square_count the_champ_google_plus_count">&nbsp;</ss></i></li><li class="theChampSharingRound"><i style="width:35px;height:35px;border-radius:999px;" alt="Skype" Title="Skype" class="theChampSharing theChampSkypeBackground" onclick='theChampPopup("https://web.skype.com/share?url=http%3A%2F%2Fcubacoopera.uccm.sld.cu%2Fdia-mundial-de-la-artritis-reumatoide%2F")'><ss style="display:block;border-radius:999px;" class="theChampSharingSvg theChampSkypeSvg"></ss><ss class="the_champ_square_count the_champ_Skype_count">&nbsp;</ss></i></li><li class="theChampSharingRound"><ss class="the_champ_square_count">&nbsp;</ss><i style="width:35px;height:35px;border-radius:999px;" title="More" alt="More" class="theChampSharing theChampMoreBackground" onclick="theChampMoreSharingPopup(this, 'http://cubacoopera.uccm.sld.cu/dia-mundial-de-la-artritis-reumatoide/', 'D%C3%ADa%20Mundial%20de%20la%20Artritis%20Reumatoide', '')" ><ss style="display:block" class="theChampSharingSvg theChampMoreSvg"></ss></i></li><li class="theChampSharingRound"><ss class="the_champ_square_count">&nbsp;</ss><i style="width:35px;height:35px;border-radius:999px;margin-left:9px !important;" title="Total Shares" alt="Total Shares" class="theChampSharing theChampTCBackground"></i></li></ul><div style="clear:both"></div></div><div style='clear: both'></div>					<a href="http://cubacoopera.uccm.sld.cu/dia-mundial-de-la-artritis-reumatoide/" class="more-link">Leer más...<span class="screen-reader-text"> Día Mundial de la Artritis Reumatoide</span></a><!-- wp-default -->
								</div> <!-- end .entry-content -->
					</article><!-- end .post -->	<article id="post-191381" class="post-191381 post type-post status-publish format-standard has-post-thumbnail hentry category-belice tag-cubacoopera-cubaporlasalud-cuba">
					<div class="post-image-content">
				<figure class="post-featured-image">
					<a href="http://cubacoopera.uccm.sld.cu/el-dia-mundial-de-la-salud-mental-se-celebra-cada-diez-de-octubre/" title="Día Mundial de la Salud Mental">
					<img width="1024" height="512" src="http://cubacoopera.uccm.sld.cu/wp-content/uploads/2020/10/Diseño-sin-título-4-1024x512.jpg" class="attachment-magbook-featured-blog size-magbook-featured-blog wp-post-image" alt="" />					</a>
				</figure><!-- end.post-featured-image  -->
			</div><!-- end.post-image-content -->
					<header class="entry-header">
				 
					<div class="entry-meta">
							<span class="cats-links">
								<a class="cl-1652" href="http://cubacoopera.uccm.sld.cu/UCCM/bmc/belice/">Belice</a>
				</span><!-- end .cat-links -->
					</div>
								<h2 class="entry-title"> <a href="http://cubacoopera.uccm.sld.cu/el-dia-mundial-de-la-salud-mental-se-celebra-cada-diez-de-octubre/" title="Día Mundial de la Salud Mental"> Día Mundial de la Salud Mental </a> </h2> <!-- end.entry-title -->

				<div class="entry-meta"><span class="author vcard"><a href="http://cubacoopera.uccm.sld.cu/author/cubacooperauccm/" title="Día Mundial de la Salud Mental"><i class="fa fa-user-o"></i> Yanet Herrera Sierra</a></span><span class="posted-on"><a href="http://cubacoopera.uccm.sld.cu/el-dia-mundial-de-la-salud-mental-se-celebra-cada-diez-de-octubre/" title="12 octubre, 2020"><i class="fa fa-calendar-check-o"></i>12 octubre, 2020</a></span>							<span class="comments">
							<a href="http://cubacoopera.uccm.sld.cu/el-dia-mundial-de-la-salud-mental-se-celebra-cada-diez-de-octubre/#respond"><i class="fa fa-comments-o"></i> No Comments</a> </span>
												<span class="tag-links">
								<a href="http://cubacoopera.uccm.sld.cu/CubaCoopera/cubacoopera-cubaporlasalud-cuba/" rel="tag">CubaCoopera CubaPorLaSalud Cuba</a>							</span> <!-- end .tag-links -->
						</div> <!-- end .entry-meta -->			</header><!-- end .entry-header -->
			<div class="entry-content">
				<p>El  Día Mundial de la Salud Mental  se conmemora todos los 10 de octubre; el objetivo que persigue, es el de recordar que la salud de cada individuo es la sólida base para la construcción de vidas plenas y satisfactorias. </p>
<div style='clear: both'></div><div  class='the_champ_sharing_container the_champ_horizontal_sharing' super-socializer-data-href="http://cubacoopera.uccm.sld.cu/el-dia-mundial-de-la-salud-mental-se-celebra-cada-diez-de-octubre/"><div class='the_champ_sharing_title' style="font-weight:bold" >Compartir</div><ul class="the_champ_sharing_ul"><li class="theChampSharingRound"><i style="width:35px;height:35px;border-radius:999px;" alt="Facebook" Title="Facebook" class="theChampSharing theChampFacebookBackground" onclick='theChampPopup("https://www.facebook.com/sharer/sharer.php?u=http%3A%2F%2Fcubacoopera.uccm.sld.cu%2Fel-dia-mundial-de-la-salud-mental-se-celebra-cada-diez-de-octubre%2F")'><ss style="display:block;border-radius:999px;" class="theChampSharingSvg theChampFacebookSvg"></ss><ss class="the_champ_square_count the_champ_facebook_count">&nbsp;</ss></i></li><li class="theChampSharingRound"><i style="width:35px;height:35px;border-radius:999px;" alt="Twitter" Title="Twitter" class="theChampSharing theChampTwitterBackground" onclick='theChampPopup("http://twitter.com/intent/tweet?text=D%C3%ADa%20Mundial%20de%20la%20Salud%20Mental&url=http%3A%2F%2Fcubacoopera.uccm.sld.cu%2Fel-dia-mundial-de-la-salud-mental-se-celebra-cada-diez-de-octubre%2F")'><ss style="display:block;border-radius:999px;" class="theChampSharingSvg theChampTwitterSvg"></ss><ss class="the_champ_square_count the_champ_twitter_count">&nbsp;</ss></i></li><li class="theChampSharingRound"><i style="width:35px;height:35px;border-radius:999px;" alt="Linkedin" Title="Linkedin" class="theChampSharing theChampLinkedinBackground" onclick='theChampPopup("http://www.linkedin.com/shareArticle?mini=true&url=http%3A%2F%2Fcubacoopera.uccm.sld.cu%2Fel-dia-mundial-de-la-salud-mental-se-celebra-cada-diez-de-octubre%2F&title=D%C3%ADa%20Mundial%20de%20la%20Salud%20Mental")'><ss style="display:block;border-radius:999px;" class="theChampSharingSvg theChampLinkedinSvg"></ss><ss class="the_champ_square_count the_champ_linkedin_count">&nbsp;</ss></i></li><li class="theChampSharingRound"><i style="width:35px;height:35px;border-radius:999px;" alt="Whatsapp" Title="Whatsapp" class="theChampSharing theChampWhatsappBackground"><a href="https://web.whatsapp.com/send?text=D%C3%ADa%20Mundial%20de%20la%20Salud%20Mental http%3A%2F%2Fcubacoopera.uccm.sld.cu%2Fel-dia-mundial-de-la-salud-mental-se-celebra-cada-diez-de-octubre%2F" rel="nofollow noopener" target="_blank"><ss style="display:block" class="theChampSharingSvg theChampWhatsappSvg"></ss><ss class="the_champ_square_count the_champ_whatsapp_count">&nbsp;</ss></a></i></li><li class="theChampSharingRound"><i style="width:35px;height:35px;border-radius:999px;" alt="Telegram" Title="Telegram" class="theChampSharing theChampTelegramBackground" onclick='theChampPopup("https://telegram.me/share/url?url=http%3A%2F%2Fcubacoopera.uccm.sld.cu%2Fel-dia-mundial-de-la-salud-mental-se-celebra-cada-diez-de-octubre%2F&text=D%C3%ADa%20Mundial%20de%20la%20Salud%20Mental")'><ss style="display:block;border-radius:999px;" class="theChampSharingSvg theChampTelegramSvg"></ss><ss class="the_champ_square_count the_champ_Telegram_count">&nbsp;</ss></i></li><li class="theChampSharingRound"><i style="width:35px;height:35px;border-radius:999px;" alt="Google plus" Title="Google plus" class="theChampSharing theChampGoogleplusBackground" onclick='theChampPopup("https://plus.google.com/share?url=http%3A%2F%2Fcubacoopera.uccm.sld.cu%2Fel-dia-mundial-de-la-salud-mental-se-celebra-cada-diez-de-octubre%2F")'><ss style="display:block;border-radius:999px;" class="theChampSharingSvg theChampGoogleplusSvg"></ss><ss class="the_champ_square_count the_champ_google_plus_count">&nbsp;</ss></i></li><li class="theChampSharingRound"><i style="width:35px;height:35px;border-radius:999px;" alt="Skype" Title="Skype" class="theChampSharing theChampSkypeBackground" onclick='theChampPopup("https://web.skype.com/share?url=http%3A%2F%2Fcubacoopera.uccm.sld.cu%2Fel-dia-mundial-de-la-salud-mental-se-celebra-cada-diez-de-octubre%2F")'><ss style="display:block;border-radius:999px;" class="theChampSharingSvg theChampSkypeSvg"></ss><ss class="the_champ_square_count the_champ_Skype_count">&nbsp;</ss></i></li><li class="theChampSharingRound"><ss class="the_champ_square_count">&nbsp;</ss><i style="width:35px;height:35px;border-radius:999px;" title="More" alt="More" class="theChampSharing theChampMoreBackground" onclick="theChampMoreSharingPopup(this, 'http://cubacoopera.uccm.sld.cu/el-dia-mundial-de-la-salud-mental-se-celebra-cada-diez-de-octubre/', 'D%C3%ADa%20Mundial%20de%20la%20Salud%20Mental', '')" ><ss style="display:block" class="theChampSharingSvg theChampMoreSvg"></ss></i></li><li class="theChampSharingRound"><ss class="the_champ_square_count">&nbsp;</ss><i style="width:35px;height:35px;border-radius:999px;margin-left:9px !important;" title="Total Shares" alt="Total Shares" class="theChampSharing theChampTCBackground"></i></li></ul><div style="clear:both"></div></div><div style='clear: both'></div>					<a href="http://cubacoopera.uccm.sld.cu/el-dia-mundial-de-la-salud-mental-se-celebra-cada-diez-de-octubre/" class="more-link">Leer más...<span class="screen-reader-text"> Día Mundial de la Salud Mental</span></a><!-- wp-default -->
								</div> <!-- end .entry-content -->
					</article><!-- end .post -->	<article id="post-191363" class="post-191363 post type-post status-publish format-standard has-post-thumbnail hentry category-bmc category-guatemala category-pensamientos tag-cuba tag-cubacoopera tag-cubaporlasalud tag-cubaporlavida">
					<div class="post-image-content">
				<figure class="post-featured-image">
					<a href="http://cubacoopera.uccm.sld.cu/nuestro-amor-por-la-profesion/" title="Amor por la profesión">
					<img width="480" height="370" src="http://cubacoopera.uccm.sld.cu/wp-content/uploads/2020/10/121055299_1299012043809082_2907563764011048258_n-1.jpg" class="attachment-magbook-featured-blog size-magbook-featured-blog wp-post-image" alt="" srcset="http://cubacoopera.uccm.sld.cu/wp-content/uploads/2020/10/121055299_1299012043809082_2907563764011048258_n-1.jpg 480w, http://cubacoopera.uccm.sld.cu/wp-content/uploads/2020/10/121055299_1299012043809082_2907563764011048258_n-1-250x193.jpg 250w" sizes="(max-width: 480px) 100vw, 480px" />					</a>
				</figure><!-- end.post-featured-image  -->
			</div><!-- end.post-image-content -->
					<header class="entry-header">
				 
					<div class="entry-meta">
							<span class="cats-links">
								<a class="cl-1620" href="http://cubacoopera.uccm.sld.cu/UCCM/bmc/">Brigadas Médicas Cubanas</a>
									<a class="cl-1655" href="http://cubacoopera.uccm.sld.cu/UCCM/bmc/guatemala/">Guatemala</a>
									<a class="cl-1824" href="http://cubacoopera.uccm.sld.cu/UCCM/pensamientos/">Pensamientos internacionalista</a>
				</span><!-- end .cat-links -->
					</div>
								<h2 class="entry-title"> <a href="http://cubacoopera.uccm.sld.cu/nuestro-amor-por-la-profesion/" title="Amor por la profesión"> Amor por la profesión </a> </h2> <!-- end.entry-title -->

				<div class="entry-meta"><span class="author vcard"><a href="http://cubacoopera.uccm.sld.cu/author/raisarp/" title="Amor por la profesión"><i class="fa fa-user-o"></i> Raisa Beatriz Rodríguez Pedro</a></span><span class="posted-on"><a href="http://cubacoopera.uccm.sld.cu/nuestro-amor-por-la-profesion/" title="12 octubre, 2020"><i class="fa fa-calendar-check-o"></i>12 octubre, 2020</a></span>							<span class="comments">
							<a href="http://cubacoopera.uccm.sld.cu/nuestro-amor-por-la-profesion/#respond"><i class="fa fa-comments-o"></i> No Comments</a> </span>
												<span class="tag-links">
								<a href="http://cubacoopera.uccm.sld.cu/CubaCoopera/cuba/" rel="tag">Cuba</a><a href="http://cubacoopera.uccm.sld.cu/CubaCoopera/cubacoopera/" rel="tag">Cubacoopera</a><a href="http://cubacoopera.uccm.sld.cu/CubaCoopera/cubaporlasalud/" rel="tag">CubaPorLaSalud</a><a href="http://cubacoopera.uccm.sld.cu/CubaCoopera/cubaporlavida/" rel="tag">CubaPorLaVida</a>							</span> <!-- end .tag-links -->
						</div> <!-- end .entry-meta -->			</header><!-- end .entry-header -->
			<div class="entry-content">
				<p>Aliviar sufrimientos, prevenir  enfermedades, combatir pandemias, apoyar a los que sufren desastres  naturales, compartir todo nuestro conocimiento, y ayudar a los más  necesitados. Nadie nos obliga a hurgar en la ciencia buscando un futuro  mejor.</p>
<div style='clear: both'></div><div  class='the_champ_sharing_container the_champ_horizontal_sharing' super-socializer-data-href="http://cubacoopera.uccm.sld.cu/nuestro-amor-por-la-profesion/"><div class='the_champ_sharing_title' style="font-weight:bold" >Compartir</div><ul class="the_champ_sharing_ul"><li class="theChampSharingRound"><i style="width:35px;height:35px;border-radius:999px;" alt="Facebook" Title="Facebook" class="theChampSharing theChampFacebookBackground" onclick='theChampPopup("https://www.facebook.com/sharer/sharer.php?u=http%3A%2F%2Fcubacoopera.uccm.sld.cu%2Fnuestro-amor-por-la-profesion%2F")'><ss style="display:block;border-radius:999px;" class="theChampSharingSvg theChampFacebookSvg"></ss><ss class="the_champ_square_count the_champ_facebook_count">&nbsp;</ss></i></li><li class="theChampSharingRound"><i style="width:35px;height:35px;border-radius:999px;" alt="Twitter" Title="Twitter" class="theChampSharing theChampTwitterBackground" onclick='theChampPopup("http://twitter.com/intent/tweet?text=Amor%20por%20la%20profesi%C3%B3n&url=http%3A%2F%2Fcubacoopera.uccm.sld.cu%2Fnuestro-amor-por-la-profesion%2F")'><ss style="display:block;border-radius:999px;" class="theChampSharingSvg theChampTwitterSvg"></ss><ss class="the_champ_square_count the_champ_twitter_count">&nbsp;</ss></i></li><li class="theChampSharingRound"><i style="width:35px;height:35px;border-radius:999px;" alt="Linkedin" Title="Linkedin" class="theChampSharing theChampLinkedinBackground" onclick='theChampPopup("http://www.linkedin.com/shareArticle?mini=true&url=http%3A%2F%2Fcubacoopera.uccm.sld.cu%2Fnuestro-amor-por-la-profesion%2F&title=Amor%20por%20la%20profesi%C3%B3n")'><ss style="display:block;border-radius:999px;" class="theChampSharingSvg theChampLinkedinSvg"></ss><ss class="the_champ_square_count the_champ_linkedin_count">&nbsp;</ss></i></li><li class="theChampSharingRound"><i style="width:35px;height:35px;border-radius:999px;" alt="Whatsapp" Title="Whatsapp" class="theChampSharing theChampWhatsappBackground"><a href="https://web.whatsapp.com/send?text=Amor%20por%20la%20profesi%C3%B3n http%3A%2F%2Fcubacoopera.uccm.sld.cu%2Fnuestro-amor-por-la-profesion%2F" rel="nofollow noopener" target="_blank"><ss style="display:block" class="theChampSharingSvg theChampWhatsappSvg"></ss><ss class="the_champ_square_count the_champ_whatsapp_count">&nbsp;</ss></a></i></li><li class="theChampSharingRound"><i style="width:35px;height:35px;border-radius:999px;" alt="Telegram" Title="Telegram" class="theChampSharing theChampTelegramBackground" onclick='theChampPopup("https://telegram.me/share/url?url=http%3A%2F%2Fcubacoopera.uccm.sld.cu%2Fnuestro-amor-por-la-profesion%2F&text=Amor%20por%20la%20profesi%C3%B3n")'><ss style="display:block;border-radius:999px;" class="theChampSharingSvg theChampTelegramSvg"></ss><ss class="the_champ_square_count the_champ_Telegram_count">&nbsp;</ss></i></li><li class="theChampSharingRound"><i style="width:35px;height:35px;border-radius:999px;" alt="Google plus" Title="Google plus" class="theChampSharing theChampGoogleplusBackground" onclick='theChampPopup("https://plus.google.com/share?url=http%3A%2F%2Fcubacoopera.uccm.sld.cu%2Fnuestro-amor-por-la-profesion%2F")'><ss style="display:block;border-radius:999px;" class="theChampSharingSvg theChampGoogleplusSvg"></ss><ss class="the_champ_square_count the_champ_google_plus_count">&nbsp;</ss></i></li><li class="theChampSharingRound"><i style="width:35px;height:35px;border-radius:999px;" alt="Skype" Title="Skype" class="theChampSharing theChampSkypeBackground" onclick='theChampPopup("https://web.skype.com/share?url=http%3A%2F%2Fcubacoopera.uccm.sld.cu%2Fnuestro-amor-por-la-profesion%2F")'><ss style="display:block;border-radius:999px;" class="theChampSharingSvg theChampSkypeSvg"></ss><ss class="the_champ_square_count the_champ_Skype_count">&nbsp;</ss></i></li><li class="theChampSharingRound"><ss class="the_champ_square_count">&nbsp;</ss><i style="width:35px;height:35px;border-radius:999px;" title="More" alt="More" class="theChampSharing theChampMoreBackground" onclick="theChampMoreSharingPopup(this, 'http://cubacoopera.uccm.sld.cu/nuestro-amor-por-la-profesion/', 'Amor%20por%20la%20profesi%C3%B3n', '')" ><ss style="display:block" class="theChampSharingSvg theChampMoreSvg"></ss></i></li><li class="theChampSharingRound"><ss class="the_champ_square_count">&nbsp;</ss><i style="width:35px;height:35px;border-radius:999px;margin-left:9px !important;" title="Total Shares" alt="Total Shares" class="theChampSharing theChampTCBackground"></i></li></ul><div style="clear:both"></div></div><div style='clear: both'></div>					<a href="http://cubacoopera.uccm.sld.cu/nuestro-amor-por-la-profesion/" class="more-link">Leer más...<span class="screen-reader-text"> Amor por la profesión</span></a><!-- wp-default -->
								</div> <!-- end .entry-content -->
					</article><!-- end .post -->	<article id="post-191341" class="post-191341 post type-post status-publish format-standard has-post-thumbnail hentry category-bmc category-djibuti category-pensamientos tag-cuba tag-cubacoopera tag-cubaporlasalud tag-cubaporlavida">
					<div class="post-image-content">
				<figure class="post-featured-image">
					<a href="http://cubacoopera.uccm.sld.cu/pediatras-trabajan-con-mucha-entrega-y-amor/" title="Pediatras trabajan con mucha entrega y amor">
					<img width="174" height="235" src="http://cubacoopera.uccm.sld.cu/wp-content/uploads/2020/10/121258353_2006271896177135_2152090295267954902_n-4.jpg" class="attachment-magbook-featured-blog size-magbook-featured-blog wp-post-image" alt="" />					</a>
				</figure><!-- end.post-featured-image  -->
			</div><!-- end.post-image-content -->
					<header class="entry-header">
				 
					<div class="entry-meta">
							<span class="cats-links">
								<a class="cl-1620" href="http://cubacoopera.uccm.sld.cu/UCCM/bmc/">Brigadas Médicas Cubanas</a>
									<a class="cl-1628" href="http://cubacoopera.uccm.sld.cu/UCCM/bmc/djibuti/">Djibuti</a>
									<a class="cl-1824" href="http://cubacoopera.uccm.sld.cu/UCCM/pensamientos/">Pensamientos internacionalista</a>
				</span><!-- end .cat-links -->
					</div>
								<h2 class="entry-title"> <a href="http://cubacoopera.uccm.sld.cu/pediatras-trabajan-con-mucha-entrega-y-amor/" title="Pediatras trabajan con mucha entrega y amor"> Pediatras trabajan con mucha entrega y amor </a> </h2> <!-- end.entry-title -->

				<div class="entry-meta"><span class="author vcard"><a href="http://cubacoopera.uccm.sld.cu/author/raisarp/" title="Pediatras trabajan con mucha entrega y amor"><i class="fa fa-user-o"></i> Raisa Beatriz Rodríguez Pedro</a></span><span class="posted-on"><a href="http://cubacoopera.uccm.sld.cu/pediatras-trabajan-con-mucha-entrega-y-amor/" title="12 octubre, 2020"><i class="fa fa-calendar-check-o"></i>12 octubre, 2020</a></span>							<span class="comments">
							<a href="http://cubacoopera.uccm.sld.cu/pediatras-trabajan-con-mucha-entrega-y-amor/#respond"><i class="fa fa-comments-o"></i> No Comments</a> </span>
												<span class="tag-links">
								<a href="http://cubacoopera.uccm.sld.cu/CubaCoopera/cuba/" rel="tag">Cuba</a><a href="http://cubacoopera.uccm.sld.cu/CubaCoopera/cubacoopera/" rel="tag">Cubacoopera</a><a href="http://cubacoopera.uccm.sld.cu/CubaCoopera/cubaporlasalud/" rel="tag">CubaPorLaSalud</a><a href="http://cubacoopera.uccm.sld.cu/CubaCoopera/cubaporlavida/" rel="tag">CubaPorLaVida</a>							</span> <!-- end .tag-links -->
						</div> <!-- end .entry-meta -->			</header><!-- end .entry-header -->
			<div class="entry-content">
				<p>La Dra. María del  Carmen Mora y la Dra. Bárbara Morrell son pediatras que trabajan con  mucha entrega y amor con sus infantes abogando por acciones encaminadas a  la promoción y prevención de enfermedades que eventualmente pueden  afectar de corto a largo plazo el desarrollo integral del niño. </p>
<div style='clear: both'></div><div  class='the_champ_sharing_container the_champ_horizontal_sharing' super-socializer-data-href="http://cubacoopera.uccm.sld.cu/pediatras-trabajan-con-mucha-entrega-y-amor/"><div class='the_champ_sharing_title' style="font-weight:bold" >Compartir</div><ul class="the_champ_sharing_ul"><li class="theChampSharingRound"><i style="width:35px;height:35px;border-radius:999px;" alt="Facebook" Title="Facebook" class="theChampSharing theChampFacebookBackground" onclick='theChampPopup("https://www.facebook.com/sharer/sharer.php?u=http%3A%2F%2Fcubacoopera.uccm.sld.cu%2Fpediatras-trabajan-con-mucha-entrega-y-amor%2F")'><ss style="display:block;border-radius:999px;" class="theChampSharingSvg theChampFacebookSvg"></ss><ss class="the_champ_square_count the_champ_facebook_count">&nbsp;</ss></i></li><li class="theChampSharingRound"><i style="width:35px;height:35px;border-radius:999px;" alt="Twitter" Title="Twitter" class="theChampSharing theChampTwitterBackground" onclick='theChampPopup("http://twitter.com/intent/tweet?text=Pediatras%20trabajan%20con%20mucha%20entrega%20y%20amor&url=http%3A%2F%2Fcubacoopera.uccm.sld.cu%2Fpediatras-trabajan-con-mucha-entrega-y-amor%2F")'><ss style="display:block;border-radius:999px;" class="theChampSharingSvg theChampTwitterSvg"></ss><ss class="the_champ_square_count the_champ_twitter_count">&nbsp;</ss></i></li><li class="theChampSharingRound"><i style="width:35px;height:35px;border-radius:999px;" alt="Linkedin" Title="Linkedin" class="theChampSharing theChampLinkedinBackground" onclick='theChampPopup("http://www.linkedin.com/shareArticle?mini=true&url=http%3A%2F%2Fcubacoopera.uccm.sld.cu%2Fpediatras-trabajan-con-mucha-entrega-y-amor%2F&title=Pediatras%20trabajan%20con%20mucha%20entrega%20y%20amor")'><ss style="display:block;border-radius:999px;" class="theChampSharingSvg theChampLinkedinSvg"></ss><ss class="the_champ_square_count the_champ_linkedin_count">&nbsp;</ss></i></li><li class="theChampSharingRound"><i style="width:35px;height:35px;border-radius:999px;" alt="Whatsapp" Title="Whatsapp" class="theChampSharing theChampWhatsappBackground"><a href="https://web.whatsapp.com/send?text=Pediatras%20trabajan%20con%20mucha%20entrega%20y%20amor http%3A%2F%2Fcubacoopera.uccm.sld.cu%2Fpediatras-trabajan-con-mucha-entrega-y-amor%2F" rel="nofollow noopener" target="_blank"><ss style="display:block" class="theChampSharingSvg theChampWhatsappSvg"></ss><ss class="the_champ_square_count the_champ_whatsapp_count">&nbsp;</ss></a></i></li><li class="theChampSharingRound"><i style="width:35px;height:35px;border-radius:999px;" alt="Telegram" Title="Telegram" class="theChampSharing theChampTelegramBackground" onclick='theChampPopup("https://telegram.me/share/url?url=http%3A%2F%2Fcubacoopera.uccm.sld.cu%2Fpediatras-trabajan-con-mucha-entrega-y-amor%2F&text=Pediatras%20trabajan%20con%20mucha%20entrega%20y%20amor")'><ss style="display:block;border-radius:999px;" class="theChampSharingSvg theChampTelegramSvg"></ss><ss class="the_champ_square_count the_champ_Telegram_count">&nbsp;</ss></i></li><li class="theChampSharingRound"><i style="width:35px;height:35px;border-radius:999px;" alt="Google plus" Title="Google plus" class="theChampSharing theChampGoogleplusBackground" onclick='theChampPopup("https://plus.google.com/share?url=http%3A%2F%2Fcubacoopera.uccm.sld.cu%2Fpediatras-trabajan-con-mucha-entrega-y-amor%2F")'><ss style="display:block;border-radius:999px;" class="theChampSharingSvg theChampGoogleplusSvg"></ss><ss class="the_champ_square_count the_champ_google_plus_count">&nbsp;</ss></i></li><li class="theChampSharingRound"><i style="width:35px;height:35px;border-radius:999px;" alt="Skype" Title="Skype" class="theChampSharing theChampSkypeBackground" onclick='theChampPopup("https://web.skype.com/share?url=http%3A%2F%2Fcubacoopera.uccm.sld.cu%2Fpediatras-trabajan-con-mucha-entrega-y-amor%2F")'><ss style="display:block;border-radius:999px;" class="theChampSharingSvg theChampSkypeSvg"></ss><ss class="the_champ_square_count the_champ_Skype_count">&nbsp;</ss></i></li><li class="theChampSharingRound"><ss class="the_champ_square_count">&nbsp;</ss><i style="width:35px;height:35px;border-radius:999px;" title="More" alt="More" class="theChampSharing theChampMoreBackground" onclick="theChampMoreSharingPopup(this, 'http://cubacoopera.uccm.sld.cu/pediatras-trabajan-con-mucha-entrega-y-amor/', 'Pediatras%20trabajan%20con%20mucha%20entrega%20y%20amor', '')" ><ss style="display:block" class="theChampSharingSvg theChampMoreSvg"></ss></i></li><li class="theChampSharingRound"><ss class="the_champ_square_count">&nbsp;</ss><i style="width:35px;height:35px;border-radius:999px;margin-left:9px !important;" title="Total Shares" alt="Total Shares" class="theChampSharing theChampTCBackground"></i></li></ul><div style="clear:both"></div></div><div style='clear: both'></div>					<a href="http://cubacoopera.uccm.sld.cu/pediatras-trabajan-con-mucha-entrega-y-amor/" class="more-link">Leer más...<span class="screen-reader-text"> Pediatras trabajan con mucha entrega y amor</span></a><!-- wp-default -->
								</div> <!-- end .entry-content -->
					</article><!-- end .post -->	<article id="post-191336" class="post-191336 post type-post status-publish format-standard has-post-thumbnail hentry category-uccm tag-cubacoopera-cubaporlasalud-cuba">
					<div class="post-image-content">
				<figure class="post-featured-image">
					<a href="http://cubacoopera.uccm.sld.cu/responsabilidad-en-la-uccm-ante-la-covid-19/" title="Responsabilidad ante la COVID-19 en la UCCM">
					<img width="384" height="258" src="http://cubacoopera.uccm.sld.cu/wp-content/uploads/2020/10/4-1.jpg" class="attachment-magbook-featured-blog size-magbook-featured-blog wp-post-image" alt="" srcset="http://cubacoopera.uccm.sld.cu/wp-content/uploads/2020/10/4-1.jpg 384w, http://cubacoopera.uccm.sld.cu/wp-content/uploads/2020/10/4-1-250x168.jpg 250w, http://cubacoopera.uccm.sld.cu/wp-content/uploads/2020/10/4-1-272x182.jpg 272w" sizes="(max-width: 384px) 100vw, 384px" />					</a>
				</figure><!-- end.post-featured-image  -->
			</div><!-- end.post-image-content -->
					<header class="entry-header">
				 
					<div class="entry-meta">
							<span class="cats-links">
								<a class="cl-107" href="http://cubacoopera.uccm.sld.cu/UCCM/uccm/">UCCM</a>
				</span><!-- end .cat-links -->
					</div>
								<h2 class="entry-title"> <a href="http://cubacoopera.uccm.sld.cu/responsabilidad-en-la-uccm-ante-la-covid-19/" title="Responsabilidad ante la COVID-19 en la UCCM"> Responsabilidad ante la COVID-19 en la UCCM </a> </h2> <!-- end.entry-title -->

				<div class="entry-meta"><span class="author vcard"><a href="http://cubacoopera.uccm.sld.cu/author/tsalgado/" title="Responsabilidad ante la COVID-19 en la UCCM"><i class="fa fa-user-o"></i> Lic. Tania Salgado Grimón</a></span><span class="posted-on"><a href="http://cubacoopera.uccm.sld.cu/responsabilidad-en-la-uccm-ante-la-covid-19/" title="12 octubre, 2020"><i class="fa fa-calendar-check-o"></i>12 octubre, 2020</a></span>							<span class="comments">
							<a href="http://cubacoopera.uccm.sld.cu/responsabilidad-en-la-uccm-ante-la-covid-19/#respond"><i class="fa fa-comments-o"></i> No Comments</a> </span>
												<span class="tag-links">
								<a href="http://cubacoopera.uccm.sld.cu/CubaCoopera/cubacoopera-cubaporlasalud-cuba/" rel="tag">CubaCoopera CubaPorLaSalud Cuba</a>							</span> <!-- end .tag-links -->
						</div> <!-- end .entry-meta -->			</header><!-- end .entry-header -->
			<div class="entry-content">
				<p>En la Unidad Central de Colaboración Médica (UCCM) se ratifica que en la nueva etapa por la que transita Cuba es significativa la responsabilidad consciente y permanente de sus trabajadores ante la COVID-19; en especial para contribuir a la salud del pueblo y apoyar los esfuerzos del Estado. Se precisa ser disciplinado, respetar los protocolos médicos y las medidas higiénico-sanitarias. Cuidémonos por la circulación de la pandemia y de los asintomáticos; es trascendental la percepción del peligro sin alarmas o histerias.</p>
<div style='clear: both'></div><div  class='the_champ_sharing_container the_champ_horizontal_sharing' super-socializer-data-href="http://cubacoopera.uccm.sld.cu/responsabilidad-en-la-uccm-ante-la-covid-19/"><div class='the_champ_sharing_title' style="font-weight:bold" >Compartir</div><ul class="the_champ_sharing_ul"><li class="theChampSharingRound"><i style="width:35px;height:35px;border-radius:999px;" alt="Facebook" Title="Facebook" class="theChampSharing theChampFacebookBackground" onclick='theChampPopup("https://www.facebook.com/sharer/sharer.php?u=http%3A%2F%2Fcubacoopera.uccm.sld.cu%2Fresponsabilidad-en-la-uccm-ante-la-covid-19%2F")'><ss style="display:block;border-radius:999px;" class="theChampSharingSvg theChampFacebookSvg"></ss><ss class="the_champ_square_count the_champ_facebook_count">&nbsp;</ss></i></li><li class="theChampSharingRound"><i style="width:35px;height:35px;border-radius:999px;" alt="Twitter" Title="Twitter" class="theChampSharing theChampTwitterBackground" onclick='theChampPopup("http://twitter.com/intent/tweet?text=Responsabilidad%20ante%20la%20COVID-19%20en%20la%20UCCM&url=http%3A%2F%2Fcubacoopera.uccm.sld.cu%2Fresponsabilidad-en-la-uccm-ante-la-covid-19%2F")'><ss style="display:block;border-radius:999px;" class="theChampSharingSvg theChampTwitterSvg"></ss><ss class="the_champ_square_count the_champ_twitter_count">&nbsp;</ss></i></li><li class="theChampSharingRound"><i style="width:35px;height:35px;border-radius:999px;" alt="Linkedin" Title="Linkedin" class="theChampSharing theChampLinkedinBackground" onclick='theChampPopup("http://www.linkedin.com/shareArticle?mini=true&url=http%3A%2F%2Fcubacoopera.uccm.sld.cu%2Fresponsabilidad-en-la-uccm-ante-la-covid-19%2F&title=Responsabilidad%20ante%20la%20COVID-19%20en%20la%20UCCM")'><ss style="display:block;border-radius:999px;" class="theChampSharingSvg theChampLinkedinSvg"></ss><ss class="the_champ_square_count the_champ_linkedin_count">&nbsp;</ss></i></li><li class="theChampSharingRound"><i style="width:35px;height:35px;border-radius:999px;" alt="Whatsapp" Title="Whatsapp" class="theChampSharing theChampWhatsappBackground"><a href="https://web.whatsapp.com/send?text=Responsabilidad%20ante%20la%20COVID-19%20en%20la%20UCCM http%3A%2F%2Fcubacoopera.uccm.sld.cu%2Fresponsabilidad-en-la-uccm-ante-la-covid-19%2F" rel="nofollow noopener" target="_blank"><ss style="display:block" class="theChampSharingSvg theChampWhatsappSvg"></ss><ss class="the_champ_square_count the_champ_whatsapp_count">&nbsp;</ss></a></i></li><li class="theChampSharingRound"><i style="width:35px;height:35px;border-radius:999px;" alt="Telegram" Title="Telegram" class="theChampSharing theChampTelegramBackground" onclick='theChampPopup("https://telegram.me/share/url?url=http%3A%2F%2Fcubacoopera.uccm.sld.cu%2Fresponsabilidad-en-la-uccm-ante-la-covid-19%2F&text=Responsabilidad%20ante%20la%20COVID-19%20en%20la%20UCCM")'><ss style="display:block;border-radius:999px;" class="theChampSharingSvg theChampTelegramSvg"></ss><ss class="the_champ_square_count the_champ_Telegram_count">&nbsp;</ss></i></li><li class="theChampSharingRound"><i style="width:35px;height:35px;border-radius:999px;" alt="Google plus" Title="Google plus" class="theChampSharing theChampGoogleplusBackground" onclick='theChampPopup("https://plus.google.com/share?url=http%3A%2F%2Fcubacoopera.uccm.sld.cu%2Fresponsabilidad-en-la-uccm-ante-la-covid-19%2F")'><ss style="display:block;border-radius:999px;" class="theChampSharingSvg theChampGoogleplusSvg"></ss><ss class="the_champ_square_count the_champ_google_plus_count">&nbsp;</ss></i></li><li class="theChampSharingRound"><i style="width:35px;height:35px;border-radius:999px;" alt="Skype" Title="Skype" class="theChampSharing theChampSkypeBackground" onclick='theChampPopup("https://web.skype.com/share?url=http%3A%2F%2Fcubacoopera.uccm.sld.cu%2Fresponsabilidad-en-la-uccm-ante-la-covid-19%2F")'><ss style="display:block;border-radius:999px;" class="theChampSharingSvg theChampSkypeSvg"></ss><ss class="the_champ_square_count the_champ_Skype_count">&nbsp;</ss></i></li><li class="theChampSharingRound"><ss class="the_champ_square_count">&nbsp;</ss><i style="width:35px;height:35px;border-radius:999px;" title="More" alt="More" class="theChampSharing theChampMoreBackground" onclick="theChampMoreSharingPopup(this, 'http://cubacoopera.uccm.sld.cu/responsabilidad-en-la-uccm-ante-la-covid-19/', 'Responsabilidad%20ante%20la%20COVID-19%20en%20la%20UCCM', '')" ><ss style="display:block" class="theChampSharingSvg theChampMoreSvg"></ss></i></li><li class="theChampSharingRound"><ss class="the_champ_square_count">&nbsp;</ss><i style="width:35px;height:35px;border-radius:999px;margin-left:9px !important;" title="Total Shares" alt="Total Shares" class="theChampSharing theChampTCBackground"></i></li></ul><div style="clear:both"></div></div><div style='clear: both'></div>					<a href="http://cubacoopera.uccm.sld.cu/responsabilidad-en-la-uccm-ante-la-covid-19/" class="more-link">Leer más...<span class="screen-reader-text"> Responsabilidad ante la COVID-19 en la UCCM</span></a><!-- wp-default -->
								</div> <!-- end .entry-content -->
					</article><!-- end .post -->	<article id="post-191337" class="post-191337 post type-post status-publish format-standard has-post-thumbnail hentry category-bmc category-principales_resultados category-venezuela tag-cuba tag-cubacoopera tag-cubaporlasalud tag-cubaporlavida">
					<div class="post-image-content">
				<figure class="post-featured-image">
					<a href="http://cubacoopera.uccm.sld.cu/reconocimiento-a-los-profesionales-de-la-salud/" title="Reconocimiento a  los profesionales de la Salud">
					<img width="317" height="317" src="http://cubacoopera.uccm.sld.cu/wp-content/uploads/2020/10/121054155_1054503381651942_4672591384676288624_o-1.jpg" class="attachment-magbook-featured-blog size-magbook-featured-blog wp-post-image" alt="" srcset="http://cubacoopera.uccm.sld.cu/wp-content/uploads/2020/10/121054155_1054503381651942_4672591384676288624_o-1.jpg 317w, http://cubacoopera.uccm.sld.cu/wp-content/uploads/2020/10/121054155_1054503381651942_4672591384676288624_o-1-250x250.jpg 250w" sizes="(max-width: 317px) 100vw, 317px" />					</a>
				</figure><!-- end.post-featured-image  -->
			</div><!-- end.post-image-content -->
					<header class="entry-header">
				 
					<div class="entry-meta">
							<span class="cats-links">
								<a class="cl-1620" href="http://cubacoopera.uccm.sld.cu/UCCM/bmc/">Brigadas Médicas Cubanas</a>
									<a class="cl-1733" href="http://cubacoopera.uccm.sld.cu/UCCM/contenidos/principales_resultados/">Principales resultados</a>
									<a class="cl-1661" href="http://cubacoopera.uccm.sld.cu/UCCM/bmc/venezuela/">Venezuela</a>
				</span><!-- end .cat-links -->
					</div>
								<h2 class="entry-title"> <a href="http://cubacoopera.uccm.sld.cu/reconocimiento-a-los-profesionales-de-la-salud/" title="Reconocimiento a  los profesionales de la Salud"> Reconocimiento a  los profesionales de la Salud </a> </h2> <!-- end.entry-title -->

				<div class="entry-meta"><span class="author vcard"><a href="http://cubacoopera.uccm.sld.cu/author/raisarp/" title="Reconocimiento a  los profesionales de la Salud"><i class="fa fa-user-o"></i> Raisa Beatriz Rodríguez Pedro</a></span><span class="posted-on"><a href="http://cubacoopera.uccm.sld.cu/reconocimiento-a-los-profesionales-de-la-salud/" title="12 octubre, 2020"><i class="fa fa-calendar-check-o"></i>12 octubre, 2020</a></span>							<span class="comments">
							<a href="http://cubacoopera.uccm.sld.cu/reconocimiento-a-los-profesionales-de-la-salud/#respond"><i class="fa fa-comments-o"></i> No Comments</a> </span>
												<span class="tag-links">
								<a href="http://cubacoopera.uccm.sld.cu/CubaCoopera/cuba/" rel="tag">Cuba</a><a href="http://cubacoopera.uccm.sld.cu/CubaCoopera/cubacoopera/" rel="tag">Cubacoopera</a><a href="http://cubacoopera.uccm.sld.cu/CubaCoopera/cubaporlasalud/" rel="tag">CubaPorLaSalud</a><a href="http://cubacoopera.uccm.sld.cu/CubaCoopera/cubaporlavida/" rel="tag">CubaPorLaVida</a>							</span> <!-- end .tag-links -->
						</div> <!-- end .entry-meta -->			</header><!-- end .entry-header -->
			<div class="entry-content">
				<p>Médicos de #Cuba, valientes de la brigada que enfrenta la #COVID19 en el Hospital Intermedio de Campaña ubicado en el Poliedro de #Caracas,  recibe el sello Che Guevara por el día del médico y la medicina  integral comunitaria, el Dr. Luis Arley González, en nombre de todos los  profesionales. </p>
<div style='clear: both'></div><div  class='the_champ_sharing_container the_champ_horizontal_sharing' super-socializer-data-href="http://cubacoopera.uccm.sld.cu/reconocimiento-a-los-profesionales-de-la-salud/"><div class='the_champ_sharing_title' style="font-weight:bold" >Compartir</div><ul class="the_champ_sharing_ul"><li class="theChampSharingRound"><i style="width:35px;height:35px;border-radius:999px;" alt="Facebook" Title="Facebook" class="theChampSharing theChampFacebookBackground" onclick='theChampPopup("https://www.facebook.com/sharer/sharer.php?u=http%3A%2F%2Fcubacoopera.uccm.sld.cu%2Freconocimiento-a-los-profesionales-de-la-salud%2F")'><ss style="display:block;border-radius:999px;" class="theChampSharingSvg theChampFacebookSvg"></ss><ss class="the_champ_square_count the_champ_facebook_count">&nbsp;</ss></i></li><li class="theChampSharingRound"><i style="width:35px;height:35px;border-radius:999px;" alt="Twitter" Title="Twitter" class="theChampSharing theChampTwitterBackground" onclick='theChampPopup("http://twitter.com/intent/tweet?text=Reconocimiento%20a%20%20los%20profesionales%20de%20la%20Salud&url=http%3A%2F%2Fcubacoopera.uccm.sld.cu%2Freconocimiento-a-los-profesionales-de-la-salud%2F")'><ss style="display:block;border-radius:999px;" class="theChampSharingSvg theChampTwitterSvg"></ss><ss class="the_champ_square_count the_champ_twitter_count">&nbsp;</ss></i></li><li class="theChampSharingRound"><i style="width:35px;height:35px;border-radius:999px;" alt="Linkedin" Title="Linkedin" class="theChampSharing theChampLinkedinBackground" onclick='theChampPopup("http://www.linkedin.com/shareArticle?mini=true&url=http%3A%2F%2Fcubacoopera.uccm.sld.cu%2Freconocimiento-a-los-profesionales-de-la-salud%2F&title=Reconocimiento%20a%20%20los%20profesionales%20de%20la%20Salud")'><ss style="display:block;border-radius:999px;" class="theChampSharingSvg theChampLinkedinSvg"></ss><ss class="the_champ_square_count the_champ_linkedin_count">&nbsp;</ss></i></li><li class="theChampSharingRound"><i style="width:35px;height:35px;border-radius:999px;" alt="Whatsapp" Title="Whatsapp" class="theChampSharing theChampWhatsappBackground"><a href="https://web.whatsapp.com/send?text=Reconocimiento%20a%20%20los%20profesionales%20de%20la%20Salud http%3A%2F%2Fcubacoopera.uccm.sld.cu%2Freconocimiento-a-los-profesionales-de-la-salud%2F" rel="nofollow noopener" target="_blank"><ss style="display:block" class="theChampSharingSvg theChampWhatsappSvg"></ss><ss class="the_champ_square_count the_champ_whatsapp_count">&nbsp;</ss></a></i></li><li class="theChampSharingRound"><i style="width:35px;height:35px;border-radius:999px;" alt="Telegram" Title="Telegram" class="theChampSharing theChampTelegramBackground" onclick='theChampPopup("https://telegram.me/share/url?url=http%3A%2F%2Fcubacoopera.uccm.sld.cu%2Freconocimiento-a-los-profesionales-de-la-salud%2F&text=Reconocimiento%20a%20%20los%20profesionales%20de%20la%20Salud")'><ss style="display:block;border-radius:999px;" class="theChampSharingSvg theChampTelegramSvg"></ss><ss class="the_champ_square_count the_champ_Telegram_count">&nbsp;</ss></i></li><li class="theChampSharingRound"><i style="width:35px;height:35px;border-radius:999px;" alt="Google plus" Title="Google plus" class="theChampSharing theChampGoogleplusBackground" onclick='theChampPopup("https://plus.google.com/share?url=http%3A%2F%2Fcubacoopera.uccm.sld.cu%2Freconocimiento-a-los-profesionales-de-la-salud%2F")'><ss style="display:block;border-radius:999px;" class="theChampSharingSvg theChampGoogleplusSvg"></ss><ss class="the_champ_square_count the_champ_google_plus_count">&nbsp;</ss></i></li><li class="theChampSharingRound"><i style="width:35px;height:35px;border-radius:999px;" alt="Skype" Title="Skype" class="theChampSharing theChampSkypeBackground" onclick='theChampPopup("https://web.skype.com/share?url=http%3A%2F%2Fcubacoopera.uccm.sld.cu%2Freconocimiento-a-los-profesionales-de-la-salud%2F")'><ss style="display:block;border-radius:999px;" class="theChampSharingSvg theChampSkypeSvg"></ss><ss class="the_champ_square_count the_champ_Skype_count">&nbsp;</ss></i></li><li class="theChampSharingRound"><ss class="the_champ_square_count">&nbsp;</ss><i style="width:35px;height:35px;border-radius:999px;" title="More" alt="More" class="theChampSharing theChampMoreBackground" onclick="theChampMoreSharingPopup(this, 'http://cubacoopera.uccm.sld.cu/reconocimiento-a-los-profesionales-de-la-salud/', 'Reconocimiento%20a%20%20los%20profesionales%20de%20la%20Salud', '')" ><ss style="display:block" class="theChampSharingSvg theChampMoreSvg"></ss></i></li><li class="theChampSharingRound"><ss class="the_champ_square_count">&nbsp;</ss><i style="width:35px;height:35px;border-radius:999px;margin-left:9px !important;" title="Total Shares" alt="Total Shares" class="theChampSharing theChampTCBackground"></i></li></ul><div style="clear:both"></div></div><div style='clear: both'></div>					<a href="http://cubacoopera.uccm.sld.cu/reconocimiento-a-los-profesionales-de-la-salud/" class="more-link">Leer más...<span class="screen-reader-text"> Reconocimiento a  los profesionales de la Salud</span></a><!-- wp-default -->
								</div> <!-- end .entry-content -->
					</article><!-- end .post -->	<article id="post-191323" class="post-191323 post type-post status-publish format-standard has-post-thumbnail hentry category-uccm tag-cubacoopera-cubaporlasalud-cuba">
					<div class="post-image-content">
				<figure class="post-featured-image">
					<a href="http://cubacoopera.uccm.sld.cu/inicio-de-las-luchas-independentistas-en-cuba/" title="Inicio de las luchas independentistas en Cuba">
					<img width="265" height="155" src="http://cubacoopera.uccm.sld.cu/wp-content/uploads/2020/10/índice-2.jpg" class="attachment-magbook-featured-blog size-magbook-featured-blog wp-post-image" alt="" srcset="http://cubacoopera.uccm.sld.cu/wp-content/uploads/2020/10/índice-2.jpg 265w, http://cubacoopera.uccm.sld.cu/wp-content/uploads/2020/10/índice-2-250x146.jpg 250w" sizes="(max-width: 265px) 100vw, 265px" />					</a>
				</figure><!-- end.post-featured-image  -->
			</div><!-- end.post-image-content -->
					<header class="entry-header">
				 
					<div class="entry-meta">
							<span class="cats-links">
								<a class="cl-107" href="http://cubacoopera.uccm.sld.cu/UCCM/uccm/">UCCM</a>
				</span><!-- end .cat-links -->
					</div>
								<h2 class="entry-title"> <a href="http://cubacoopera.uccm.sld.cu/inicio-de-las-luchas-independentistas-en-cuba/" title="Inicio de las luchas independentistas en Cuba"> Inicio de las luchas independentistas en Cuba </a> </h2> <!-- end.entry-title -->

				<div class="entry-meta"><span class="author vcard"><a href="http://cubacoopera.uccm.sld.cu/author/mario/" title="Inicio de las luchas independentistas en Cuba"><i class="fa fa-user-o"></i> Mario Barrientos Rodriguez</a></span><span class="posted-on"><a href="http://cubacoopera.uccm.sld.cu/inicio-de-las-luchas-independentistas-en-cuba/" title="10 octubre, 2020"><i class="fa fa-calendar-check-o"></i>10 octubre, 2020</a></span>							<span class="comments">
							<a href="http://cubacoopera.uccm.sld.cu/inicio-de-las-luchas-independentistas-en-cuba/#respond"><i class="fa fa-comments-o"></i> No Comments</a> </span>
												<span class="tag-links">
								<a href="http://cubacoopera.uccm.sld.cu/CubaCoopera/cubacoopera-cubaporlasalud-cuba/" rel="tag">CubaCoopera CubaPorLaSalud Cuba</a>							</span> <!-- end .tag-links -->
						</div> <!-- end .entry-meta -->			</header><!-- end .entry-header -->
			<div class="entry-content">
				<p>Cada 10 de Octubre deviene ocasión especial para reafirmar el sentimiento patriótico y revolucionario que dio inicio a las luchas independentistas de Cuba. Ese hecho protagonizado por Carlos Manuel de Céspedes proporcionó un giro a la historia de Cuba y marcó la radicalización del pensamiento  revolucionario. Fue el inicio de un proceso independentista continuo hasta 1959 cuando el pueblo cubano logra definitiva y legítimamente su  libertad y soberanía.</p>
<div style='clear: both'></div><div  class='the_champ_sharing_container the_champ_horizontal_sharing' super-socializer-data-href="http://cubacoopera.uccm.sld.cu/inicio-de-las-luchas-independentistas-en-cuba/"><div class='the_champ_sharing_title' style="font-weight:bold" >Compartir</div><ul class="the_champ_sharing_ul"><li class="theChampSharingRound"><i style="width:35px;height:35px;border-radius:999px;" alt="Facebook" Title="Facebook" class="theChampSharing theChampFacebookBackground" onclick='theChampPopup("https://www.facebook.com/sharer/sharer.php?u=http%3A%2F%2Fcubacoopera.uccm.sld.cu%2Finicio-de-las-luchas-independentistas-en-cuba%2F")'><ss style="display:block;border-radius:999px;" class="theChampSharingSvg theChampFacebookSvg"></ss><ss class="the_champ_square_count the_champ_facebook_count">&nbsp;</ss></i></li><li class="theChampSharingRound"><i style="width:35px;height:35px;border-radius:999px;" alt="Twitter" Title="Twitter" class="theChampSharing theChampTwitterBackground" onclick='theChampPopup("http://twitter.com/intent/tweet?text=Inicio%20de%20las%20luchas%20independentistas%20en%20Cuba&url=http%3A%2F%2Fcubacoopera.uccm.sld.cu%2Finicio-de-las-luchas-independentistas-en-cuba%2F")'><ss style="display:block;border-radius:999px;" class="theChampSharingSvg theChampTwitterSvg"></ss><ss class="the_champ_square_count the_champ_twitter_count">&nbsp;</ss></i></li><li class="theChampSharingRound"><i style="width:35px;height:35px;border-radius:999px;" alt="Linkedin" Title="Linkedin" class="theChampSharing theChampLinkedinBackground" onclick='theChampPopup("http://www.linkedin.com/shareArticle?mini=true&url=http%3A%2F%2Fcubacoopera.uccm.sld.cu%2Finicio-de-las-luchas-independentistas-en-cuba%2F&title=Inicio%20de%20las%20luchas%20independentistas%20en%20Cuba")'><ss style="display:block;border-radius:999px;" class="theChampSharingSvg theChampLinkedinSvg"></ss><ss class="the_champ_square_count the_champ_linkedin_count">&nbsp;</ss></i></li><li class="theChampSharingRound"><i style="width:35px;height:35px;border-radius:999px;" alt="Whatsapp" Title="Whatsapp" class="theChampSharing theChampWhatsappBackground"><a href="https://web.whatsapp.com/send?text=Inicio%20de%20las%20luchas%20independentistas%20en%20Cuba http%3A%2F%2Fcubacoopera.uccm.sld.cu%2Finicio-de-las-luchas-independentistas-en-cuba%2F" rel="nofollow noopener" target="_blank"><ss style="display:block" class="theChampSharingSvg theChampWhatsappSvg"></ss><ss class="the_champ_square_count the_champ_whatsapp_count">&nbsp;</ss></a></i></li><li class="theChampSharingRound"><i style="width:35px;height:35px;border-radius:999px;" alt="Telegram" Title="Telegram" class="theChampSharing theChampTelegramBackground" onclick='theChampPopup("https://telegram.me/share/url?url=http%3A%2F%2Fcubacoopera.uccm.sld.cu%2Finicio-de-las-luchas-independentistas-en-cuba%2F&text=Inicio%20de%20las%20luchas%20independentistas%20en%20Cuba")'><ss style="display:block;border-radius:999px;" class="theChampSharingSvg theChampTelegramSvg"></ss><ss class="the_champ_square_count the_champ_Telegram_count">&nbsp;</ss></i></li><li class="theChampSharingRound"><i style="width:35px;height:35px;border-radius:999px;" alt="Google plus" Title="Google plus" class="theChampSharing theChampGoogleplusBackground" onclick='theChampPopup("https://plus.google.com/share?url=http%3A%2F%2Fcubacoopera.uccm.sld.cu%2Finicio-de-las-luchas-independentistas-en-cuba%2F")'><ss style="display:block;border-radius:999px;" class="theChampSharingSvg theChampGoogleplusSvg"></ss><ss class="the_champ_square_count the_champ_google_plus_count">&nbsp;</ss></i></li><li class="theChampSharingRound"><i style="width:35px;height:35px;border-radius:999px;" alt="Skype" Title="Skype" class="theChampSharing theChampSkypeBackground" onclick='theChampPopup("https://web.skype.com/share?url=http%3A%2F%2Fcubacoopera.uccm.sld.cu%2Finicio-de-las-luchas-independentistas-en-cuba%2F")'><ss style="display:block;border-radius:999px;" class="theChampSharingSvg theChampSkypeSvg"></ss><ss class="the_champ_square_count the_champ_Skype_count">&nbsp;</ss></i></li><li class="theChampSharingRound"><ss class="the_champ_square_count">&nbsp;</ss><i style="width:35px;height:35px;border-radius:999px;" title="More" alt="More" class="theChampSharing theChampMoreBackground" onclick="theChampMoreSharingPopup(this, 'http://cubacoopera.uccm.sld.cu/inicio-de-las-luchas-independentistas-en-cuba/', 'Inicio%20de%20las%20luchas%20independentistas%20en%20Cuba', '')" ><ss style="display:block" class="theChampSharingSvg theChampMoreSvg"></ss></i></li><li class="theChampSharingRound"><ss class="the_champ_square_count">&nbsp;</ss><i style="width:35px;height:35px;border-radius:999px;margin-left:9px !important;" title="Total Shares" alt="Total Shares" class="theChampSharing theChampTCBackground"></i></li></ul><div style="clear:both"></div></div><div style='clear: both'></div>					<a href="http://cubacoopera.uccm.sld.cu/inicio-de-las-luchas-independentistas-en-cuba/" class="more-link">Leer más...<span class="screen-reader-text"> Inicio de las luchas independentistas en Cuba</span></a><!-- wp-default -->
								</div> <!-- end .entry-content -->
					</article><!-- end .post -->
	<nav class="navigation pagination" role="navigation">
		<h2 class="screen-reader-text">Navegación de entradas</h2>
		<div class="nav-links"><span aria-current='page' class='page-numbers current'><span class="meta-nav screen-reader-text">Page </span>1</span>
<a class='page-numbers' href='http://cubacoopera.uccm.sld.cu/page/2/'><span class="meta-nav screen-reader-text">Page </span>2</a>
<span class="page-numbers dots">&hellip;</span>
<a class='page-numbers' href='http://cubacoopera.uccm.sld.cu/page/1803/'><span class="meta-nav screen-reader-text">Page </span>1.803</a>
<a class="next page-numbers" href="http://cubacoopera.uccm.sld.cu/page/2/"><i class="fa fa-angle-double-right"></i><span class="screen-reader-text">Next page</span></a></div>
	</nav>		</main><!-- end #main -->
	</div> <!-- #primary -->

<aside id="secondary" class="widget-area">
    <!-- Category Box Widget Two Layout-1 ============================================= --><aside id="magbook_category_box_two_column_widgets-4" class="widget widget-cat-box-2">
		
			<div class="box-two-layout-1">
				<div class="cat-box-two-wrap clearfix">
					<div class="widget widget-cat-box-left">
													<h2 class="widget-title">
																	<span>MEDIDAS EN CONTRA DE LA COVID-19 </span>
															</h2>
						<div class="cat-box-two-primary">							<article id="post-191434" class="post-191434 post type-post status-publish format-standard has-post-thumbnail hentry category-brigadashr category-coronavirus category-uccm tag-cubacoopera-cubaporlasalud-cuba">
																	<div class="cat-box-two-image">
										<figure class="post-featured-image">
											<a title="Premio Nobel de la Paz para las Brigadas Henry Reeve" href="http://cubacoopera.uccm.sld.cu/premio-nobel-de-la-paz-para-henry-reeve/"><img width="425" height="456" src="http://cubacoopera.uccm.sld.cu/wp-content/uploads/2020/10/Premio-Nobel-de-la-Paz-para-las-Brigadas-Henry-Reeve-1.jpg" class="attachment-magbook-featured-image size-magbook-featured-image wp-post-image" alt="" srcset="http://cubacoopera.uccm.sld.cu/wp-content/uploads/2020/10/Premio-Nobel-de-la-Paz-para-las-Brigadas-Henry-Reeve-1.jpg 425w, http://cubacoopera.uccm.sld.cu/wp-content/uploads/2020/10/Premio-Nobel-de-la-Paz-para-las-Brigadas-Henry-Reeve-1-233x250.jpg 233w, http://cubacoopera.uccm.sld.cu/wp-content/uploads/2020/10/Premio-Nobel-de-la-Paz-para-las-Brigadas-Henry-Reeve-1-411x441.jpg 411w" sizes="(max-width: 425px) 100vw, 425px" /></a>
										</figure> <!-- end .post-featured-image -->
									</div> <!-- end .cat-box-two-image -->
																<div class="cat-box-two-text">
									<header class="entry-header">
										<div class="entry-meta">	<span class="cats-links">
								<a class="cl-2028" href="http://cubacoopera.uccm.sld.cu/UCCM/brigadashr/">Brigadas Henry Reeve</a>
									<a class="cl-1802" href="http://cubacoopera.uccm.sld.cu/UCCM/coronavirus/">Coronavirus</a>
									<a class="cl-107" href="http://cubacoopera.uccm.sld.cu/UCCM/uccm/">UCCM</a>
				</span><!-- end .cat-links -->
</div> <!-- end .entry-meta -->											<h2 class="entry-title">
												<a title="Premio Nobel de la Paz para las Brigadas Henry Reeve" href="http://cubacoopera.uccm.sld.cu/premio-nobel-de-la-paz-para-henry-reeve/">Premio Nobel de la Paz para las Brigadas Henry Reeve</a>
											</h2> <!-- end.entry-title -->
											<div class="entry-meta"><span class="author vcard"><a href="http://cubacoopera.uccm.sld.cu/author/cubacooperauccm/" title="Premio Nobel de la Paz para las Brigadas Henry Reeve"><i class="fa fa-user-o"></i> Yanet Herrera Sierra</a></span><span class="posted-on"><a href="http://cubacoopera.uccm.sld.cu/premio-nobel-de-la-paz-para-henry-reeve/" title="13 octubre, 2020"><i class="fa fa-calendar-check-o"></i>13 octubre, 2020</a></span>														<span class="comments">
														<a href="http://cubacoopera.uccm.sld.cu/premio-nobel-de-la-paz-para-henry-reeve/#respond"><i class="fa fa-comments-o"></i> No Comments</a> </span>
												</div> <!-- end .entry-meta -->																				</header>
									<!-- end .entry-header -->
									<div class="entry-content">
										<p>Presenta oficialmente solicitud de Premio Nobel 2021 para las Brigadas Médicas Cubanas Henry Reeve el presidente del grupo parlamentario de amistad Francia-Cuba</p>
<div style='clear: both'></div><div  class='the_champ_sharing_container the_champ_horizontal_sharing' super-socializer-data-href="http://cubacoopera.uccm.sld.cu/premio-nobel-de-la-paz-para-henry-reeve/"><div class='the_champ_sharing_title' style="font-weight:bold" >Compartir</div><ul class="the_champ_sharing_ul"><li class="theChampSharingRound"><i style="width:35px;height:35px;border-radius:999px;" alt="Facebook" Title="Facebook" class="theChampSharing theChampFacebookBackground" onclick='theChampPopup("https://www.facebook.com/sharer/sharer.php?u=http%3A%2F%2Fcubacoopera.uccm.sld.cu%2Fpremio-nobel-de-la-paz-para-henry-reeve%2F")'><ss style="display:block;border-radius:999px;" class="theChampSharingSvg theChampFacebookSvg"></ss><ss class="the_champ_square_count the_champ_facebook_count">&nbsp;</ss></i></li><li class="theChampSharingRound"><i style="width:35px;height:35px;border-radius:999px;" alt="Twitter" Title="Twitter" class="theChampSharing theChampTwitterBackground" onclick='theChampPopup("http://twitter.com/intent/tweet?text=Premio%20Nobel%20de%20la%20Paz%20para%20las%20Brigadas%20Henry%20Reeve&url=http%3A%2F%2Fcubacoopera.uccm.sld.cu%2Fpremio-nobel-de-la-paz-para-henry-reeve%2F")'><ss style="display:block;border-radius:999px;" class="theChampSharingSvg theChampTwitterSvg"></ss><ss class="the_champ_square_count the_champ_twitter_count">&nbsp;</ss></i></li><li class="theChampSharingRound"><i style="width:35px;height:35px;border-radius:999px;" alt="Linkedin" Title="Linkedin" class="theChampSharing theChampLinkedinBackground" onclick='theChampPopup("http://www.linkedin.com/shareArticle?mini=true&url=http%3A%2F%2Fcubacoopera.uccm.sld.cu%2Fpremio-nobel-de-la-paz-para-henry-reeve%2F&title=Premio%20Nobel%20de%20la%20Paz%20para%20las%20Brigadas%20Henry%20Reeve")'><ss style="display:block;border-radius:999px;" class="theChampSharingSvg theChampLinkedinSvg"></ss><ss class="the_champ_square_count the_champ_linkedin_count">&nbsp;</ss></i></li><li class="theChampSharingRound"><i style="width:35px;height:35px;border-radius:999px;" alt="Whatsapp" Title="Whatsapp" class="theChampSharing theChampWhatsappBackground"><a href="https://web.whatsapp.com/send?text=Premio%20Nobel%20de%20la%20Paz%20para%20las%20Brigadas%20Henry%20Reeve http%3A%2F%2Fcubacoopera.uccm.sld.cu%2Fpremio-nobel-de-la-paz-para-henry-reeve%2F" rel="nofollow noopener" target="_blank"><ss style="display:block" class="theChampSharingSvg theChampWhatsappSvg"></ss><ss class="the_champ_square_count the_champ_whatsapp_count">&nbsp;</ss></a></i></li><li class="theChampSharingRound"><i style="width:35px;height:35px;border-radius:999px;" alt="Telegram" Title="Telegram" class="theChampSharing theChampTelegramBackground" onclick='theChampPopup("https://telegram.me/share/url?url=http%3A%2F%2Fcubacoopera.uccm.sld.cu%2Fpremio-nobel-de-la-paz-para-henry-reeve%2F&text=Premio%20Nobel%20de%20la%20Paz%20para%20las%20Brigadas%20Henry%20Reeve")'><ss style="display:block;border-radius:999px;" class="theChampSharingSvg theChampTelegramSvg"></ss><ss class="the_champ_square_count the_champ_Telegram_count">&nbsp;</ss></i></li><li class="theChampSharingRound"><i style="width:35px;height:35px;border-radius:999px;" alt="Google plus" Title="Google plus" class="theChampSharing theChampGoogleplusBackground" onclick='theChampPopup("https://plus.google.com/share?url=http%3A%2F%2Fcubacoopera.uccm.sld.cu%2Fpremio-nobel-de-la-paz-para-henry-reeve%2F")'><ss style="display:block;border-radius:999px;" class="theChampSharingSvg theChampGoogleplusSvg"></ss><ss class="the_champ_square_count the_champ_google_plus_count">&nbsp;</ss></i></li><li class="theChampSharingRound"><i style="width:35px;height:35px;border-radius:999px;" alt="Skype" Title="Skype" class="theChampSharing theChampSkypeBackground" onclick='theChampPopup("https://web.skype.com/share?url=http%3A%2F%2Fcubacoopera.uccm.sld.cu%2Fpremio-nobel-de-la-paz-para-henry-reeve%2F")'><ss style="display:block;border-radius:999px;" class="theChampSharingSvg theChampSkypeSvg"></ss><ss class="the_champ_square_count the_champ_Skype_count">&nbsp;</ss></i></li><li class="theChampSharingRound"><ss class="the_champ_square_count">&nbsp;</ss><i style="width:35px;height:35px;border-radius:999px;" title="More" alt="More" class="theChampSharing theChampMoreBackground" onclick="theChampMoreSharingPopup(this, 'http://cubacoopera.uccm.sld.cu/premio-nobel-de-la-paz-para-henry-reeve/', 'Premio%20Nobel%20de%20la%20Paz%20para%20las%20Brigadas%20Henry%20Reeve', '')" ><ss style="display:block" class="theChampSharingSvg theChampMoreSvg"></ss></i></li><li class="theChampSharingRound"><ss class="the_champ_square_count">&nbsp;</ss><i style="width:35px;height:35px;border-radius:999px;margin-left:9px !important;" title="Total Shares" alt="Total Shares" class="theChampSharing theChampTCBackground"></i></li></ul><div style="clear:both"></div></div><div style='clear: both'></div>									</div>
									<!-- end .entry-content -->
								</div> <!-- end .cat-box-text -->
							</article> <!-- end .post -->
							</div><div class="cat-box-two-secondary">							<article id="post-191237" class="post-191237 post type-post status-publish format-standard has-post-thumbnail hentry category-coronavirus category-haiti tag-cubacoopera-cubaporlasalud-cuba">
																	<div class="cat-box-two-image">
										<figure class="post-featured-image">
											<a title="Brigada Henry Reeve que enfrentó a la COVID-19 en Haití" href="http://cubacoopera.uccm.sld.cu/brigada-henry-reeve-2/"><img width="142" height="225" src="http://cubacoopera.uccm.sld.cu/wp-content/uploads/2020/10/nuevo-brigada.jpg" class="attachment-magbook-featured-image size-magbook-featured-image wp-post-image" alt="" /></a>
										</figure> <!-- end .post-featured-image -->
									</div> <!-- end .cat-box-two-image -->
																<div class="cat-box-two-text">
									<header class="entry-header">
										<div class="entry-meta">	<span class="cats-links">
								<a class="cl-1802" href="http://cubacoopera.uccm.sld.cu/UCCM/coronavirus/">Coronavirus</a>
									<a class="cl-1667" href="http://cubacoopera.uccm.sld.cu/UCCM/bmc/haiti/">Haití</a>
				</span><!-- end .cat-links -->
</div> <!-- end .entry-meta -->											<h2 class="entry-title">
												<a title="Brigada Henry Reeve que enfrentó a la COVID-19 en Haití" href="http://cubacoopera.uccm.sld.cu/brigada-henry-reeve-2/">Brigada Henry Reeve que enfrentó a la COVID-19 en Haití</a>
											</h2> <!-- end.entry-title -->
											<div class="entry-meta"><span class="author vcard"><a href="http://cubacoopera.uccm.sld.cu/author/cubacooperauccm/" title="Brigada Henry Reeve que enfrentó a la COVID-19 en Haití"><i class="fa fa-user-o"></i> Yanet Herrera Sierra</a></span><span class="posted-on"><a href="http://cubacoopera.uccm.sld.cu/brigada-henry-reeve-2/" title="9 octubre, 2020"><i class="fa fa-calendar-check-o"></i>9 octubre, 2020</a></span></div> <!-- end .entry-meta -->																				</header>
									<!-- end .entry-header -->
									<div class="entry-content">
										<p>La Brigada Henry Reeve en Haití, trabaja en el enfrentamiento a la COVID-19, ejemplo de ello es el Licenciado en Higiene y  Epidemiología Jorge Enrique Alonso Canino.</p>
<div style='clear: both'></div><div  class='the_champ_sharing_container the_champ_horizontal_sharing' super-socializer-data-href="http://cubacoopera.uccm.sld.cu/brigada-henry-reeve-2/"><div class='the_champ_sharing_title' style="font-weight:bold" >Compartir</div><ul class="the_champ_sharing_ul"><li class="theChampSharingRound"><i style="width:35px;height:35px;border-radius:999px;" alt="Facebook" Title="Facebook" class="theChampSharing theChampFacebookBackground" onclick='theChampPopup("https://www.facebook.com/sharer/sharer.php?u=http%3A%2F%2Fcubacoopera.uccm.sld.cu%2Fbrigada-henry-reeve-2%2F")'><ss style="display:block;border-radius:999px;" class="theChampSharingSvg theChampFacebookSvg"></ss><ss class="the_champ_square_count the_champ_facebook_count">&nbsp;</ss></i></li><li class="theChampSharingRound"><i style="width:35px;height:35px;border-radius:999px;" alt="Twitter" Title="Twitter" class="theChampSharing theChampTwitterBackground" onclick='theChampPopup("http://twitter.com/intent/tweet?text=Brigada%20Henry%20Reeve%20que%20enfrent%C3%B3%20a%20la%20COVID-19%20en%20Hait%C3%AD&url=http%3A%2F%2Fcubacoopera.uccm.sld.cu%2Fbrigada-henry-reeve-2%2F")'><ss style="display:block;border-radius:999px;" class="theChampSharingSvg theChampTwitterSvg"></ss><ss class="the_champ_square_count the_champ_twitter_count">&nbsp;</ss></i></li><li class="theChampSharingRound"><i style="width:35px;height:35px;border-radius:999px;" alt="Linkedin" Title="Linkedin" class="theChampSharing theChampLinkedinBackground" onclick='theChampPopup("http://www.linkedin.com/shareArticle?mini=true&url=http%3A%2F%2Fcubacoopera.uccm.sld.cu%2Fbrigada-henry-reeve-2%2F&title=Brigada%20Henry%20Reeve%20que%20enfrent%C3%B3%20a%20la%20COVID-19%20en%20Hait%C3%AD")'><ss style="display:block;border-radius:999px;" class="theChampSharingSvg theChampLinkedinSvg"></ss><ss class="the_champ_square_count the_champ_linkedin_count">&nbsp;</ss></i></li><li class="theChampSharingRound"><i style="width:35px;height:35px;border-radius:999px;" alt="Whatsapp" Title="Whatsapp" class="theChampSharing theChampWhatsappBackground"><a href="https://web.whatsapp.com/send?text=Brigada%20Henry%20Reeve%20que%20enfrent%C3%B3%20a%20la%20COVID-19%20en%20Hait%C3%AD http%3A%2F%2Fcubacoopera.uccm.sld.cu%2Fbrigada-henry-reeve-2%2F" rel="nofollow noopener" target="_blank"><ss style="display:block" class="theChampSharingSvg theChampWhatsappSvg"></ss><ss class="the_champ_square_count the_champ_whatsapp_count">&nbsp;</ss></a></i></li><li class="theChampSharingRound"><i style="width:35px;height:35px;border-radius:999px;" alt="Telegram" Title="Telegram" class="theChampSharing theChampTelegramBackground" onclick='theChampPopup("https://telegram.me/share/url?url=http%3A%2F%2Fcubacoopera.uccm.sld.cu%2Fbrigada-henry-reeve-2%2F&text=Brigada%20Henry%20Reeve%20que%20enfrent%C3%B3%20a%20la%20COVID-19%20en%20Hait%C3%AD")'><ss style="display:block;border-radius:999px;" class="theChampSharingSvg theChampTelegramSvg"></ss><ss class="the_champ_square_count the_champ_Telegram_count">&nbsp;</ss></i></li><li class="theChampSharingRound"><i style="width:35px;height:35px;border-radius:999px;" alt="Google plus" Title="Google plus" class="theChampSharing theChampGoogleplusBackground" onclick='theChampPopup("https://plus.google.com/share?url=http%3A%2F%2Fcubacoopera.uccm.sld.cu%2Fbrigada-henry-reeve-2%2F")'><ss style="display:block;border-radius:999px;" class="theChampSharingSvg theChampGoogleplusSvg"></ss><ss class="the_champ_square_count the_champ_google_plus_count">&nbsp;</ss></i></li><li class="theChampSharingRound"><i style="width:35px;height:35px;border-radius:999px;" alt="Skype" Title="Skype" class="theChampSharing theChampSkypeBackground" onclick='theChampPopup("https://web.skype.com/share?url=http%3A%2F%2Fcubacoopera.uccm.sld.cu%2Fbrigada-henry-reeve-2%2F")'><ss style="display:block;border-radius:999px;" class="theChampSharingSvg theChampSkypeSvg"></ss><ss class="the_champ_square_count the_champ_Skype_count">&nbsp;</ss></i></li><li class="theChampSharingRound"><ss class="the_champ_square_count">&nbsp;</ss><i style="width:35px;height:35px;border-radius:999px;" title="More" alt="More" class="theChampSharing theChampMoreBackground" onclick="theChampMoreSharingPopup(this, 'http://cubacoopera.uccm.sld.cu/brigada-henry-reeve-2/', 'Brigada%20Henry%20Reeve%20que%20enfrent%C3%B3%20a%20la%20COVID-19%20en%20Hait%C3%AD', '')" ><ss style="display:block" class="theChampSharingSvg theChampMoreSvg"></ss></i></li><li class="theChampSharingRound"><ss class="the_champ_square_count">&nbsp;</ss><i style="width:35px;height:35px;border-radius:999px;margin-left:9px !important;" title="Total Shares" alt="Total Shares" class="theChampSharing theChampTCBackground"></i></li></ul><div style="clear:both"></div></div><div style='clear: both'></div>									</div>
									<!-- end .entry-content -->
								</div> <!-- end .cat-box-text -->
							</article> <!-- end .post -->
							</div>					</div> <!-- end .widget-cat-box-left -->

					<div class="widget widget-cat-box-right">
											</div> <!-- end .widget-cat-box-right -->
				</div> <!-- end .cat-box-wrap -->
			</div><!-- end .box-two-layout-1 -->

	</aside><!-- end .widget-cat-box --><aside id="adwidget_htmlwidget-2" class="widget AdWidget_HTMLWidget"><div style='text-align: center;'><a class="twitter-timeline" data-width="320" data-height="520" data-dnt="true" href="https://twitter.com/cuba_coopera?ref_src=twsrc%5Etfw">Tweets by Cubacoopera</a> <script async src="https://platform.twitter.com/widgets.js" charset="utf-8"></script> </div></aside>		<aside id="recent-posts-3" class="widget widget_recent_entries">		<h2 class="widget-title">Al Dia</h2>		<ul>
											<li>
					<a href="http://cubacoopera.uccm.sld.cu/salva-vidas-donando-organos/">Salva vidas donando órganos</a>
									</li>
											<li>
					<a href="http://cubacoopera.uccm.sld.cu/logros-alcanzados-en-temas-de-salud/">Logros alcanzados en temas de salud</a>
									</li>
											<li>
					<a href="http://cubacoopera.uccm.sld.cu/verdaderos-protagonistas-de-la-ensenanza-en-medicina/">Verdaderos protagonistas de la enseñanza en Medicina</a>
									</li>
											<li>
					<a href="http://cubacoopera.uccm.sld.cu/nuestro-homenaje-para-todos-los-medicos-cubanos/">Nuestro homenaje para todos los médicos cubanos</a>
									</li>
											<li>
					<a href="http://cubacoopera.uccm.sld.cu/premio-nobel-de-la-paz-para-henry-reeve/">Premio Nobel de la Paz para las Brigadas Henry Reeve</a>
									</li>
					</ul>
		</aside><aside id="shortcodes-ultimate-4" class="widget shortcodes-ultimate"><h2 class="widget-title">Brigadas médicas Cubanas contra la Covid-19</h2><div class="textwidget"><div class="wonderpluginslider-container" id="wonderpluginslider-container-3" style="max-width:840px;margin:0 auto;padding-left:0px;padding-right:0px;padding-top:0px;padding-bottom:0px;"><div class="wonderpluginslider" id="wonderpluginslider-3" data-sliderid="3" data-width="840" data-height="360" data-skin="classic" data-autoplay="true" data-randomplay="false" data-loadimageondemand="false" data-transitiononfirstslide="false" data-autoplayvideo="false" data-isresponsive="true" data-fullwidth="false" data-isfullscreen="false" data-ratioresponsive="false" data-showtext="true" data-showtimer="true" data-showbottomshadow="true" data-navshowpreview="true" data-textautohide="true" data-pauseonmouseover="false" data-lightboxresponsive="true" data-lightboxshownavigation="false" data-lightboxshowtitle="true" data-lightboxshowdescription="false" data-texteffectresponsive="true" data-donotinit="false" data-addinitscript="false" data-lightboxfullscreenmode="false" data-lightboxcloseonoverlay="true" data-lightboxvideohidecontrols="false" data-lightboxnogroup="false" data-lightboxshowsocial="false" data-lightboxshowfacebook="true" data-lightboxshowtwitter="true" data-lightboxshowpinterest="true" data-lightboxsocialrotateeffect="true" data-showsocial="false" data-showfacebook="true" data-showtwitter="true" data-showpinterest="true" data-socialrotateeffect="true" data-triggerresize="false" data-lightboxautoslide="false" data-lightboxshowtimer="true" data-lightboxshowplaybutton="true" data-lightboxalwaysshownavarrows="false" data-lightboxshowtitleprefix="true" data-titleusealt="false" data-scalemode="fill" data-arrowstyle="mouseover" data-transition="slice" data-loop="0" data-border="0" data-slideinterval="8000" data-arrowimage="arrows-32-32-0.png" data-arrowwidth="32" data-arrowheight="32" data-arrowtop="50" data-arrowmargin="8" data-navstyle="bullets" data-navimage="bullet-24-24-0.png" data-navwidth="24" data-navheight="24" data-navspacing="8" data-navmarginx="16" data-navmarginy="16" data-navposition="bottom" data-playvideoimage="playvideo-64-64-0.png" data-playvideoimagewidth="64" data-playvideoimageheight="64" data-lightboxthumbwidth="90" data-lightboxthumbheight="60" data-lightboxthumbtopmargin="12" data-lightboxthumbbottommargin="4" data-lightboxbarheight="64" data-lightboxtitlebottomcss="color:#333; font-size:14px; font-family:Armata,sans-serif,Arial; overflow:hidden; text-align:left;" data-lightboxdescriptionbottomcss="color:#333; font-size:12px; font-family:Arial,Helvetica,sans-serif; overflow:hidden; text-align:left; margin:4px 0px 0px; padding: 0px;" data-textformat="Bottom bar" data-textpositionstatic="bottom" data-textpositiondynamic="bottomleft" data-paddingleft="0" data-paddingright="0" data-paddingtop="0" data-paddingbottom="0" data-texteffectresponsivesize="640" data-textleftrightpercentforstatic="40" data-fadeduration="1000" data-crossfadeduration="1000" data-slideduration="1000" data-elasticduration="1000" data-sliceduration="1500" data-blindsduration="1500" data-blocksduration="1500" data-shuffleduration="1500" data-tilesduration="2000" data-kenburnsduration="5000" data-flipduration="1500" data-flipwithzoomduration="2000" data-threedduration="1500" data-threedhorizontalduration="1500" data-threedwithzoomduration="2500" data-threedhorizontalwithzoomduration="2200" data-threedflipduration="1500" data-threedflipwithzoomduration="2000" data-threedtilesduration="2000" data-threedfallback="flip" data-threedhorizontalfallback="flip" data-threedwithzoomfallback="flipwithzoom" data-threedhorizontalwithzoomfallback="flipwithzoom" data-threedflipfallback="flip" data-threedflipwithzoomfallback="flipwithzoom" data-threedtilesfallback="tiles" data-ratiomediumscreen="800" data-ratiomediumheight="1.2" data-ratiosmallscreen="480" data-ratiosmallheight="1.6" data-socialmode="mouseover" data-socialposition="position:absolute;top:8px;right:8px;" data-socialpositionsmallscreen="position:absolute;top:8px;right:8px;" data-socialdirection="horizontal" data-socialbuttonsize="32" data-socialbuttonfontsize="18" data-lightboxsocialposition="position:absolute;top:100%;right:0;" data-lightboxsocialpositionsmallscreen="position:absolute;top:100%;right:0;left:0;" data-lightboxsocialdirection="horizontal" data-lightboxsocialbuttonsize="32" data-lightboxsocialbuttonfontsize="18" data-lightboxtitlestyle="bottom" data-lightboximagepercentage="75" data-lightboxdefaultvideovolume="1" data-lightboxoverlaybgcolor="#000" data-lightboxoverlayopacity="0.9" data-lightboxbgcolor="#fff" data-lightboxtitleprefix="%NUM / %TOTAL" data-lightboxtitleinsidecss="color:#fff; font-size:16px; font-family:Arial,Helvetica,sans-serif; overflow:hidden; text-align:left;" data-lightboxdescriptioninsidecss="color:#fff; font-size:12px; font-family:Arial,Helvetica,sans-serif; overflow:hidden; text-align:left; margin:4px 0px 0px; padding: 0px;" data-triggerresizedelay="100" data-lightboxslideinterval="5000" data-lightboxtimerposition="bottom" data-lightboxtimercolor="#dc572e" data-lightboxtimeropacity="1" data-lightboxbordersize="8" data-lightboxborderradius="0" data-textcss="display:block; padding:12px; text-align:left;" data-textbgcss="display:block; position:absolute; top:0px; left:0px; width:100%; height:100%; background-color:#333333; opacity:0.6; filter:alpha(opacity=60);" data-titlecss="display:block; position:relative; font:bold 14px &quot;Lucida Sans Unicode&quot;,&quot;Lucida Grande&quot;,sans-serif,Arial; color:#fff;" data-descriptioncss="display:block; position:relative; font:12px &quot;Lucida Sans Unicode&quot;,&quot;Lucida Grande&quot;,sans-serif,Arial; color:#fff; margin-top:8px;" data-buttoncss="display:block; position:relative; margin-top:8px;" data-titlecssresponsive="font-size:12px;" data-descriptioncssresponsive="display:none !important;" data-buttoncssresponsive="" data-jsfolder="http://cubacoopera.uccm.sld.cu/wp-content/plugins/wonderplugin-slider/engine/" style="display:none;" ><ul class="amazingslider-slides" style="display:none;"><li><a href="https://www.youtube.com/embed/dbFZ0bBO7ks" data-width="960" data-height="540" class="html5lightbox"><img class="amazingsliderimg" src="https://img.youtube.com/vi/dbFZ0bBO7ks/0.jpg" alt="Colaboradores cubanos en Trinidad y Tobago en lucha contra el Covid-19" title="Colaboradores cubanos en Trinidad y Tobago en lucha contra el Covid-19" data-description="" /></a></li><li><a href="https://www.youtube.com/embed/-_sm5PJgFhc?t=14s" data-width="960" data-height="540" class="html5lightbox"><img class="amazingsliderimg" src="https://img.youtube.com/vi/-_sm5PJgFhc/0.jpg" alt="Profesionales de la salud cubana en Qatar luchan contra el Covid-19" title="Profesionales de la salud cubana en Qatar luchan contra el Covid-19" data-description="" /></a></li><li><a href="https://www.youtube.com/embed/aCZb9WlJntU" data-width="960" data-height="540" class="html5lightbox"><img class="amazingsliderimg" src="https://img.youtube.com/vi/aCZb9WlJntU/0.jpg" alt="" title="" data-description="" /></a></li><li><a href="https://www.youtube.com/embed/cN-4tZ_4Z-o" data-width="960" data-height="540" class="html5lightbox"><img class="amazingsliderimg" src="https://img.youtube.com/vi/cN-4tZ_4Z-o/0.jpg" alt="" title="" data-description="" /></a></li><li><a href="https://www.youtube.com/embed/ZfEJLhmCQUY" data-width="960" data-height="540" class="html5lightbox"><img class="amazingsliderimg" src="https://img.youtube.com/vi/ZfEJLhmCQUY/0.jpg" alt="" title="" data-description="" /></a></li><li><a href="https://www.youtube.com/embed/jVBPzYLHWHo" data-width="960" data-height="540" data-description="Cuba siempre luchará por la salud mundial, para que un mundo mejor sea posible." class="html5lightbox"><img class="amazingsliderimg" src="https://img.youtube.com/vi/jVBPzYLHWHo/0.jpg" alt="Cuba siempre apoyará la salud mundial" title="Cuba siempre apoyará la salud mundial" data-description="Cuba siempre luchará por la salud mundial, para que un mundo mejor sea posible." /></a></li><li><a href="https://www.youtube.com/embed/R1qtZOIwFxg" data-width="960" data-height="540" data-description="Aplausos para héroes anónimos, que nos ayudan a luchar contra la Covid-19." class="html5lightbox"><img class="amazingsliderimg" src="https://img.youtube.com/vi/R1qtZOIwFxg/0.jpg" alt="Aplausos para héroes anónimos" title="Aplausos para héroes anónimos" data-description="Aplausos para héroes anónimos, que nos ayudan a luchar contra la Covid-19." /></a></li></ul><ul class="amazingslider-thumbnails" style="display:none;"><li><img class="amazingsliderthumbnailimg" src="https://img.youtube.com/vi/dbFZ0bBO7ks/0.jpg" alt="Colaboradores cubanos en Trinidad y Tobago en lucha contra el Covid-19" title="Colaboradores cubanos en Trinidad y Tobago en lucha contra el Covid-19" data-description="" /></li><li><img class="amazingsliderthumbnailimg" src="https://img.youtube.com/vi/-_sm5PJgFhc/0.jpg" alt="Profesionales de la salud cubana en Qatar luchan contra el Covid-19" title="Profesionales de la salud cubana en Qatar luchan contra el Covid-19" data-description="" /></li><li><img class="amazingsliderthumbnailimg" src="https://img.youtube.com/vi/aCZb9WlJntU/0.jpg" alt="" title="" data-description="" /></li><li><img class="amazingsliderthumbnailimg" src="https://img.youtube.com/vi/cN-4tZ_4Z-o/0.jpg" alt="" title="" data-description="" /></li><li><img class="amazingsliderthumbnailimg" src="https://img.youtube.com/vi/ZfEJLhmCQUY/0.jpg" alt="" title="" data-description="" /></li><li><img class="amazingsliderthumbnailimg" src="https://img.youtube.com/vi/jVBPzYLHWHo/0.jpg" alt="Cuba siempre apoyará la salud mundial" title="Cuba siempre apoyará la salud mundial" data-description="Cuba siempre luchará por la salud mundial, para que un mundo mejor sea posible." /></li><li><img class="amazingsliderthumbnailimg" src="https://img.youtube.com/vi/R1qtZOIwFxg/0.jpg" alt="Aplausos para héroes anónimos" title="Aplausos para héroes anónimos" data-description="Aplausos para héroes anónimos, que nos ayudan a luchar contra la Covid-19." /></li></ul><div class="wonderplugin-engine"><a href="http://www.wonderplugin.com/wordpress-slider/" title="WordPress Slider Plugin">WordPress Slider Plugin</a></div></div></div></div></aside><aside id="calendar-2" class="widget widget_calendar"><h2 class="widget-title">Calendario de post</h2><div id="calendar_wrap" class="calendar_wrap"><table id="wp-calendar">
	<caption>octubre 2020</caption>
	<thead>
	<tr>
		<th scope="col" title="lunes">L</th>
		<th scope="col" title="martes">M</th>
		<th scope="col" title="miércoles">X</th>
		<th scope="col" title="jueves">J</th>
		<th scope="col" title="viernes">V</th>
		<th scope="col" title="sábado">S</th>
		<th scope="col" title="domingo">D</th>
	</tr>
	</thead>

	<tfoot>
	<tr>
		<td colspan="3" id="prev"><a href="http://cubacoopera.uccm.sld.cu/2020/09/">&laquo; Sep</a></td>
		<td class="pad">&nbsp;</td>
		<td colspan="3" id="next" class="pad">&nbsp;</td>
	</tr>
	</tfoot>

	<tbody>
	<tr>
		<td colspan="3" class="pad">&nbsp;</td><td><a href="http://cubacoopera.uccm.sld.cu/2020/10/01/" aria-label="Entradas publicadas el 1 October, 2020">1</a></td><td>2</td><td>3</td><td>4</td>
	</tr>
	<tr>
		<td><a href="http://cubacoopera.uccm.sld.cu/2020/10/05/" aria-label="Entradas publicadas el 5 October, 2020">5</a></td><td><a href="http://cubacoopera.uccm.sld.cu/2020/10/06/" aria-label="Entradas publicadas el 6 October, 2020">6</a></td><td><a href="http://cubacoopera.uccm.sld.cu/2020/10/07/" aria-label="Entradas publicadas el 7 October, 2020">7</a></td><td><a href="http://cubacoopera.uccm.sld.cu/2020/10/08/" aria-label="Entradas publicadas el 8 October, 2020">8</a></td><td><a href="http://cubacoopera.uccm.sld.cu/2020/10/09/" aria-label="Entradas publicadas el 9 October, 2020">9</a></td><td><a href="http://cubacoopera.uccm.sld.cu/2020/10/10/" aria-label="Entradas publicadas el 10 October, 2020">10</a></td><td>11</td>
	</tr>
	<tr>
		<td><a href="http://cubacoopera.uccm.sld.cu/2020/10/12/" aria-label="Entradas publicadas el 12 October, 2020">12</a></td><td><a href="http://cubacoopera.uccm.sld.cu/2020/10/13/" aria-label="Entradas publicadas el 13 October, 2020">13</a></td><td id="today"><a href="http://cubacoopera.uccm.sld.cu/2020/10/14/" aria-label="Entradas publicadas el 14 October, 2020">14</a></td><td>15</td><td>16</td><td>17</td><td>18</td>
	</tr>
	<tr>
		<td>19</td><td>20</td><td>21</td><td>22</td><td>23</td><td>24</td><td>25</td>
	</tr>
	<tr>
		<td>26</td><td>27</td><td>28</td><td>29</td><td>30</td><td>31</td>
		<td class="pad" colspan="1">&nbsp;</td>
	</tr>
	</tbody>
	</table></div></aside><aside id="ai1ec_agenda_widget-4" class="widget widget_ai1ec_agenda_widget">

	<h2 class="widget-title">Eventos</h2>

<style>
<!--

-->
</style>
<div class="timely ai1ec-agenda-widget-view ai1ec-clearfix">

			<p class="ai1ec-no-results">
			No hay nuevos eventos.
		</p>
	 
			<div class="ai1ec-subscribe-buttons-widget">
							<a class="ai1ec-btn ai1ec-btn-default ai1ec-btn-xs ai1ec-pull-right
					ai1ec-calendar-link"
					href="&#x3F;ai1ec&#x3D;">
					Ver Calendario
					<i class="ai1ec-fa ai1ec-fa-arrow-right"></i>
				</a>
			
							<div class="ai1ec-subscribe-dropdown ai1ec-dropdown ai1ec-btn
	ai1ec-btn-default ai1ec-btn-xs">
	<span role="button" class="ai1ec-dropdown-toggle ai1ec-subscribe"
			data-toggle="ai1ec-dropdown">
		<i class="ai1ec-fa ai1ec-icon-rss ai1ec-fa-lg ai1ec-fa-fw"></i>
		<span class="ai1ec-hidden-xs">
							Añadir
						<span class="ai1ec-caret"></span>
		</span>
	</span>
			<ul class="ai1ec-dropdown-menu ai1ec-pull-left" role="menu">
		<li>
			<a class="ai1ec-tooltip-trigger ai1ec-tooltip-auto" target="_blank"
				data-placement="right" title="Copia esta URL de tu propio calendario Timely o haz click para agregar a tu calendario de texto-enriquecido"
				href="http&#x3A;&#x2F;&#x2F;cubacoopera.uccm.sld.cu&#x2F;&#x3F;plugin&#x3D;all-in-one-event-calendar&amp;controller&#x3D;ai1ec_exporter_controller&amp;action&#x3D;export_events">
				<i class="ai1ec-fa ai1ec-fa-lg ai1ec-fa-fw ai1ec-icon-timely"></i>
				Añadir a Timely Calendar
			</a>
		</li>
		<li>
			<a class="ai1ec-tooltip-trigger ai1ec-tooltip-auto" target="_blank"
			  data-placement="right" title="Suscribete a este calendario en tu Google Calendar"
			  href="https://www.google.com/calendar/render?cid=http&#x25;3A&#x25;2F&#x25;2Fcubacoopera.uccm.sld.cu&#x25;2F&#x25;3Fplugin&#x25;3Dall-in-one-event-calendar&#x25;26controller&#x25;3Dai1ec_exporter_controller&#x25;26action&#x25;3Dexport_events&#x25;26no_html&#x25;3Dtrue&#x25;26&#x25;26">
				<i class="ai1ec-fa ai1ec-icon-google ai1ec-fa-lg ai1ec-fa-fw"></i>
				Añadir a Google
			</a>
		</li>
		<li>
			<a class="ai1ec-tooltip-trigger ai1ec-tooltip-auto" target="_blank"
			  data-placement="right" title="Suscríbete a este calendario en tu MS Outlook"
			  href="webcal&#x3A;&#x2F;&#x2F;cubacoopera.uccm.sld.cu&#x2F;&#x3F;plugin&#x3D;all-in-one-event-calendar&amp;controller&#x3D;ai1ec_exporter_controller&amp;action&#x3D;export_events&amp;no_html&#x3D;true">
				<i class="ai1ec-fa ai1ec-icon-windows ai1ec-fa-lg ai1ec-fa-fw"></i>
				Agregar a Outlook
			</a>
		</li>
		<li>
			<a class="ai1ec-tooltip-trigger ai1ec-tooltip-auto" target="_blank"
			  data-placement="right" title="Suscríbete a este calendario en Apple Calendar / iCal"
			  href="webcal&#x3A;&#x2F;&#x2F;cubacoopera.uccm.sld.cu&#x2F;&#x3F;plugin&#x3D;all-in-one-event-calendar&amp;controller&#x3D;ai1ec_exporter_controller&amp;action&#x3D;export_events&amp;no_html&#x3D;true">
				<i class="ai1ec-fa ai1ec-icon-apple ai1ec-fa-lg ai1ec-fa-fw"></i>
				Agregar a Apple Calendar
			</a>
		</li>
		<li>
						<a class="ai1ec-tooltip-trigger ai1ec-tooltip-auto"
			  data-placement="right" title="Suscribirse a este calendario en la agenda de otro de texto plano"
			  href="http&#x3A;&#x2F;&#x2F;cubacoopera.uccm.sld.cu&#x2F;&#x3F;plugin&#x3D;all-in-one-event-calendar&amp;controller&#x3D;ai1ec_exporter_controller&amp;action&#x3D;export_events&amp;no_html&#x3D;true">
				<i class="ai1ec-fa ai1ec-icon-calendar ai1ec-fa-fw"></i>
				Agregar a otro calendario
			</a>
		</li>
		<li>
			<a class="ai1ec-tooltip-trigger ai1ec-tooltip-auto"
			  data-placement="right" title=""
			  href="http&#x3A;&#x2F;&#x2F;cubacoopera.uccm.sld.cu&#x2F;&#x3F;plugin&#x3D;all-in-one-event-calendar&amp;controller&#x3D;ai1ec_exporter_controller&amp;action&#x3D;export_events&xml=true">
				<i class="ai1ec-fa ai1ec-fa-file-text ai1ec-fa-lg ai1ec-fa-fw"></i>
				Export to XML
			</a>
		</li>
	</ul>
</div>

					</div>
	 
</div>



</aside><aside id="recent-comments-4" class="widget widget_recent_comments"><h2 class="widget-title">Comentarios recientes</h2><ul id="recentcomments"><li class="recentcomments"><span class="comment-author-link"><a href='https://laptopalanyerler.blogspot.com/' rel='external nofollow' class='url'>Laptop Alan Yerler</a></span> en <a href="http://cubacoopera.uccm.sld.cu/evento/centro-oftalmologico-de-la-mision-milagro-en-siguatepeque-bmc-honduras/#comment-828202">Centro Oftalmológico de la Misión Milagro en Siguatepeque. BMC Honduras.</a></li><li class="recentcomments"><span class="comment-author-link"><a href='https://masaloku.com.tr/' rel='external nofollow' class='url'>Masal</a></span> en <a href="http://cubacoopera.uccm.sld.cu/de-fidel-sobre-camilo/#comment-828201">De Fidel Castro sobre Camilo Cienfuegos</a></li><li class="recentcomments"><span class="comment-author-link"><a href='https://masaloku.com.tr/' rel='external nofollow' class='url'>Masal</a></span> en <a href="http://cubacoopera.uccm.sld.cu/de-vilma-espin-guillois-sobre-camilo-3/#comment-828200">De Vilma Espín Guillois sobre Camilo Cienfuegos</a></li><li class="recentcomments"><span class="comment-author-link"><a href='https://svrtechnologies.com/sap-mm-online-training/' rel='external nofollow' class='url'>SVR Technologies</a></span> en <a href="http://cubacoopera.uccm.sld.cu/evento/centro-oftalmologico-de-la-mision-milagro-en-siguatepeque-bmc-honduras/#comment-828199">Centro Oftalmológico de la Misión Milagro en Siguatepeque. BMC Honduras.</a></li><li class="recentcomments"><span class="comment-author-link"><a href='https://www.baytronik.com/gaziantep-google-seo/' rel='external nofollow' class='url'>Gaziantep Google</a></span> en <a href="http://cubacoopera.uccm.sld.cu/bmc-en-burkina-faso-taller-de-actualizacion-en-ira-por-coronavirus/#comment-828198">Taller de actualización en IRA por coronavirus</a></li></ul></aside></aside><!-- end #secondary -->
</div><!-- end .wrap -->
</div><!-- end #content -->
<!-- Footer Start ============================================= -->
<footer id="colophon" class="site-footer">
	<div class="widget-wrap" >
		<div class="wrap">
			<div class="widget-area">
			<div class="column-4"><aside id="login_wid-2" class="widget widget_login_wid"><h3 class="widget-title">Iniciar sesión</h3>		<script>
			function closeMessage(){jQuery('.error_wid_login').hide();}
			jQuery(document).ready(function () {
				jQuery('#login').validate({ errorClass: "lw-error" });
			});
		</script>
	<div id="log_forms" class="log_forms login_wid-2">
        

		
<form name="login" id="login" method="post" action="" autocomplete="off" >


<input type="hidden" name="option" value="afo_user_login" />
<input type="hidden" name="redirect" value="http://cubacoopera.uccm.sld.cu/" />
<div class="log-form-group">
	<label for="userusername">Nombre de usuario </label>
	<input type="text" name="userusername" id="userusername" required />
</div>
<div class="log-form-group">
	<label for="userpassword">Contraseña </label>
	<input type="password" name="userpassword" id="userpassword" required />
</div>



<div class="log-form-group"><label for="remember"> Remember Me</label> <input type="checkbox" name="remember" id="remember" value="Yes"        /></div>
<div class="log-form-group"><input name="login" type="submit" value="Iniciar sesión" /></div>
<div class="log-form-group extra-links">
	</div>
</form>


</div></aside></div><!-- end .column4  --><div class="column-4"><aside id="wp_statistics_widget-4" class="widget widget_wp_statistics_widget"><h3 class="widget-title">Estadísticas</h3><ul><li><label>Usuarios online: </label>5</li><li><label>Visitas hoy: </label>3.517</li><li><label>Visitas los últimos 7 días: </label>41.098</li><li><label>Visitas los últimos 30 días: </label>262.525</li><li><label>Visitas los últimos 365 días: </label>3.165.300</li><li><label>Promedio de posts: </label>17</li><li><label>Fecha de última entrada: </label>14 octubre, 2020</li></ul></aside></div><!--end .column4  --><div class="column-4"><aside id="newsletterwidget-2" class="widget widget_newsletterwidget"><div class="tnp tnp-widget"><form method="post" action="http://cubacoopera.uccm.sld.cu/?na=s" onsubmit="return newsletter_check(this)">

<input type="hidden" name="nlang" value="">
<input type="hidden" name="nr" value="widget">
<input type='hidden' name='nl[]' value='0'>
<div class="tnp-field tnp-field-firstname"><label>Primer nombre o nombre completo</label><input class="tnp-firstname" type="text" name="nn" ></div>
<div class="tnp-field tnp-field-email"><label>Email</label><input class="tnp-email" type="email" name="ne" required></div>
<div class="tnp-field tnp-field-privacy"><label><input type="checkbox" name="ny" required class="tnp-privacy">&nbsp;para continuar, debe eceptar los terminos de privacidad</label></div>
<div class="tnp-field tnp-field-button"><input class="tnp-submit" type="submit" value="Subscríbete" >
</div>
</form>
</div>
</aside></div><!--end .column4  --><div class="column-4"><aside id="categories-6" class="widget widget_categories"><h3 class="widget-title">Búsqueda por temas</h3><form action="http://cubacoopera.uccm.sld.cu" method="get"><label class="screen-reader-text" for="cat">Búsqueda por temas</label><select  name='cat' id='cat' class='postform' >
	<option value='-1'>Elegir categoría</option>
	<option class="level-0" value="2028">Brigadas Henry Reeve&nbsp;&nbsp;(71)</option>
	<option class="level-0" value="1620">Brigadas Médicas Cubanas&nbsp;&nbsp;(1.843)</option>
	<option class="level-1" value="1621">&nbsp;&nbsp;&nbsp;África del sur&nbsp;&nbsp;(12)</option>
	<option class="level-1" value="1622">&nbsp;&nbsp;&nbsp;Angola&nbsp;&nbsp;(11)</option>
	<option class="level-1" value="1662">&nbsp;&nbsp;&nbsp;Antigua y Barbuda&nbsp;&nbsp;(1)</option>
	<option class="level-1" value="1674">&nbsp;&nbsp;&nbsp;Arabia Saudita&nbsp;&nbsp;(4)</option>
	<option class="level-1" value="1651">&nbsp;&nbsp;&nbsp;Argelia&nbsp;&nbsp;(234)</option>
	<option class="level-1" value="1663">&nbsp;&nbsp;&nbsp;Bahamas&nbsp;&nbsp;(1)</option>
	<option class="level-1" value="1675">&nbsp;&nbsp;&nbsp;Bahreín&nbsp;&nbsp;(1)</option>
	<option class="level-1" value="1652">&nbsp;&nbsp;&nbsp;Belice&nbsp;&nbsp;(35)</option>
	<option class="level-1" value="1623">&nbsp;&nbsp;&nbsp;Botswana&nbsp;&nbsp;(5)</option>
	<option class="level-1" value="1624">&nbsp;&nbsp;&nbsp;Burkina Faso&nbsp;&nbsp;(44)</option>
	<option class="level-1" value="1625">&nbsp;&nbsp;&nbsp;Cabo Verde&nbsp;&nbsp;(13)</option>
	<option class="level-1" value="1626">&nbsp;&nbsp;&nbsp;Chad&nbsp;&nbsp;(1)</option>
	<option class="level-1" value="1653">&nbsp;&nbsp;&nbsp;Chile&nbsp;&nbsp;(1)</option>
	<option class="level-1" value="1678">&nbsp;&nbsp;&nbsp;China&nbsp;&nbsp;(6)</option>
	<option class="level-1" value="1627">&nbsp;&nbsp;&nbsp;Congo&nbsp;&nbsp;(7)</option>
	<option class="level-1" value="1664">&nbsp;&nbsp;&nbsp;Curazao&nbsp;&nbsp;(1)</option>
	<option class="level-1" value="1628">&nbsp;&nbsp;&nbsp;Djibuti&nbsp;&nbsp;(45)</option>
	<option class="level-1" value="1665">&nbsp;&nbsp;&nbsp;Dominica&nbsp;&nbsp;(5)</option>
	<option class="level-1" value="1629">&nbsp;&nbsp;&nbsp;Eritrea&nbsp;&nbsp;(1)</option>
	<option class="level-1" value="1647">&nbsp;&nbsp;&nbsp;Eswatini&nbsp;&nbsp;(3)</option>
	<option class="level-1" value="1630">&nbsp;&nbsp;&nbsp;Etiopía&nbsp;&nbsp;(8)</option>
	<option class="level-1" value="1631">&nbsp;&nbsp;&nbsp;Gabón&nbsp;&nbsp;(5)</option>
	<option class="level-1" value="1632">&nbsp;&nbsp;&nbsp;Gambia&nbsp;&nbsp;(59)</option>
	<option class="level-1" value="1633">&nbsp;&nbsp;&nbsp;Ghana&nbsp;&nbsp;(5)</option>
	<option class="level-1" value="1666">&nbsp;&nbsp;&nbsp;Granada&nbsp;&nbsp;(1)</option>
	<option class="level-1" value="1655">&nbsp;&nbsp;&nbsp;Guatemala&nbsp;&nbsp;(64)</option>
	<option class="level-1" value="1634">&nbsp;&nbsp;&nbsp;Guinea Bissau&nbsp;&nbsp;(100)</option>
	<option class="level-1" value="1635">&nbsp;&nbsp;&nbsp;Guinea Conakry&nbsp;&nbsp;(55)</option>
	<option class="level-1" value="1636">&nbsp;&nbsp;&nbsp;Guinea Ecuatorial&nbsp;&nbsp;(2)</option>
	<option class="level-1" value="1656">&nbsp;&nbsp;&nbsp;Guyana&nbsp;&nbsp;(1)</option>
	<option class="level-1" value="1667">&nbsp;&nbsp;&nbsp;Haití&nbsp;&nbsp;(195)</option>
	<option class="level-1" value="1668">&nbsp;&nbsp;&nbsp;Jamaica&nbsp;&nbsp;(10)</option>
	<option class="level-1" value="1637">&nbsp;&nbsp;&nbsp;Kenya&nbsp;&nbsp;(5)</option>
	<option class="level-1" value="1684">&nbsp;&nbsp;&nbsp;Kiribati&nbsp;&nbsp;(1)</option>
	<option class="level-1" value="1677">&nbsp;&nbsp;&nbsp;Kuwait&nbsp;&nbsp;(1)</option>
	<option class="level-1" value="1638">&nbsp;&nbsp;&nbsp;Lesoto&nbsp;&nbsp;(2)</option>
	<option class="level-1" value="1639">&nbsp;&nbsp;&nbsp;Liberia&nbsp;&nbsp;(1)</option>
	<option class="level-1" value="1640">&nbsp;&nbsp;&nbsp;Mauritania&nbsp;&nbsp;(5)</option>
	<option class="level-1" value="1641">&nbsp;&nbsp;&nbsp;Mozambique&nbsp;&nbsp;(66)</option>
	<option class="level-1" value="1642">&nbsp;&nbsp;&nbsp;Namibia&nbsp;&nbsp;(70)</option>
	<option class="level-1" value="1643">&nbsp;&nbsp;&nbsp;Níger&nbsp;&nbsp;(3)</option>
	<option class="level-1" value="1680">&nbsp;&nbsp;&nbsp;Qatar&nbsp;&nbsp;(390)</option>
	<option class="level-1" value="1644">&nbsp;&nbsp;&nbsp;RASD&nbsp;&nbsp;(2)</option>
	<option class="level-1" value="1658">&nbsp;&nbsp;&nbsp;República Dominicana&nbsp;&nbsp;(1)</option>
	<option class="level-1" value="1669">&nbsp;&nbsp;&nbsp;San Cristóbal y Nieves&nbsp;&nbsp;(1)</option>
	<option class="level-1" value="1670">&nbsp;&nbsp;&nbsp;San Vicente y las Granadinas&nbsp;&nbsp;(24)</option>
	<option class="level-1" value="1646">&nbsp;&nbsp;&nbsp;Seychelles&nbsp;&nbsp;(1)</option>
	<option class="level-1" value="1659">&nbsp;&nbsp;&nbsp;Surinam&nbsp;&nbsp;(4)</option>
	<option class="level-1" value="1648">&nbsp;&nbsp;&nbsp;Tanzania&nbsp;&nbsp;(3)</option>
	<option class="level-1" value="1681">&nbsp;&nbsp;&nbsp;Timor Leste&nbsp;&nbsp;(61)</option>
	<option class="level-1" value="1672">&nbsp;&nbsp;&nbsp;Trinidad y Tobago&nbsp;&nbsp;(51)</option>
	<option class="level-1" value="1682">&nbsp;&nbsp;&nbsp;Turquia&nbsp;&nbsp;(1)</option>
	<option class="level-1" value="1649">&nbsp;&nbsp;&nbsp;Uganda&nbsp;&nbsp;(1)</option>
	<option class="level-1" value="1660">&nbsp;&nbsp;&nbsp;Uruguay&nbsp;&nbsp;(1)</option>
	<option class="level-1" value="1661">&nbsp;&nbsp;&nbsp;Venezuela&nbsp;&nbsp;(216)</option>
	<option class="level-1" value="1683">&nbsp;&nbsp;&nbsp;Vietnam&nbsp;&nbsp;(1)</option>
	<option class="level-1" value="1650">&nbsp;&nbsp;&nbsp;Zimbabwe&nbsp;&nbsp;(18)</option>
	<option class="level-0" value="1731">Contenidos&nbsp;&nbsp;(109)</option>
	<option class="level-1" value="1734">&nbsp;&nbsp;&nbsp;Historia de la Colaboración Médica&nbsp;&nbsp;(26)</option>
	<option class="level-1" value="1732">&nbsp;&nbsp;&nbsp;Historias de vida&nbsp;&nbsp;(47)</option>
	<option class="level-1" value="1733">&nbsp;&nbsp;&nbsp;Principales resultados&nbsp;&nbsp;(49)</option>
	<option class="level-0" value="1802">Coronavirus&nbsp;&nbsp;(296)</option>
	<option class="level-0" value="1824">Pensamientos internacionalista&nbsp;&nbsp;(33)</option>
	<option class="level-0" value="107">UCCM&nbsp;&nbsp;(25.395)</option>
</select>
</form>
<script type='text/javascript'>
/* <![CDATA[ */
(function() {
	var dropdown = document.getElementById( "cat" );
	function onCatChange() {
		if ( dropdown.options[ dropdown.selectedIndex ].value > 0 ) {
			dropdown.parentNode.submit();
		}
	}
	dropdown.onchange = onCatChange;
})();
/* ]]> */
</script>

			</aside></div><!--end .column4-->			</div> <!-- end .widget-area -->
		</div><!-- end .wrap -->
	</div> <!-- end .widget-wrap -->
		<div class="site-info">
		<div class="wrap">
						<div class="copyright-wrap clearfix">
				<div class="copyright">					<a title="Cubacoopera" target="_blank" href="http://cubacoopera.uccm.sld.cu/">Cubacoopera</a> : 
									Este portal Web pretende aumentar los niveles de visibilidad en el ciberespacio de los servicios profesionales médicos, académicos y de salud que se prestan a partir de la cooperación médica internacional, constituir un medio para el intercambio directo y de retroalimentación con los colaboradores cubanos de la salud y de estos con sus familiares, contribuir a la socialización de los contenidos publicados, generar tráfico de usuarios y servir como herramienta para medir el alcance e impacto de la divulgación de los resultados de la colaboración médica cubana en el exterior. <a title="" target="_blank" href="https://themefreesia.com"> </a>
									 <a title="WordPress" target="_blank" href="https://wordpress.org"></a> <br> <strong>Autor: Ing. Jorge Luis Peña Millán 								</div>
							</div> <!-- end .copyright-wrap -->
			<div style="clear:both;"></div>
		</div> <!-- end .wrap -->
	</div> <!-- end .site-info -->
				<a class="go-to-top">
				<span class="icon-bg"></span>
				<span class="back-to-top-text">Top</span>
				<i class="fa fa-angle-up back-to-top-icon"></i>
			</a>
		<div class="page-overlay"></div>
</footer> <!-- end #colophon -->
</div><!-- end .site-content-contain -->
</div><!-- end #page -->
<div id="wondergridgallerylightbox_options" data-skinsfoldername="skins/default/"  data-jsfolder="http://cubacoopera.uccm.sld.cu/wp-content/plugins/wonderplugin-gridgallery/engine/" style="display:none;"></div><!-- WP-TopBar_5.34 :: DB: 5.09 :: Number of TopBars Selected: 0 :: Rotate TopBars: no -->
	<div id="fb-root"></div>
	<script type='text/javascript'>
/* <![CDATA[ */
var wpcf7 = {"apiSettings":{"root":"http:\/\/cubacoopera.uccm.sld.cu\/wp-json\/contact-form-7\/v1","namespace":"contact-form-7\/v1"},"cached":"1"};
/* ]]> */
</script>
<script type='text/javascript' src='http://cubacoopera.uccm.sld.cu/wp-content/plugins/contact-form-7/includes/js/scripts.js?ver=5.1.1'></script>
<script type='text/javascript' src='http://cubacoopera.uccm.sld.cu/wp-includes/js/jquery/jquery.form.min.js?ver=4.2.1'></script>
<script type='text/javascript' src='http://cubacoopera.uccm.sld.cu/wp-content/plugins/sb-popular-posts-tabbed-widget/includes/js/tabs.js?ver=1'></script>
<script type='text/javascript' src='http://cubacoopera.uccm.sld.cu/wp-content/themes/magbook/js/magbook-main.js?ver=5.2.4'></script>
<script type='text/javascript' src='http://cubacoopera.uccm.sld.cu/wp-content/themes/magbook/assets/sticky/jquery.sticky.min.js?ver=5.2.4'></script>
<script type='text/javascript' src='http://cubacoopera.uccm.sld.cu/wp-content/themes/magbook/assets/sticky/sticky-settings.js?ver=5.2.4'></script>
<script type='text/javascript' src='http://cubacoopera.uccm.sld.cu/wp-content/themes/magbook/js/navigation.js?ver=5.2.4'></script>
<script type='text/javascript' src='http://cubacoopera.uccm.sld.cu/wp-content/themes/magbook/js/jquery.flexslider-min.js?ver=5.2.4'></script>
<script type='text/javascript'>
/* <![CDATA[ */
var magbook_slider_value = {"magbook_animation_effect":"fade","magbook_slideshowSpeed":"5000","magbook_animationSpeed":"700"};
/* ]]> */
</script>
<script type='text/javascript' src='http://cubacoopera.uccm.sld.cu/wp-content/themes/magbook/js/flexslider-setting.js?ver=5.2.4'></script>
<script type='text/javascript' src='http://cubacoopera.uccm.sld.cu/wp-content/plugins/super-socializer/js/front/combined.js?ver=7.12.11'></script>
<script type='text/javascript'>
/* <![CDATA[ */
var newsletter = {"messages":{"email_error":"El Email escrito no es correcto","name_error":"Por favor, escriba su primer nombre o nombre completo","surname_error":"Por favor, escriba sus apellidos","profile_error":"A mandatory field is not filled in","privacy_error":"Usted debe aceptar los terminos de privacidad"},"profile_max":"20"};
/* ]]> */
</script>
<script type='text/javascript' src='http://cubacoopera.uccm.sld.cu/wp-content/plugins/newsletter/subscription/validate.js?ver=6.3.2'></script>
<script type='text/javascript' src='http://cubacoopera.uccm.sld.cu/wp-includes/js/wp-embed.min.js?ver=5.2.4'></script>
<script type='text/javascript' src='http://cubacoopera.uccm.sld.cu/?ai1ec_render_js=common_frontend&#038;is_backend=false&#038;ver=2.5.36'></script>
</body>
</html>