#include "hip/hip_runtime.h"


<!DOCTYPE html>
<html lang="es">
<head>
    <meta charset='utf-8'>
    <meta name='viewport' content='width=device-width'>
    <meta http-equiv="X-UA-Compatible" content="IE=edge,chrome=1">
    <title>No confiarse ante la COVID-19, clave para el éxito | Presidencia y Gobierno de la República de Cuba</title>
    <link rel="alternate" type="application/rss+xml" title="RSS 2.0" href="https://www.presidencia.gob.cu/es/feed/">
    <link rel="alternate" type="text/xml" title="RSS .92" href="https://www.presidencia.gob.cu/es/feed/">
    <link rel="icon" href="/static/assets/img/favicon/favicon.ico">
    <link rel="shortcut icon" href="/static/assets/img/favicon/favicon.ico">
    <meta name="theme-color" content="#062f4f">
    
    <link rel="canonical" href="https://www.presidencia.gob.cu/es/noticias/no-confiarse-ante-la-covid-19-clave-para-el-exito/"/>
    <meta name="keywords" content="Noticias,COVID-19,Salud,coronavirus">
    <meta name="description" content="Una tendencia a la disminución de los pacientes positivos a la COVID-19 que se encuentran hospitalizados marcó el inicio de la jornada de este miércoles, en la cual  el grupo temporal de trabajo para la prevención y control del nuevo coronavirus, que encabezan el Presidente de la República y el Primer Ministro, analizó la situación epidemiológica de las provincias de Sancti Spíritus, La Habana y Ciego de Ávila.">
    <!-- OpenGraph -->
    <meta property="og:site_name" content="Presidencia de Cuba">
    <meta property="og:title" content="No confiarse ante la COVID-19, clave para el éxito">
    <meta property="og:url" content="https://www.presidencia.gob.cu/es/noticias/no-confiarse-ante-la-covid-19-clave-para-el-exito/">
    <meta property="og:description" content="Una tendencia a la disminución de los pacientes positivos a la COVID-19 que se encuentran hospitalizados marcó el inicio de la jornada de este miércoles, en la cual  el grupo temporal de trabajo para la prevención y control del nuevo coronavirus, que encabezan el Presidente de la República y el Primer Ministro, analizó la situación epidemiológica de las provincias de Sancti Spíritus, La Habana y Ciego de Ávila.">
    <meta property="og:image" content="https://www.presidencia.gob.cu/media/filer/public/2020/10/07/2020-10-07_reunion_coronavirus_7-10_er_1.jpg">
    <meta property="og:type" content="Article">
    <meta property="og:locale" content="es">
    <meta property="article:author" content="Yaima Puig Meneses">
    <meta property="article:published_time" content="2020-10-07T20:54:16">
    <meta property="article:modified_time" content="2020-10-07T20:54:16">
    <meta property="article:section" content="Noticias">
    <meta property="article:tag" content="Salud"><meta property="article:tag" content="COVID-19"><meta property="article:tag" content="Reunión de Trabajo">
    <!-- TwitterCards -->
    <meta name="twitter:site" content="@PresidenciaCuba">
    <meta name="twitter:domain" content="www.presidencia.gob.cu">
    <meta name="twitter:card" content="summary_large_image">
    <meta name="twitter:creator" content="@PresidenciaCuba">
    <meta name="twitter:url" content="https://www.presidencia.gob.cu/es/noticias/no-confiarse-ante-la-covid-19-clave-para-el-exito/">
    <meta name="twitter:title" content="No confiarse ante la COVID-19, clave para el éxito">
    <meta name="twitter:description" content="Una tendencia a la disminución de los pacientes positivos a la COVID-19 que se encuentran hospitalizados marcó el inicio de la jornada de este miércoles, en la cual  el grupo temporal de trabajo para la prevención y control del nuevo coronavirus, que encabezan el Presidente de la República y el Primer Ministro, analizó la situación epidemiológica de las provincias de Sancti Spíritus, La Habana y Ciego de Ávila.">
    <meta name="twitter:image" content="https://www.presidencia.gob.cu/media/filer/public/2020/10/07/2020-10-07_reunion_coronavirus_7-10_er_1.jpg">



    <link href="/static/assets/plugins/bootstrap/css/bootstrap.min.css" rel="stylesheet">
    <link href="/static/assets/plugins/font-awesome/css/font-awesome.min.css" rel="stylesheet">
    <link href="/static/assets/plugins/simple-line-icons/simple-line-icons.min.css" rel="stylesheet">
    <link href="/static/assets/plugins/ssk/css/social-share-kit.min.css" rel="stylesheet">
    <link href="/static/assets/plugins/animate/animate.min.css" rel="stylesheet">
    <link href="/static/assets/plugins/swiper/css/swiper.min.css" rel="stylesheet">
    <link href="/static/assets/plugins/lightGallery/css/lightgallery.min.css" rel="stylesheet">
    <link href="/static/assets/plugins/cubanMap/ammap/ammap.min.css" rel="stylesheet">
    <link href="/static/assets/plugins/360/css/pannellum.min.css" rel="stylesheet">
    <link href="/static/assets/css/style.min.css" rel="stylesheet">
    

    <script>let _paq = window._paq || [];
    _paq.push(["trackPageView"]);
    _paq.push(["enableLinkTracking"]);
    (function () {
        let a = "//stat.presidencia.gob.cu/";
        _paq.push(["setTrackerUrl", a + "matomo.php"]);
        _paq.push(["setSiteId", "1"]);
        let e = document, c = e.createElement("script"), b = e.getElementsByTagName("script")[0];
        c.type = "text/javascript";
        c.async = !0;
        c.defer = !0;
        c.src = a + "matomo.js";
        b.parentNode.insertBefore(c, b)
    })();</script>
    <script async defer src="//telus.redcuba.cu/tracker/telus.min.js" id="telus-tracker" data-id="68"></script>
</head>
<body class="site">



    <div class="ssk-tools ssk-group ssk-sticky ssk-left ssk-center ssk-lg">
        
            <a href="/es/pdf/no-confiarse-ante-la-covid-19-clave-para-el-exito-1317/" class="ssk ssk-alizarin" title="PDF">
                <span class="iconm-generate-pdf"></span>
            </a>
        
        <a href="" class="ssk ssk-email"></a>
        <a href="" class="ssk ssk-facebook"></a>
        <a href="" class="ssk ssk-twitter"></a>
        <a href="" class="ssk ssk-pinterest"></a>
        <a class="ssk ssk-black hidden-xs" id="inc-font"><span class="iconm-increase-font"></span></a>
        <a class="ssk ssk-dark-gray hidden-xs" id="dec-font"><span class="iconm-decrease-font"></span></a>
        
    </div>




<div id="nav-header" class="header">
    <div class="progress-container">
        <div class="progress-bar" id="scrollspy_progress_bar"></div>
    </div>
    <div class="container">
        <a href="/" class="logo-presidencia"></a>
        <div class="mobi-toggler">
            <div></div>
        </div>
        
<nav class="header-navigation pull-right">
    <ul>
        

    <li class="dropdown">
        <a class="dropdown-toggle" data-toggle="dropdown"
           data-target="/es/presidencia/"
           href="/es/presidencia/">
            Presidencia <b class="caret"></b>
        </a>
        
            <ul class="dropdown-menu">
                

    <li class="">
        <a 
           href="/es/presidencia/condecoraciones/">
            Condecoraciones
        </a>
        
    </li>

    <li class="">
        <a 
           href="/es/presidencia/intervenciones/">
            Intervenciones
        </a>
        
    </li>

    <li class="">
        <a 
           href="/es/presidencia/sede/">
            Sede
        </a>
        
    </li>

            </ul>
        
    </li>

    <li class="dropdown">
        <a class="dropdown-toggle" data-toggle="dropdown"
           data-target="/es/gobierno/"
           href="/es/gobierno/">
            Gobierno <b class="caret"></b>
        </a>
        
            <ul class="dropdown-menu">
                

    <li class="">
        <a 
           href="/es/gobierno/programas-priorizados/">
            Programas
        </a>
        
    </li>

            </ul>
        
    </li>

    <li class="selected active ">
        <a 
           href="/es/noticias/">
            Noticias
        </a>
        
    </li>

    <li class="">
        <a 
           href="/es/pensar-como-pais/">
            Pensar como país
        </a>
        
    </li>

    <li class="dropdown">
        <a class="dropdown-toggle" data-toggle="dropdown"
           data-target="/es/cuba/"
           href="/es/cuba/">
            Cuba <b class="caret"></b>
        </a>
        
            <ul class="dropdown-menu">
                

    <li class="">
        <a 
           href="/es/cuba/estado-cubano/">
            Estado
        </a>
        
    </li>

    <li class="">
        <a 
           href="/es/cuba/covid-19/">
            COVID-19
        </a>
        
    </li>

            </ul>
        
    </li>

    <li class="">
        <a 
           href="/es/galerias/">
            Galerías
        </a>
        
    </li>

    <li class="">
        <a 
           href="/es/contacto/">
            Contacto
        </a>
        
    </li>

        <li class="menu-search">
            <i class="icon icon-magnifier search-btn p-0"></i>
            <div class="search-box">
                <form action="/es/search/" method="get"><input type='hidden' name='csrfmiddlewaretoken' value='d88vrRZJwYabWc3Vh57Cfl7ZsQQ0FZR9VJywcbRb0RyCCrGcJ2SFVeP4qvzWhloD' />
                    <div class="input-group">
                        <input id="id_q" name="q" class="input-text form-control" type="search"
                               placeholder="Buscar en Presidencia">
                        <span class="input-group-btn">
                      <button class="btn btn-primary" type="submit">Buscar</button>
                    </span>
                    </div>
                </form>
            </div>
        </li>
        <li class="dropdown">
            <a class="dropdown-toggle" data-toggle="dropdown">
                <i class="icon icon-globe"></i>
                es
            </a>
            <ul class="dropdown-menu">
                



<li class="lang active">
	<a href="/es/noticias/no-confiarse-ante-la-covid-19-clave-para-el-exito/" title="Change to language: Español">Español</a>
</li>

<li class="lang">
	<a href="/es/noticias/no-confiarse-ante-la-covid-19-clave-para-el-exito/" title="Change to language: English">English</a>
</li>



            </ul>
        </li>
        <li class="social-networks">
            <div class="social-icon">
                <a class="icon icon-social-facebook color-white hover-facebook"
                   href="https://www.facebook.com/PresidenciaDeCuba"
                   title="facebook" target="_blank"></a>
                <a class="icon icon-social-youtube color-white hover-youtube"
                   href="https://www.youtube.com/channel/UCFD9TPE5iL0_g3CqYdnXoWA"
                   title="youtube" target="_blank"></a>
                <a class="icon icon-social-instagram color-white hover-instagram"
                   href="https://www.instagram.com/presidenciadecuba/"
                   title="instagram" target="_blank"></a>
                <a class="icon icon-social-twitter color-white hover-twitter"
                   href="https://twitter.com/PresidenciaCuba"
                   title="twitter" target="_blank"></a>
                <a class="icon icon-feed color-white hover-feed"
                   href="/es/feed/"
                   title="feed" target="_blank"></a>
            </div>
        </li>
    </ul>
</nav>
    </div>
</div>

    
    <section class="bg-color-sky-light content-sm-both container" style="border-radius: 6px">
        
    
    <div class="row container" id="parent-sidebar">
        <div class="col-lg-9 col-md-9 col-sm-12 col-xs-12 blog-posts md-margin-b-20">
            
            
    




    <article >
        <h1>
            No confiarse ante la COVID-19, clave para el éxito<br>
            
                
                    
                    <small><a class="category" href="/es/noticias/category/nacionales/">Nacionales</a></small>
                    
                
            
        </h1>
        <ul class="blog-info md-center">
            
                <li>


    
    
    


    
        <a href="/es/noticias/author/yaima-puig/">
            <img class="img-circle margin-r-5"
                 src="/media/filer_thumbnails/filer_public_thumbnails/public/2019/01/08/yaima.jpg__40x40_q85_crop_subsampling-2_upscale.jpg"
                 height="40" width="40" alt="Yaima Puig Meneses">
            <span>Yaima Puig Meneses</span>
        </a>
    
</li>
            
            <li>
                <i class="icon icon-calendar"></i>
                <time datetime="2020-10-07 20:54:16">7 de Octubre de 2020</time>
            </li>
        </ul>
        
            <ul class="blog-info md-center">
                <li><span class="hidden-xs">En este articulo:</span>
                    
                        <a href="/es/noticias/tag/salud/">
                            <i class="icon icon-tag"></i>&nbsp;Salud&nbsp;
                        </a>
                    
                        <a href="/es/noticias/tag/covid-19/">
                            <i class="icon icon-tag"></i>&nbsp;COVID-19&nbsp;
                        </a>
                    
                        <a href="/es/noticias/tag/reunion-de-trabajo/">
                            <i class="icon icon-tag"></i>&nbsp;Reunión de Trabajo
                        </a>
                    
                </li>
            </ul>
        
        
            <blockquote>
                <small>Una tendencia a la disminución de los pacientes positivos a la COVID-19 que se encuentran hospitalizados marcó el inicio de la jornada de este miércoles, en la cual  el grupo temporal de trabajo para la prevención y control del nuevo coronavirus, que encabezan el Presidente de la República y el Primer Ministro, analizó la situación epidemiológica de las provincias de Sancti Spíritus, La Habana y Ciego de Ávila.</small>
            </blockquote>
        
        <div class="margin-b-20">
            <div class="article-featured-image box-shadow">
                
                    <img class="full-width img-responsive img-rounded"
                         src="/media/filer_thumbnails/filer_public_thumbnails/public/2020/10/07/2020-10-07_reunion_coronavirus_7-10_er_1.jpg__830x600_q85_subject_location-544%2C1232_subsampling-2.jpg" width="100%" height="auto"
                         alt="No confiarse ante la COVID-19, clave para el éxito"
                         title="No confiarse ante la COVID-19, clave para el éxito">
                    
                        <div class="img-caption img-rounded-buttom">
                            <p class="small help-block">Estudios Revolución</p>
                        </div>
                    
                
            </div>
        </div>
        <div class="text-justify"><p>Cuatrocientas cincuenta y dos personas positivas a la COVID-19 permanecían hospitalizadas este miércoles en Cuba, cifra que ha comenzado a disminuir en los últimos días y se confirma como la más baja desde que en el mes de septiembre se alcanzó el pico máximo de personas ingresadas a causa del nuevo brote de la epidemia.</p>

<p>Según confirmó el ministro de Salud Pública, José Angel Portal Miranda, la mayoría de esos pacientes pertenecen a Sancti Spíritus, La Habana y Ciego de Ávila, tres de las provincias que diagnosticaron nuevos casos en esta jornada y por tanto centraron los análisis del grupo temporal de trabajo para la prevención y control del nuevo coronavirus, encabezado por el Presidente de la República, Miguel Díaz-Canel Bermúdez, y el Primer Ministro, Manuel Marrero Cruz.</p>

<p>Al referirse de manera particular a Sancti Spíritus, en fase de transmisión autóctona limitada de la enfermedad, y que en la actualidad manifiesta el escenario epidemiológico más complejo del país, el titular de Salud Pública informó sobre el cierre del evento de transmisión local en la comunidad Bernal, del municipio de Jatibonico.</p>

<p>A su vez, dio a conocer la apertura de uno nuevo en el área de salud Norte del municipio cabecera de esa misma provincia, en el cual, aun cuando no reportaron casos positivos en la jornada, se acumulaban 11 pacientes confirmados. De las muestras PCR que allí se habían realizado –puntualizó- 151 han sido negativas y estaban pendientes los resultados de 31 personas.</p>

<p>De esta manera, precisó Portal Miranda, en el país se mantienen activos 15 eventos de transmisión local, de los 92 que se han abierto desde que comenzó el enfrentamiento a la epidemia en Cuba.</p>

<p>Puntualmente sobre Sancti Spíritus, donde se han diagnosticado 139 pacientes positivos a la enfermedad en las últimas dos semanas, periodo en el que la tasa de incidencia por cada 100 000 habitantes se incrementa a 29,83, la gobernadora Teresita Romero Rodríguez comentó que allí se profundiza en los estudios poblacionales de las áreas de mayor complejidad, asociadas fundamentalmente a lugares donde residen contactos de casos positivos o sospechosos.</p>

<p>En los días más recientes, aseguró, se ha incrementado el número de muestras PCR en la provincia con el propósito de detectar la mayor cantidad de personas contagiadas y poder contener la transmisión de la enfermedad.</p>

<p>Por otra parte, el gobernador de La Habana, Reinaldo García Zapata, reconoció que si bien en la jornada la capital solo reportó un nuevo caso positivo en 4 211 muestras PCR procesadas, ello no constituye motivo para la confianza. Ahora es cuando menos confiados tenemos que estar, que cada cual, desde su responsabilidad, cumpla lo que le corresponde para lograr contener los contagios, valoró.</p>

<p>En el territorio, dijo, se continúa el seguimiento constante a la situación que presentan los 53 controles de focos activos, en los cuales se trabaja con minuciosidad. Atención diferenciada se brinda a aquellos que tienen cinco o más casos positivos, de manera que ninguna situación nos tome por sorpresa, señaló.</p>

<p>Particularmente sobre este aspecto, el Ministro de Salud Pública subrayó que el balance en la capital sigue siendo positivo, pues durante las últimas jornadas son más los controles que se cierran que los que se inician. En esta ocasión se dieron por concluidos cinco, en igual número de municipios, y solo se abrió uno en Regla.</p>

<p>Al informar sobre la situación de Ciego de Ávila, provincia donde poco a poco se ha ido retomando el control de la epidemia, el gobernador Tomás Alexis Martín Venegas precisó que los tres casos informados el miércoles tenían fuente de infección conocida y se encontraban ingresados en centros de aislamiento para contactos.</p>

<p>Como un elemento esperanzador refirió que en los últimos seis días solo se han confirmado nuevos pacientes en la ciudad capital, los otros nueve municipios se mantenían sin notificar casos. Durante la jornada se concedieron 23 altas clínicas y no había personas en estado grave ni crítico en las 204 que permanecían hospitalizadas contagiadas con la enfermedad.</p>

<p>En la sesión de trabajo, el grupo temporal también dio seguimiento a la recuperación en las provincias Pinar del Río y  Artemisa, así como en el municipio especial Isla de la Juventud, tras el paso del huracán Delta por las inmediaciones de estos territorios.</p>

<p>Según dieron a conocer las respectivas autoridades gubernamentales, las afectaciones de manera general fueron mínimas y se concentran fundamentalmente en el servicio eléctrico y la agricultura. El saldo de lluvias para las presas y embalses no ocasionó riesgos de inundaciones y ya regresan a sus hogares la mayoría de las personas evacuadas.</p>

<p>La premisa de trabajo en el país, tanto para el enfrentamiento a la COVID-19 como en la recuperación de las afectaciones provocadas por el evento climatológico, sigue estando marcada por el estricto cumplimiento de las medidas higiénico-sanitarias, esenciales para evitar nuevos brotes de la enfermedad en Cuba.</p></div>
    </article>





        </div>
        <aside class="col-lg-3 col-md-3 col-sm-12 col-xs-12 blog-sidebar md-center" id="sidebar">
            
    
<h2 class="margin-top-15">Artículos Relacionados</h2>

    <div class="grid-content grid-related-news">
        
            
            
            <article>
                <h5><a href="/es/noticias/nota-informativa-del-ministerio-de-salud-publica-07-10/">Nota informativa del Ministerio de Salud Pública (07/10)</a></h5>
                <a href="/es/noticias/nota-informativa-del-ministerio-de-salud-publica-07-10/">
                    <img class="img-responsive img-rounded full-width" src="/media/filer_thumbnails/filer_public_thumbnails/public/2020/08/28/informacion_oficial_optimized.png__250x200_q85_crop_subject_location-640%2C360_subsampling-2.jpg"
                         alt="Nota informativa del Ministerio de Salud Pública"
                         title="Nota informativa del Ministerio de Salud Pública">
                </a>
                <ul class="blog-info">
                    <li>
                        
                            por
                            <a href="/es/noticias/author/presidencia/">Presidencia</a><br>
                        
                        <i class="icon icon-calendar"></i>
                        <time datetime="2020-10-07 10:47:39">7 de Octubre de 2020</time>
                    </li>
                </ul>
            </article>
            
            
            
            <article>
                <h5><a href="/es/noticias/cuba-entre-la-covid-19-y-el-huracan-delta/">Cuba, entre la COVID-19 y el huracán Delta</a></h5>
                <a href="/es/noticias/cuba-entre-la-covid-19-y-el-huracan-delta/">
                    <img class="img-responsive img-rounded full-width" src="/media/filer_thumbnails/filer_public_thumbnails/public/2020/10/06/er_03_coronavirus_06_10_2020.jpg__250x200_q85_crop_subject_location-1703%2C137_subsampling-2.jpg"
                         alt="None"
                         title="None">
                </a>
                <ul class="blog-info">
                    <li>
                        
                            por
                            <a href="/es/noticias/author/leticia-martinez/">Leticia Martínez Hernández</a><br>
                        
                        <i class="icon icon-calendar"></i>
                        <time datetime="2020-10-06 20:26:36">6 de Octubre de 2020</time>
                    </li>
                </ul>
            </article>
            
            
            
            <article>
                <h5><a href="/es/noticias/retrocede-sancti-spiritus-a-la-fase-de-transmision-autoctona-limitada-de-la-covid-19/">Retrocede Sancti Spíritus a la fase de transmisión autóctona limitada de la COVID-19</a></h5>
                <a href="/es/noticias/retrocede-sancti-spiritus-a-la-fase-de-transmision-autoctona-limitada-de-la-covid-19/">
                    <img class="img-responsive img-rounded full-width" src="/media/filer_thumbnails/filer_public_thumbnails/public/2020/10/05/202010-05_er_03_coronavirus_05_10_2020.jpg__250x200_q85_crop_subject_location-606%2C663_subsampling-2.jpg"
                         alt="Retrocede Sancti Spíritus a la fase de transmisión autóctona limitada de la COVID-19"
                         title="Retrocede Sancti Spíritus a la fase de transmisión autóctona limitada de la COVID-19">
                </a>
                <ul class="blog-info">
                    <li>
                        
                            por
                            <a href="/es/noticias/author/yaima-puig/">Yaima Puig Meneses</a><br>
                        
                        <i class="icon icon-calendar"></i>
                        <time datetime="2020-10-05 21:22:22">5 de Octubre de 2020</time>
                    </li>
                </ul>
            </article>
            
            
            
            <article>
                <h5><a href="/es/noticias/compartiendo-responsabilidades-podemos-seguir-avanzando-hacia-una-situacion-mas-favorable/">Compartiendo responsabilidades podemos seguir avanzando hacia una situación más favorable</a></h5>
                <a href="/es/noticias/compartiendo-responsabilidades-podemos-seguir-avanzando-hacia-una-situacion-mas-favorable/">
                    <img class="img-responsive img-rounded full-width" src="/media/filer_thumbnails/filer_public_thumbnails/public/2020/10/03/2020-10-03_er_02_coronavirus_03_10_2020.jpg__250x200_q85_crop_subject_location-1726%2C663_subsampling-2.jpg"
                         alt="Compartiendo responsabilidades podemos seguir avanzando hacia una situación más favorable"
                         title="Compartiendo responsabilidades podemos seguir avanzando hacia una situación más favorable">
                </a>
                <ul class="blog-info">
                    <li>
                        
                            por
                            <a href="/es/noticias/author/yaima-puig/">Yaima Puig Meneses</a><br>
                        
                        <i class="icon icon-calendar"></i>
                        <time datetime="2020-10-03 17:24:43">3 de Octubre de 2020</time>
                    </li>
                </ul>
            </article>
            
    </div>



        </aside>
    </div>
    


    </section>
    
<div class="container">
    <img style="margin: 20px" src="data:image/svg+xml,%3Csvg version='1.1' id='Capa_1' xmlns='http://www.w3.org/2000/svg' xmlns:xlink='http://www.w3.org/1999/xlink' x='0px' y='0px' viewBox='0 0 1152 45' style='enable-background:new 0 0 1152 45;' xml:space='preserve'%3E%3Cstyle type='text/css'%3E .st0%7Bfill:%2310324E;%7D%0A%3C/style%3E%3Cg id='separator_new'%3E%3Cg id='line'%3E%3Cg id='XMLID_47_'%3E%3Cpath id='XMLID_63_' class='st0' d='M538.6,27L1.4,27C0.6,27,0,26.4,0,25.6v0c0-0.8,0.6-1.4,1.4-1.4h537.2c0.8,0,1.4,0.6,1.4,1.4 v0C540,26.4,539.4,27,538.6,27z'/%3E%3C/g%3E%3Cg id='XMLID_5_'%3E%3Cpath id='XMLID_20_' class='st0' d='M1150.6,27l-537.2,0c-0.8,0-1.4-0.6-1.4-1.4v0c0-0.8,0.6-1.4,1.4-1.4h537.2 c0.8,0,1.4,0.6,1.4,1.4v0C1152,26.4,1151.4,27,1150.6,27z'/%3E%3C/g%3E%3C/g%3E%3Cg id='star'%3E%3Cpath id='XMLID_29_' class='st0' d='M542,0c-1.1,0-1.4,0.7-0.8,1.6l33.5,41.9c0.7,0.9,1.8,0.9,2.5,0l33.5-41.9 c0.7-0.9,0.3-1.6-0.8-1.6H542z M582.9,17.9c-0.9,0.6-1.7,1.3-2.6,1.9c-0.1,0.1-0.2,0.2-0.1,0.4c0.7,2.1,1.3,4.3,1.9,6.4 c0,0,0,0.1,0,0.1c0,0.1,0,0.2-0.1,0.2c-0.1,0-0.2,0-0.3,0c-0.4-0.2-0.7-0.5-1.1-0.8c-1.5-1-3-2.1-4.5-3.1c-0.2-0.1-0.3-0.1-0.4,0 c-1,0.7-2,1.4-3,2.1c-0.9,0.6-1.7,1.2-2.6,1.8c-0.1,0.1-0.2,0.1-0.3,0c-0.1-0.1-0.1-0.2-0.1-0.3c0.4-1.4,0.8-2.7,1.2-4.1 c0.2-0.8,0.5-1.6,0.7-2.4c0-0.1,0-0.1,0-0.1c0-0.1,0-0.2,0-0.3c0,0,0,0-0.1-0.1c-0.8-0.6-1.6-1.2-2.5-1.9 c-0.9-0.7-1.8-1.4-2.8-2.1c-0.1-0.1-0.2-0.1-0.3-0.2c-0.1-0.1-0.1-0.2-0.1-0.2c0-0.1,0.1-0.2,0.2-0.2c0.7,0,1.4,0,2.1,0 c1.6,0,3.3-0.1,4.9-0.1c0.2,0,0.2-0.1,0.3-0.2c0.4-1,0.7-2.1,1.1-3.1c0.4-1.1,0.7-2.2,1.1-3.2c0-0.1,0.1-0.3,0.3-0.3 c0.2,0,0.2,0.1,0.3,0.2c0.7,2.1,1.5,4.2,2.2,6.3c0.1,0.3,0.1,0.3,0.4,0.3c1.3,0,2.6,0.1,4,0.1c0.9,0,1.9,0,2.8,0.1 c0.1,0,0.2,0,0.3,0.2c0,0.1,0,0.2-0.1,0.3C584.8,16.4,583.8,17.2,582.9,17.9z'/%3E%3C/g%3E%3C/g%3E%3C/svg%3E">
</div>


<footer>
    <div class="swiper-linear-gradient">
        <div class="content-sm container">
            <div class="swiper-slider swiper">
                <div class="swiper-wrapper">
                    <div class="swiper-slide">
                        <a target="_blank" href="http://www.fidelcastro.cu/">
                            <div class="sprite-redcuba fidel"></div>
                        </a>
                    </div>
                    <div class="swiper-slide">
                        <a target="_blank" href="http://www.cubadebate.cu/autor/raul-castro-ruz/">
                            <div class="sprite-redcuba raul"></div>
                        </a>
                    </div>
                    <div class="swiper-slide">
                        <a target="_blank" href="http://www.granma.cu/">
                            <div class="sprite-redcuba granma"></div>
                        </a>
                    </div>
                    <div class="swiper-slide">
                        <a target="_blank" href="http://www.juventudrebelde.cu/">
                            <div class="sprite-redcuba juventudrebelde"></div>
                        </a>
                    </div>
                    <div class="swiper-slide">
                        <a target="_blank" href="http://www.cubaeduca.cu/">
                            <div class="sprite-redcuba cubaeduca"></div>
                        </a>
                    </div>
                    <div class="swiper-slide">
                        <a target="_blank" href="http://www.cubarte.cult.cu/">
                            <div class="sprite-redcuba cubarte"></div>
                        </a>
                    </div>
                    <div class="swiper-slide">
                        <a target="_blank" href="http://www.ecured.cu/">
                            <div class="sprite-redcuba ecured"></div>
                        </a>
                    </div>
                    <div class="swiper-slide">
                        <a target="_blank" href="http://infomed.sld.cu/">
                            <div class="sprite-redcuba infomed"></div>
                        </a>
                    </div>
                    <div class="swiper-slide">
                        <a target="_blank" href="https://www.cuba.travel/">
                            <div class="sprite-redcuba cubatravel"></div>
                        </a>
                    </div>
                    <div class="swiper-slide">
                        <a target="_blank" href="http://www.cuba.cu/">
                            <div class="sprite-redcuba cubacu"></div>
                        </a>
                    </div>
                    <div class="swiper-slide">
                        <a target="_blank" href="http://www.cubacrece.gob.cu/">
                            <div class="sprite-redcuba cubacrece"></div>
                        </a>
                    </div>
                </div>
            </div>
        </div>
    </div>
    <div class="pre-footer">
        <div class="container padding-top-25">
            <div class="row center">
                <div class="margin-b-20">
                    <a class="sprite-prefooter minrex" target="_blank" href="http://www.minrex.gob.cu/"
                       title="Ministerio de Relaciones Exteriores de Cuba"></a>
                    <a class="sprite-prefooter tribunal" target="_blank" href="http://www.tsp.gob.cu/es"
                       title="Tribunal Supremo Popular de la República de Cuba"></a>
                    <a class="sprite-prefooter gaceta" target="_blank" href="https://www.gacetaoficial.gob.cu/"
                       title="Gaceta Oficial de la República de Cuba"></a>
                    <a class="sprite-prefooter asamblea" target="_blank" href="http://www.parlamentocubano.cu/"
                       title="Asamblea Nacional del Poder Popular"></a>
                    <a class="sprite-prefooter pcc" target="_blank" href="http://www.pcc.cu/"
                       title="Partido Comunista de Cuba"></a>
                    <a class="sprite-prefooter contraloria" target="_blank" href="http://www.contraloria.gob.cu/"
                       title="Contraloría General de la República de Cuba"></a>
                    <a class="sprite-prefooter fiscalia" target="_blank" href="http://www.fgr.gob.cu/"
                       title="Fiscalía General de la República de Cuba"></a>
                    <a class="sprite-prefooter redcuba" target="_blank" href="https://www.redcuba.cu/"
                       title="contenidos unificados para búsqueda avanzada"></a>
                </div>
                <div class="col-lg-4 col-md-4 col-sm-4 col-xs-12 margin-b-20">
                    <h4>Contacto</h4>
                    <address class="list-unstyled list-inline">
                        <span style="font-family: Montserrat-Semibold, sans-serif">Correo electrónico:</span><br>
                        <a target="_blank" href="mailto:despacho@presidencia.gob.cu">despacho@presidencia.gob.cu</a>
                    </address>
                </div>
                <div class="col-lg-4 col-md-4 col-sm-4 col-xs-12 margin-b-20">
                    <h4>Palacio de La Revolución</h4>
                    <address class="list-unstyled list-inline">
                        <span style="font-family: Montserrat-Semibold, sans-serif">Dirección:</span><br>
                        <small>
                            Ave Paseo # 1040 B e/ Carlos Manuel de Céspedes e Independencia, Plaza de La Revolución
                        </small>
                    </address>
                </div>
                <div class="col-lg-4 col-md-4 col-sm-4 col-xs-12 margin-b-20">
                    <h4>Soporte técnico</h4>
                    <address class="list-unstyled list-inline">
                        <span style="font-family: Montserrat-Semibold, sans-serif">Correo electrónico:</span><br>
                        <a target="_blank" href="mailto:webmaster@presidencia.gob.cu">webmaster@presidencia.gob.cu</a>
                    </address>
                </div>
                <div class="col-lg-12 col-md-12 col-sm-12 col-xs-12">
                    <ul class="menu-footer list-unstyled list-inline" style="text-transform: uppercase">
                        

    <li class="">
        <a 
           href="/es/presidencia/">
            Presidencia
        </a>
        
    </li>

    <li class="">
        <a 
           href="/es/gobierno/">
            Gobierno
        </a>
        
    </li>

    <li class="selected active ">
        <a 
           href="/es/noticias/">
            Noticias
        </a>
        
    </li>

    <li class="">
        <a 
           href="/es/pensar-como-pais/">
            Pensar como país
        </a>
        
    </li>

    <li class="">
        <a 
           href="/es/cuba/">
            Cuba
        </a>
        
    </li>

    <li class="">
        <a 
           href="/es/galerias/">
            Galerías
        </a>
        
    </li>

    <li class="">
        <a 
           href="/es/contacto/">
            Contacto
        </a>
        
    </li>

                    </ul>
                </div>
                <div class="col-lg-12 col-md-12 col-sm-12 col-xs-12">
                    <ul class="list-unstyled list-inline">
                        <li>
                            <a title="facebook" target="_blank"
                               href="https://www.facebook.com/PresidenciaDeCuba">
                                <i class="icon icon-social-facebook fa-2x"></i></a>
                        </li>
                        <li>
                            <a title="youtube" target="_blank"
                               href="https://www.youtube.com/channel/UCFD9TPE5iL0_g3CqYdnXoWA">
                                <i class="icon icon-social-youtube fa-2x"></i></a>
                        </li>
                        <li>
                            <a title="instagram" target="_blank"
                               href="https://www.instagram.com/presidenciadecuba/">
                                <i class="icon icon-social-instagram fa-2x"></i></a>
                        </li>
                        <li>
                            <a title="twitter" target="_blank"
                               href="https://twitter.com/PresidenciaCuba">
                                <i class="icon icon-social-twitter fa-2x"></i></a>
                        </li>
                        <li>
                            <a title="RSS" target="_blank"
                               href="/es/feed/">
                                <i class="icon icon-feed fa-2x"></i></a>
                        </li>
                    </ul>
                </div>
            </div>
        </div>
    </div>
    <div class="footer">
        <div class="d-flex justify-content-center align-items-center">
            <i class="cuba-brand-white margin-r-10" style="width: 45px;height: 20px"></i>
            <span>2020 © Palacio de La Revolución.</span>
        </div>
    </div>
</footer>
<script src="/static/assets/plugins/respond.min.js"></script>
<script src="/static/assets/plugins/jquery.min.js"></script>
<script src="/static/assets/plugins/jquery-migrate.min.js"></script>
<script src="/static/assets/plugins/bootstrap/js/bootstrap.min.js"></script>
<script src="/static/assets/plugins/jquery.easing.js"></script>
<script src="/static/assets/plugins/smooth-scroll/smooth-scroll.min.js"></script>
<script src="/static/assets/plugins/jquery.wow.min.js"></script>
<script src="/static/assets/plugins/swiper/js/swiper.jquery.min.js"></script>
<script src="/static/assets/plugins/masonry/jquery.masonry.pkgd.min.js"></script>
<script src="/static/assets/plugins/masonry/imagesloaded.pkgd.min.js"></script>
<script src="/static/assets/plugins/sticky-kit/jquery.sticky-kit.min.js"></script>
<script src="/static/assets/plugins/lightGallery/js/lightgallery.min.js"></script>
<script src="/static/assets/plugins/lightGallery/js/lg-video.min.js"></script>
<script src="/static/assets/plugins/ssk/js/social-share-kit.min.js"></script>
<script src="/static/assets/plugins/wavesurfer/wavesurfer.min.js"></script>
<script src="/static/assets/js/layout.min.js"></script>





</body>
</html>