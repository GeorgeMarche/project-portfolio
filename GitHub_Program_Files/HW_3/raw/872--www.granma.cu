#include "hip/hip_runtime.h"
<!DOCTYPE html>
<html lang="es">

<head prefix="og: http://ogp.me/ns#">
    <meta charset="utf-8">
	<meta name="google-site-verification" content="y_GTvuIlKGzzKjky35SOfpzPwzpBpGsEpE7bwkz0XP8" /> 

    <meta name="viewport" content="width=device-width, initial-scale=1.0">
    <meta name="description" content="Órgano Oficial del Comité Central del Partido Comunista de Cuba. Fundado el 3 de octubre de 1965. Actualización de noticias 24 horas.">
	<meta name="keywords" content="Cuba, Noticias, Diario Granma, Órgano oficial, Revolución Cubana, Fidel Castro, Raúl Castro, Izquierda">
	<link rel="canonical" href="http://www.granma.cu/">
    <base href="http://www.granma.cu/" />
	<meta >

    <link rel="shortcut icon" href="/static/ico/favicon.png">
  

  <!-- Etiqueta global de sitio (gtag.js) de Google Analytics -->

	
	
	    <title>Cuba informará en próximos días sobre proceso de unificación monetaria y cambiaria &rsaquo; Cuba frente a la Covid-19 &rsaquo; Granma - Órgano oficial del PCC</title>
        <style>
        @font-face {
            font-family: 'Amplitude Medium';
            src: url('/static/fonts/amplitudemedium-webfont.eot');
            src: url('/static/fonts/amplitudemedium-webfont.eot?#iefix') format('embedded-opentype'),
            url('/static/fonts/amplitudemedium-webfont.woff') format('woff'),
            url('/static/fonts/amplitudemedium-webfont.ttf') format('truetype'),
            url('/static/fonts/amplitudemedium-webfont.svg#Amplitude Medium') format('svg');
            font-weight: normal;
            font-style: normal;

        }

        @font-face {
            font-family: 'amplitude-regularregular';
            src: url('/static/fonts/amplituderegular-webfont.eot');
            src: url('/static/fonts/amplituderegular-webfont.eot?#iefix') format('embedded-opentype'),
            url('/static/fonts/amplituderegular-webfont.woff') format('woff'),
            url('/static/fonts/amplituderegular-webfont.ttf') format('truetype'),
            url('/static/fonts/amplituderegular-webfont.svg#amplitude-regularregular') format('svg');
            font-weight: normal;
            font-style: normal;

        }

        @font-face {
            font-family: 'Open Sans';
            src: url('/static/fonts/opensans-webfont.eot');
            src: url('/static/fonts/opensans-webfont.eot?#iefix') format('embedded-opentype'),
            url('/static/fonts/opensans-webfont.woff') format('woff'),
            url('/static/fonts/opensans-webfont.ttf') format('truetype'),
            url('/static/fonts/opensans-webfont.svg#Open Sans') format('svg');
            font-weight: normal;
            font-style: normal;

        }

    </style>
    <!-- Bootstrap core CSS -->
    <link href="/static/css/bootstrap.min.css" rel="stylesheet">
	
    <link href="/static/css/infografia.css" rel="stylesheet">
    <!-- Custom styles for this template -->
    <link href="/static/css/styles.css" rel="stylesheet">
	
	    <link rel="stylesheet" href="/static/css/linea/stylelinea.css">


    <link rel="stylesheet" type="text/css" href="/static/css/bar-porcent.css">

    
    <!-- HTML5 shim and Respond.js IE8 support of HTML5 elements and media queries -->
<!--[if lt IE 9]>
<script src="/static/js/html5shiv.js"></script>
<script src="/static/js/respond.min.js"></script>
<![endif]-->




    <!-- BEGIN Open Graph tags -->
    <meta property='og:type' content='article' />
    <meta property='og:title' content='Cuba informará en próximos días sobre proceso de unificación monetaria y cambiaria' />
    <meta property="og:site_name" content="Granma.cu"/>
    <meta property='og:url' content='http://www.granma.cu/cuba-covid-19/2020-10-08/cuba-informara-en-proximos-dias-sobre-proceso-de-unificacion-monetaria-y-cambiaria' />
    <meta property="og:description" content="Se dará la información completa de este complejo proceso de alcance estratégico que, aunque no es una solución mágica a todos los problemas de nuestra economía, sí ayudara a destrabar mucho de lo que es necesario ahora para que el país avance"/>
			<meta property='og:image' content='http://www.granma.cu//file/img/2014/09/medium/f0019252.jpg' />
	    <meta property='og:locale' content='es_ES'/>
          <meta property="article:tag" content="Cuba">
          <meta property="article:tag" content="Moneda Nacional">
      
<!-- END Open Graph tags -->

<!-- BEGIN Twitter Card -->
    <meta name="twitter:card" content="summary_large_image" />
    <meta name="twitter:site" content="@Granma_Digital" />
    <meta name="twitter:title" content="Cuba informará en próximos días sobre proceso de unificación monetaria y cambiaria" />
    <meta name="twitter:description" content="Se dará la información completa de este complejo proceso de alcance estratégico que, aunque no es una solución mágica a todos los problemas de nuestra economía, sí ayudara a destrabar mucho de lo que es necesario ahora para que el país avance" />
    <!-- <meta name="twitter:image" content="http://www.granma.cu//file/img/2014/09/medium/f0019252.jpg" /> -->
    <meta name="twitter:image" content="http://www.granma.cu//file/img/2014/09/medium/f0019252.jpg" />
    <meta name="twitter:url" content="http://www.granma.cu/cuba-covid-19/2020-10-08/cuba-informara-en-proximos-dias-sobre-proceso-de-unificacion-monetaria-y-cambiaria" />
   
<!-- END Twitter Card -->

</head>

<body>
    <div class="container">
        <!-- Static navbar -->
        <div class="navbar navbar-default navbar-static-top g-top-menu g-menus">
            <div class="navbar-header">
                <button type="button" class="navbar-toggle" data-toggle="collapse" data-target=".navbar-collapse-top">
                    <span class="language-globe"></span>
                </button>
                <a class="navbar-brand" href="/multimedia">MULTIMEDIA</a><span class="navbar-brand">|</span>
                <a class="navbar-brand" href="/blogs">BLOGS</a><span class="navbar-brand">|</span>
                <a class="navbar-brand" href="/impreso">EDICI&Oacute;N IMPRESA</a><span class="navbar-brand">|</span>
                 <a class="navbar-brand" href="/webviejo">SITIO WEB DE AÑOS ANTERIORES</a><span class="navbar-brand"></span>
				
                <!--a class="navbar-brand" href="/impresosesp">IMPRESOS ESPECIALES</a-->
            </div>
            <div class="navbar-collapse collapse navbar-collapse-top">
                <ul class="nav navbar-nav navbar-right">
                    <li class="text-right"><a target="_blank" href="http://en.granma.cu">English</a></li>
                    <li class="visible-md visible-lg"><span class="g-spr">|</span></li>
                    <li class="text-right"><a target="_blank" href="http://fr.granma.cu">Fran&ccedil;ais</a></li>
                    <li class="visible-md visible-lg"><span class="g-spr">|</span></li>
                    <li class="text-right"><a target="_blank" href="http://de.granma.cu">Deutsch</a></li>
                    <li class="visible-md visible-lg"><span class="g-spr">|</span></li>
                    <li class="text-right"><a target="_blank" href="http://pt.granma.cu">Portugu&ecirc;s</a></li>
                    <li class="visible-md visible-lg"><span class="g-spr">|</span></li>
                    <li class="text-right"><a target="_blank" href="http://it.granma.cu">Italiano</a></li>
                </ul>
            </div><!--/.nav-collapse -->
        </div>
        <div class="row g-mid-header">
            <div class="col-md-3 col-sm-3 g-header-logo"><a href="/"><img src="/static/img/granma-logo.png" class="img-responsive" alt="Responsive image"></a></div>
            <div class="col-md-2 col-sm-2 hidden-xs g-header-heroes"><img src="/static/img/heroes-cabezal.png" class="img-responsive" alt="Responsive image"></div>
            <div class="col-md-3 col-sm-3 g-header-titles"><p class="g-mid-header-date hidden-xs">LA HABANA, 14 DE OCTUBRE DE 2020</p><h5 class="g-mid-header-subtitle">&Oacute;RGANO OFICIAL DEL COMIT&Eacute; CENTRAL<br class="hidden-xs"/> DEL PARTIDO COMUNISTA DE CUBA</h5></div>
            <div class="col-md-4 col-sm-4 navbar-right g-header-timesocial hidden-xs">
                <div class=""><p class="g-header-timetitle text-right">ÚLTIMA ACTUALIZACIÓN</p><p class="g-header-time text-right">  01:15:54</p></div>
                <div class="pull-right g-header-social">
                    <a target="_blank" href="https://www.facebook.com/granmadigital" class="g-mid-header-fb"><span class="sr-only">facebook</span></a>
                    <a target="_blank" href="https://twitter.com/Granma_Digital" class="g-mid-header-tw"><span class="sr-only">twitter</span></a>
                    <a target="_blank" href="https://instagram.com/granmadigital" class="g-mid-header-ig"><span class="sr-only">instagram</span></a>
                    <a target="_blank" href="https://youtube.com/channel/UCCXe7yEuB8qy34HiRLp7I4A" class="g-mid-header-yt"><span class="sr-only">youtube</span></a>
                    <a target="_blank" href="http://t.me/periodicogranma" class="g-mid-header-tg"><span class="sr-only">telegram</span></a>
                    <a target="_blank" href="/feed" class="g-mid-header-rss"><span class="sr-only">rss</span></a>
                </div>
            </div>
        </div>

        <nav class="navbar navbar-default g-menus g-main-menu" role="navigation">
  <!-- Brand and toggle get grouped for better mobile display -->
  <div class="navbar-header">
    <button type="button" class="navbar-toggle" data-toggle="collapse" data-target=".navbar-main-collapse">
      <span class="sr-only">Toggle navigation</span>
      <span class="icon-bar"></span>
      <span class="icon-bar"></span>
      <span class="icon-bar"></span>
    </button>
    <a class="navbar-brand visible-xs" href="#">Cuba frente a la Covid-19</a>
  </div>

  <!-- Collect the nav links, forms, and other content for toggling -->
  <div class="collapse navbar-collapse navbar-main-collapse">

    <ul class="nav navbar-nav">    
      <li ><a href="/">Portada</a></li>
          <li ><a href="/cuba">Cuba</a></li>
          <li ><a href="/mundo">Mundo</a></li>
          <li ><a href="/deportes">Deportes</a></li>
          <li ><a href="/cultura">Cultura</a></li>
          <li ><a href="/opinion">Opinión</a></li>
          <li ><a href="/ciencia">Ciencia</a></li>
          <li ><a href="/salud">Salud</a></li>
    	
					<li class="dropdown"><a class="dropdown-toggle" data-toggle="dropdown" href="javascript:void(0);" aria-expanded="false">ESPECIALES ▼ </a>
			        <ul class="dropdown-menu">
			          			            <!--li ><a href="/cuba-china">Cuba y China</a></li-->

						<li class="" style=" font-size: 80%;  text-transform: uppercase;" ><a href="/cuba-china">Cuba y China</a></li>
						<li class="divider" ></li>
			          			            <!--li class="active"><a href="/cuba-covid-19">Cuba frente a la Covid-19</a></li-->

						<li class="" style=" font-size: 80%;  text-transform: uppercase;" class="active"><a href="/cuba-covid-19">Cuba frente a la Covid-19</a></li>
						<li class="divider" ></li>
			          			            <!--li ><a href="/conexion-toxio-2020">Conexión Tokio 2020</a></li-->

						<li class="" style=" font-size: 80%;  text-transform: uppercase;" ><a href="/conexion-toxio-2020">Conexión Tokio 2020</a></li>
						<li class="divider" ></li>
			          			            <!--li ><a href="/panamericanos-lima-2019">Panamericanos Lima 2019</a></li-->

						<li class="" style=" font-size: 80%;  text-transform: uppercase;" ><a href="/panamericanos-lima-2019">Panamericanos Lima 2019</a></li>
						<li class="divider" ></li>
			          			            <!--li ><a href="/gracias-por-la-vida">Más que médicos</a></li-->

						<li class="" style=" font-size: 80%;  text-transform: uppercase;" ><a href="/gracias-por-la-vida">Más que médicos</a></li>
						<li class="divider" ></li>
			          			            <!--li ><a href="/reforma-constitucional">Reforma Constitucional en Cuba</a></li-->

						<li class="" style=" font-size: 80%;  text-transform: uppercase;" ><a href="/reforma-constitucional">Reforma Constitucional en Cuba</a></li>
						<li class="divider" ></li>
			          					<li class="" style=" font-size: 80%;  text-transform: uppercase;" ><a href="/portadas-especiales">VER MAS...</a></li>

			        </ul>
			      </li>
    </ul>

    <form action="/archivo" class="navbar-form navbar-right" role="search">
     <div class="">
      <div class="input-group">
        <input name="q" type="search" class="form-control" placeholder="Buscar" value="">
        <span class="input-group-btn">
          <button class="btn btn-default" type="submit"><span class="glyphicon glyphicon-search"></span></button>
        </span>
      </div><!-- /input-group -->
    </div><!-- /.col-lg-6 -->

  </form>

</div><!-- /.navbar-collapse -->
</nav>
        
<div class="row g-middle-container">
	            <a href="http://www.granma.cu/cuba-covid-19"><img src="/file/banner/sec-cuba-covid-19.jpg" style="padding-right: 15px; padding-left: 15px; margin-bottom: 10px;"  class='img-responsive' alt="Banner" /></a>
            <div class="col-md-8">

	

        <article class="g-story">

            <header class="g-story-header">
                <div class="g-story-meta">
                    <h3 class="g-story-epi"><p></p></span></h3>
                    <h1 itemprop="headline" class="g-story-heading">Cuba informará en próximos días sobre proceso de unificación monetaria y cambiaria</h1>
                    <p class="g-story-description" itemprop="description">Se dará la información completa de este complejo proceso de alcance estratégico que, aunque no es una solución mágica a todos los problemas de nuestra economía, sí ayudara a destrabar mucho de lo que es necesario ahora para que el país avance</p>
                    <div class="g-story-meta-footer">
                                            <p><span class="g-story-author" itemprop="author creator" itemscope="" itemtype="http://schema.org/Person" itemid="/archivo?a=3287">Autor: <a href="/archivo?a=3287" rel="author" title="Más artículos de Yisell  Rodríguez Milán"><span class="byline-author" itemprop="name">Yisell  Rodríguez Milán</span></a> | <a title="Escribir a Yisell  Rodríguez Milán" href="mailto:yisell@granma.cu">yisell@granma.cu</a></span></p>
                                            <p><time class="dateline" datetime="2020-10-08 18:10:59">8 de octubre de 2020  18:10:59</time></p>
                    </div><!-- close story-meta-footer -->
                </div><!-- close story-meta -->
            </header>

        
            <div class="g-story-media-container">
                <figure class="g-story-media-photo" itemprop="associatedMedia" itemscope="" itemid="/file/img/2014/09/medium/f0019252.jpg" itemtype="http://schema.org/ImageObject">
                    <div class="image">
                        <img src="/file/img/2014/09/medium/f0019252.jpg" alt="Unificacion monetaria Cuba " class="" data-mediaviewer-src="/file/img/2014/09/medium/f0019252.jpg" data-mediaviewer-caption="Unificacion monetaria Cuba " data-mediaviewer-credit="Cubadebate" itemprop="url" itemid="/file/img/2014/09/medium/f0019252.jpg">
                        <meta itemprop="identifier" content="/file/img/2014/09/medium/f0019252.jpg">
                        <meta itemprop="height" content="450">
                        <meta itemprop="width" content="675">
                    </div>
                    <figcaption class="g-story-media-caption" itemprop="description">
                        <span class="caption-text"></span>
                        <strong>Foto:</strong> <span class="credit" itemprop="copyrightHolder">Cubadebate</span>
                    </figcaption>
                </figure>
            </div>

        
            <div style="overflow-y:scroll" class="story-body-textt story-content" itemprop="articleBody">
                                <p>En el caso de la Tarea Ordenamiento, que debe llevar a la unificación monetaria y cambiaria, debemos decir que está en una fase de capacitación de las personas que deben llevar a cabo su implementación, expresó el Presidente cubano Miguel Díaz-Canel Bermúdez, en el programa de la Mesa Redonda correspondiente a este jueves, cuando junto al Primer Ministro, Manuel Marrero, informó las características de la entrada de Cuba a un escenario de «nueva normalidad».</p>
<p>El mandatario pidió al pueblo no seguir noticias falsas ni «bolas» que transmiten información incompleta o errónea sobre la unificación monetaria y cambiaria. Es importante —señaló— que la población sepa que se darán todas las precisiones en su momento.</p>
<div class="col-sm-8 col-md-6 pull-left">
<div class="thumbnailrel"><img src="/file/img/2020/07/medium/f0174341.jpg" />
<div class="captionrel">
<h4 class="media-heading"><a href="/cuba/2020-10-08/participaran-presidente-y-primer-ministro-de-cuba-en-la-mesa-redonda-08-10-2020-08-10-06">En vivo: Presidente y Primer Ministro de Cuba en la Mesa Redonda</a></h4>
</div>
</div>
</div>
<p>También abordó que, en días pasados, se filtró una información de una persona que estaba participando en una actividad de capacitación para la implementación de la Tarea Ordenamiento, pero lo que se divulgó fue lo usado como ejemplo para ilustrar el proceso. Por eso, agregó, se deben seguir las divulgaciones oficiales sobre el tema.</p>
<p>La Tarea Ordenamiento lleva aparejada una reforma de salarios, pensiones y precios, puntualizó.</p>
<p>Igualmente reiteró, una vez más, como se hizo en el Informe Central al VII Congreso del Partido Comunista de Cuba, la decisión de garantizar los depósitos en cuentas bancarias en divisas extranjeras, pesos convertibles (CUC) y pesos cubanos (CUP), así como el dinero en efectivo en poder de la población.</p>
<p>Nadie tiene que tener preocupación con sus cuentas, sean en MN, CUC o MLC, incluidos nuestros colaboradores, pues tenemos información de que ya se están propagando informaciones falsas sobre sus cuentas.</p>
<p>Nosotros nos vamos a quedar con la moneda nacional, que tendrá una convertibilidad a las otras, señaló y agregó que el día que se anuncie el inicio de la unificación, no hay que ir corriendo para los bancos, pues se va a garantizar un tiempo para que las personas puedan cambiar su dinero.</p>
<p>Díaz-Canel, además, abordó los beneficios de la unificación monetaria y cambiaria para la economía nacional. La unificación traerá consigo, entre otras cuestiones, el fortalecimiento de la empresa estatal socialista, el reordenamiento del comercio interior; la modificación del sistema de distribución de divisas;  la participación de la industria nacional como importante proveedor de bienes y servicios; el incremento y diversificación de las exportaciones, el diseño e implementación de incentivos fiscales para los exportadores; y el perfeccionamiento del trabajo del sector no estatal. </p>
<p>Adelantó que, en próximos días, se dará la información completa de este complejo proceso de alcance estratégico que, aunque no es una solución mágica a todos los problemas de nuestra economía, sí ayudara a destrabar mucho de lo que es necesario ahora para que el país avance.</p>            </div>
			
			<div class="span8 trivia" itemprop="articleBody">
                
            </div>
			
			<div class="span8" itemprop="articleBody">
                
            </div>
			
			<div class="span8" itemprop="articleBody">
                            </div>
			
			<div class="story-body-textt story-content" itemprop="articleBody">
							</div>
			
            <aside>
                        <h5>Informaciones relacionadas</h5>
                <ul class="g-story-relate">
                                <li><h4><a href="/cuba-covid-19/2020-10-08/participaran-presidente-y-primer-ministro-de-cuba-en-la-mesa-redonda-08-10-2020-08-10-06">En vivo: Presidente y Primer Ministro de Cuba en la Mesa Redonda</a></h4></li>
                                <li><h4><a href="/cuba/2020-08-19/unificacion-monetaria-en-el-horizonte-de-cuba-19-08-2020-22-08-45">Unificación monetaria en el horizonte de Cuba</a></h4></li>
                                <li><h4><a href="/pensar-en-qr/2020-10-09/nuestro-socialismo-excluye-la-aplicacion-de-terapias-de-choque-09-10-2020-01-10-07">Unificación monetaria y cambiaria: Nuestro socialismo excluye la aplicación de terapias de choque</a></h4></li>
                            </ul>
                    </aside>

            <footer class="g-story-footer">
                <p class="g-story-share"><span class="g-story-share-a">COMPARTIR</span>
                    <a target="_blank" href="http://www.facebook.com/share.php?u=http://www.granma.cu/cuba-covid-19/2020-10-08/cuba-informara-en-proximos-dias-sobre-proceso-de-unificacion-monetaria-y-cambiaria" class="g-mid-header-fb"><span class="sr-only">facebook</span></a>
                    <a target="_blank" href="https://twitter.com/intent/tweet?text=Cuba informará en próximos días sobre proceso de unificación monetaria y cambiaria http://www.granma.cu/cuba-covid-19/2020-10-08/cuba-informara-en-proximos-dias-sobre-proceso-de-unificacion-monetaria-y-cambiaria Via @Granma_Digital" class="g-mid-header-tw"><span class="sr-only">twitter</span></a>
					  <a target="_blank" href="https://t.me/share/url?url=http://www.granma.cu/cuba-covid-19/2020-10-08/cuba-informara-en-proximos-dias-sobre-proceso-de-unificacion-monetaria-y-cambiaria" class="g-mid-header-tg"><span class="sr-only">telegram</span></a>
                    <a target="_blank" href="https://plus.google.com/share?url=http://www.granma.cu/cuba-covid-19/2020-10-08/cuba-informara-en-proximos-dias-sobre-proceso-de-unificacion-monetaria-y-cambiaria" class="g-mid-header-g"><span class="sr-only">google +</span></a>
                </p>
            </footer>
        </article>
		
		            <div class="g-story-comments">

    <h5 class="comment-title">COMENTAR</h5>
    <div class="row">
        <a name="comentarios"></a>
        <form class="g-comments-form" role="form">
            <div class="col-md-8 container-left">
                <div class="col-md-6 form-group container-left">
                    <label for="n">Nombre</label>
                    <input required type="text" class="form-control" id="n" placeholder="Nombre">
                </div>
                <div class="col-md-6 form-group container-right">
                    <label for="e">Email</label>
                    <input required type="email" class="form-control" id="e" placeholder="Correo electrónico">
                </div>
                <div class="col-md-12 form-group comment-textarea">
                    <label for="t">Su comentario</label>
                    <textarea required class="form-control" rows="3" name="t" id="t"></textarea>
                </div>
            </div>
            <div class="col-md-4 container-right">
                <ul class="g-story-body-list">
                    <li><p>Mostrar respeto a los criterios en sus comentarios.</p></li>
                    <li><p>No ofender, ni usar frases vulgares y/o palabras obscenas.</p></li>
                    <li><p>Nos reservaremos el derecho de moderar aquellos comentarios que no cumplan con las reglas de uso.</p></li>
                </ul>
                <button id="comment_btn" type="button" class="btn btn-default">ENVIAR</button>
            </div>
        </form>
        <img style="display:none" src="/assets/img/ldr_flat.gif">
    </div>

    <div class="g-story-comments-list">
                    <div class="media">
    <div class="pull-left">
        <p class="comment-user"><strong>Tula</strong> dijo:</p>
        <p class="g-story-comments">1</p>
        <p class="comment-date">8 de octubre de 2020</p>
        <p class="comment-time"> 23:35:19</p>
        
	
		</br>
		<button id="" class="btn btn-default respuesta" data-comment-autor="Tula" data-comment-id="1283017" data-path="" style="display: block; padding: 1px 5px; background-color: #7C6865; color: white; border-color: #7C6865;">Responder</button>

		
    </div>
    <div class="media-body">
        <p class="comment-message">Solo nos queda apoyar las decisiones tomadas por nuestro gobierno. De esta situación, también saldremos airosos.</p>

    </div>
	
	        

</div>
                    <div class="media">
    <div class="pull-left">
        <p class="comment-user"><strong>carlos lugo rodriguez</strong> dijo:</p>
        <p class="g-story-comments">2</p>
        <p class="comment-date">9 de octubre de 2020</p>
        <p class="comment-time"> 00:14:34</p>
        
	
		</br>
		<button id="" class="btn btn-default respuesta" data-comment-autor="carlos lugo rodriguez" data-comment-id="1283027" data-path="" style="display: block; padding: 1px 5px; background-color: #7C6865; color: white; border-color: #7C6865;">Responder</button>

		
    </div>
    <div class="media-body">
        <p class="comment-message">Al fin es un paso necesario para organizar el sistema financiero y la economia unificandola y evitando tantos tipos de cambio diferentes que distorcionan la realidad de inversion,costo,gastosy utilidades o perdidas. </p>

    </div>
	
	        

</div>
                    <div class="media">
    <div class="pull-left">
        <p class="comment-user"><strong>Orlando Roberto</strong> dijo:</p>
        <p class="g-story-comments">3</p>
        <p class="comment-date">9 de octubre de 2020</p>
        <p class="comment-time"> 00:14:01</p>
        
	
		</br>
		<button id="" class="btn btn-default respuesta" data-comment-autor="Orlando Roberto" data-comment-id="1283029" data-path="" style="display: block; padding: 1px 5px; background-color: #7C6865; color: white; border-color: #7C6865;">Responder</button>

		
    </div>
    <div class="media-body">
        <p class="comment-message">Excelente explicación por parte de nuestro Presidente, realmente era necesario y qué bueno que el gobierno está dando estos pasos con el objetivo de crear una seguridad para la población en cuanto a la oleada de falsas noticias respecto al tema. Debemos estar atentos a todos los programas de la Mesa Redonda donde el Gobierno intervenga para explicar el tema para así tener dominio del tema en cuestión y poder emitir una opinión oportuna y coherente. Gracias Cuba por seguir avanzando, confío en Cuba, confío en la Revolución</p>

    </div>
	
	        

</div>
                    <div class="media">
    <div class="pull-left">
        <p class="comment-user"><strong>manu</strong> dijo:</p>
        <p class="g-story-comments">4</p>
        <p class="comment-date">9 de octubre de 2020</p>
        <p class="comment-time"> 09:02:56</p>
        
	
		</br>
		<button id="" class="btn btn-default respuesta" data-comment-autor="manu" data-comment-id="1283249" data-path="" style="display: block; padding: 1px 5px; background-color: #7C6865; color: white; border-color: #7C6865;">Responder</button>

		
    </div>
    <div class="media-body">
        <p class="comment-message">HAY QUE APOYAR A NUESTRO PRESIDENTE,LA UNIFICACIÓN ES MUY NECESARIA,ESO AYUDARIA SABER COMO QUIEN SE BENEFICIAN MÁS DE LOS BIENES Y SERVICIOS , EL USO DE LAS TAGETAS PARA COBRAR BIENES Y SERVICIOS CONTROLA MÁS EL DINERO NEGRO CIRCULANTE,LA LIBRETA ESTA AYUDA SOCIAL DEBÍA DARSE A  NUCLEOS FAMILIARES QUE EN SU CONJUNTO NO LLEGUEN AL SALARIO PROMEDIO ,MINIMO  AL AÑO.Y LAS AYUDAS POR SERVICIO SOCIAL A MEDICAMENTOS, ALIMENTOS ,Y OTROS RESPONDAN IGUAL QUE EN SU CONJUNTO EL NUCLEO FAMILIAR ESTE POR DEBAJO DEL SALARIO MINIMO.PERO PARA HACER TODO ESTO NECESITAN UNOS SERVICIOS SOCIALES PROFECIONALES Y CON SUPERVICIÓN, Y MÁS PERSONAL.NINGÚN ESTADO ESTADO, Y MÁS SOCIALISTA DEJARA DESAMPARADO A LOS MÁS HUMILDE, PERO TODO DEBE HACERSE CON ORDEN, Y MÁS COMO ESTA LA ECONOMIA Y COMO SE AVECINA. SUERTE. SALUDOS.</p>

    </div>
	
	        

</div>
                    <div class="media">
    <div class="pull-left">
        <p class="comment-user"><strong>Luis D Cruz Estevez</strong> dijo:</p>
        <p class="g-story-comments">5</p>
        <p class="comment-date">9 de octubre de 2020</p>
        <p class="comment-time"> 12:51:18</p>
        
	
		</br>
		<button id="" class="btn btn-default respuesta" data-comment-autor="Luis D Cruz Estevez" data-comment-id="1283415" data-path="" style="display: block; padding: 1px 5px; background-color: #7C6865; color: white; border-color: #7C6865;">Responder</button>

		
    </div>
    <div class="media-body">
        <p class="comment-message">Es bueno subir salario pero no hacemos nada si suben los precios de los productos es como no hacer nada</p>

    </div>
	
	        

</div>
                    <div class="media">
    <div class="pull-left">
        <p class="comment-user"><strong>Midalys</strong> dijo:</p>
        <p class="g-story-comments">6</p>
        <p class="comment-date">9 de octubre de 2020</p>
        <p class="comment-time"> 13:32:32</p>
        
	
		</br>
		<button id="" class="btn btn-default respuesta" data-comment-autor="Midalys" data-comment-id="1283431" data-path="" style="display: block; padding: 1px 5px; background-color: #7C6865; color: white; border-color: #7C6865;">Responder</button>

		
    </div>
    <div class="media-body">
        <p class="comment-message">q bien  que nuestro país se está  queriendo reordenar para mejorra nuestra economía, pero lo que   se necesita es que cando eso suceda  exista una correspondencia entre salario y prrcios de los productos en el Mercado </p>

    </div>
	
	        

</div>
                    <div class="media">
    <div class="pull-left">
        <p class="comment-user"><strong>Jorge Chi</strong> dijo:</p>
        <p class="g-story-comments">7</p>
        <p class="comment-date">9 de octubre de 2020</p>
        <p class="comment-time"> 13:42:43</p>
        
	
		</br>
		<button id="" class="btn btn-default respuesta" data-comment-autor="Jorge Chi" data-comment-id="1283439" data-path="" style="display: block; padding: 1px 5px; background-color: #7C6865; color: white; border-color: #7C6865;">Responder</button>

		
    </div>
    <div class="media-body">
        <p class="comment-message">Mí criterio particular es que nuestro presidente, así como nuestro primer ministro, han cumplido muy bien con su deber ante el pueblo que representan, al informarnos sobre todo lo que se va a implementar, para evitar que sigan corriendo las bolas, por personas que quieren lograr el disgusto solamente, y que no lo hacen con buenas intenciones.</p>

    </div>
	
	        

</div>
                    <div class="media">
    <div class="pull-left">
        <p class="comment-user"><strong>Angel</strong> dijo:</p>
        <p class="g-story-comments">8</p>
        <p class="comment-date">9 de octubre de 2020</p>
        <p class="comment-time"> 13:55:43</p>
        
	
		</br>
		<button id="" class="btn btn-default respuesta" data-comment-autor="Angel" data-comment-id="1283445" data-path="" style="display: block; padding: 1px 5px; background-color: #7C6865; color: white; border-color: #7C6865;">Responder</button>

		
    </div>
    <div class="media-body">
        <p class="comment-message">Presidente.... solo le diré una cosa, con una palabra le haré saber mi sentido y pensamiento hacia este país y Revolución, y hacia usted también claro... mi palabra es GRACIAS</p>

    </div>
	
	        

</div>
                    <div class="media">
    <div class="pull-left">
        <p class="comment-user"><strong>María Elena López Chacón</strong> dijo:</p>
        <p class="g-story-comments">9</p>
        <p class="comment-date">9 de octubre de 2020</p>
        <p class="comment-time"> 15:46:56</p>
        
	
		</br>
		<button id="" class="btn btn-default respuesta" data-comment-autor="María Elena López Chacón" data-comment-id="1283519" data-path="" style="display: block; padding: 1px 5px; background-color: #7C6865; color: white; border-color: #7C6865;">Responder</button>

		
    </div>
    <div class="media-body">
        <p class="comment-message">El pueblo agradece las informaciones que se abordan en la Mesa Redonda, es la mejor manera de contrarestar cualquier desinformación. Sabemos que el País tiene que avanzar, así lo establecen los documentos de 7mo Congreso del Partido. Apoyo las medidas, a la Dirección del País y nuestra Revolución.</p>

    </div>
	
	        

</div>
                    <div class="media">
    <div class="pull-left">
        <p class="comment-user"><strong>Haile</strong> dijo:</p>
        <p class="g-story-comments">10</p>
        <p class="comment-date">10 de octubre de 2020</p>
        <p class="comment-time"> 15:26:24</p>
        
	
		</br>
		<button id="" class="btn btn-default respuesta" data-comment-autor="Haile" data-comment-id="1284037" data-path="" style="display: block; padding: 1px 5px; background-color: #7C6865; color: white; border-color: #7C6865;">Responder</button>

		
    </div>
    <div class="media-body">
        <p class="comment-message">Es necesario mantener al corriente a la población de las mesas redondas q van a explicar el proceso para que no corran las bolas por ejemplo deberían publicar qué día y a qué hora es la mesa redonda en la q van a explicar todo gracias por su información</p>

    </div>
	
	        

</div>
                    <div class="media">
    <div class="pull-left">
        <p class="comment-user"><strong>Carlos Eduardo Ojea Rodríguez </strong> dijo:</p>
        <p class="g-story-comments">11</p>
        <p class="comment-date">10 de octubre de 2020</p>
        <p class="comment-time"> 21:50:32</p>
        
	
		</br>
		<button id="" class="btn btn-default respuesta" data-comment-autor="Carlos Eduardo Ojea Rodríguez " data-comment-id="1284235" data-path="" style="display: block; padding: 1px 5px; background-color: #7C6865; color: white; border-color: #7C6865;">Responder</button>

		
    </div>
    <div class="media-body">
        <p class="comment-message">Muy bien por nuestro presidente viva Cuba Libre. </p>

    </div>
	
	        

</div>
        </div>

       
	   
	   <div class="modal wide hide fade " id="quick_reply_read" >
    <div class="modal-body">
	<button type="button" class="close" data-dismiss="modal" style="cursor: pointer; background: transparent none repeat scroll 0% 0%; border: 0px none; margin-top: 2px; float: right; font-size: 24px; font-weight: bold; line-height: 20px; color: #000; text-shadow: 0px 1px 0px #FFF; opacity: 0.2;">×</button>
    <h3>Responder comentario</h3>   
	<div class="row">
        <form class="g-comments-form" role="form">
            <div class="col-md-8 container-left">
                <div class="col-md-6 form-group container-left">
                    <label for="nom">Nombre</label>
                    <input required type="text" class="form-control" id="nom" placeholder="Nombre">
                </div>
                <div class="col-md-6 form-group container-right">
                    <label for="email">Email</label>
                    <input required type="email" class="form-control" id="email" placeholder="Correo electrónico">
                </div>
                <div class="col-md-12 form-group comment-textarea">
                    <label for="texto">Su comentario</label>
                </div>
                    <textarea required class="form-control" rows="3" name="texto" id="texto" placeholder="Escriba su respuesta aquí"></textarea>
            </div>
            <div class="col-md-4 container-right">
                <ul class="g-story-body-list">
                    <li><h4>Mostrar respeto a los criterios en sus comentarios.</h4></li>
                    <li><h4>No ofender, ni usar frases vulgares y/o palabras obscenas.</h4></li>
                    <li><h4>Nos reservaremos el derecho de moderar aquellos comentarios que no cumplan con las reglas de uso.</h4></li>
         
			 </ul>
			<button id="quick_reply_read_saver" class="btn btn-default" data-coment-id="" data-path="http://www.granma.cu/cms/comentarios/home/reply">RESPONDER</button>
            <button href="#" class="btn btn-default" data-dismiss="modal">CANCELAR</button>
			
			</div>
        </form>
    </div> 
  </div>
    
</div>
    
</div>
<script>
    var _t_id = 4534071;
    var _t_ty = 0;
</script>        	
    </div><!--/column-->

    <div role="navigation" id="sidebar" class="col-md-4 hidden-xs sidebar">
            <div class="col-md-12 col-sm-6 sidebar-widget">
            <a  title="Información del Minsap" href="/informacion-minsap"><img src="/file/banner/banner_minsap" class="img-responsive" alt="Información del Minsap"></a>        </div>
            <div class="col-md-12 col-sm-6 sidebar-widget">
            <a  title="Fake News: La mentira tiene patas cortas" href="fake-news"><img src="/file/banner/banner_fake_news" class="img-responsive" alt="Fake News: La mentira tiene patas cortas"></a>        </div>
            <div class="col-md-12 col-sm-6 sidebar-widget">
            <a  title="Consejos y trucos contra la Covid-19" href="/consejos-covid"><img src="/file/banner/banner_trucos_covid19" class="img-responsive" alt="Consejos y trucos contra la Covid-19"></a>        </div>
            <div class="col-md-12 col-sm-6 sidebar-widget">
            <div class="category-widget">
    <h3 class="category-widget-header">Cuba frente a la Covid-19</h3>
    <figure>
        <a href="cuba-covid-19/2020-06-13/presidente-de-cuba-y-cientifico-de-la-uh-publican-analisis-sobre-la-gestion-gubernamental-y-ciencia-ante-la-covid-19"><img src="/file/img/2020/06/thumb/f0171517.jpg" class="img-responsive" alt="Proteccion contra el coronavirus ,el pueblo de la capital recibe a la Brigada Henry Reeve que llega a la patria." ></a>
    </figure>
    <article class="category-widget-texts">
        <h2><a href="cuba-covid-19/2020-06-13/presidente-de-cuba-y-cientifico-de-la-uh-publican-analisis-sobre-la-gestion-gubernamental-y-ciencia-ante-la-covid-19">Presidente de Cuba y científico de la UH publican análisis sobre la gestión gubernamental y el papel de la ciencia ante la COVID-19</a></h2>
        <p><span style="color: #000000; font-family: Verdana, Arial, Helvetica, sans-serif; font-size: small;">El objetivo central de esta contribución es reflexionar sobre las experiencias acumuladas en el enfrentamiento a la COVID-19 con relación al vínculo entre los científicos y el Gobierno y sistematizarlas</span></p>        <p class="g-story-comments"><a href="cuba-covid-19/2020-06-13/presidente-de-cuba-y-cientifico-de-la-uh-publican-analisis-sobre-la-gestion-gubernamental-y-ciencia-ante-la-covid-19#comentarios">5 COMENTARIOS</a></p>
    </article>
</div>        </div>
            <div class="col-md-12 col-sm-6 sidebar-widget">
            <a  title="Titulares de Granma en tu móvil" href="/titulares-de-granma-en-tu-movil"><img src="/file/banner/Titular en tu movil.jpg" class="img-responsive" alt="Titulares de Granma en tu móvil"></a>        </div>
            <div class="col-md-12 col-sm-6 sidebar-widget">
            <h3 class="sidebar-widget-header">Grupos de WhatsApp y Telegram del Periódico Granma</h3>

<!--para Youtube con los relacionados quitadosss-->
<iframe src="https://www.youtube.com/embed/TsG61Gwddc4?rel=0" frameborder="0" allowfullscreen="allowfullscreen"></iframe>

        </div>
            <div class="col-md-12 col-sm-6 sidebar-widget">
            <a  title="Todo Salud" href="/todo-salud"><img src="/file/banner/todo-salud-banner.jpg" class="img-responsive" alt="Todo Salud"></a>        </div>
            <div class="col-md-12 col-sm-6 sidebar-widget">
            <h3 class="sidebar-widget-header">En profundidad</h3>
<div class="sugerencias">
    <article>
        <h4 class="media-heading"><a href="/consejos-covid/2020-10-14/china-ofrece-credito-a-america-latina-y-el-caribe-para-acceder-a-vacuna-contra-la-covid-19-29-07-2020-23-07-15">China ofrece crédito a América Latina y el Caribe para acceder a vacuna contra la COVID-19</a></h4>
            
        <a href="/consejos-covid/2020-10-14/china-ofrece-credito-a-america-latina-y-el-caribe-para-acceder-a-vacuna-contra-la-covid-19-29-07-2020-23-07-15"><img src="/file/img/2020/07/thumb/f0173725.jpg" class="img-responsive" alt="coronavirus"></a>
        </article>
    <ul class="media-list">
            <li class="media">
                    <a class="pull-left" href="/consejos-covid/2020-10-14/oms-alarmada-por-impacto-de-la-covid-19-en-pueblos-nativos-de-las-americas-27-07-2020-23-07-52">
                <img class="img-responsive" src="/file/img/2020/07/thumb/f0173725.jpg" alt="coronavirus">
            </a>
                    <div class="media-body">
                <h4 class="media-heading"><a href="/consejos-covid/2020-10-14/oms-alarmada-por-impacto-de-la-covid-19-en-pueblos-nativos-de-las-americas-27-07-2020-23-07-52">OMS alarmada por impacto de la COVID-19 en pueblos nativos de las Américas</a></h4>
            </div>
        </li>
            <li class="media">
                    <a class="pull-left" href="/consejos-covid/2020-10-14/la-primera-vacuna-rusa-contra-la-covid-19-esta-lista-22-07-2020-00-07-12">
                <img class="img-responsive" src="/file/img/2020/07/thumb/f0173725.jpg" alt="coronavirus">
            </a>
                    <div class="media-body">
                <h4 class="media-heading"><a href="/consejos-covid/2020-10-14/la-primera-vacuna-rusa-contra-la-covid-19-esta-lista-22-07-2020-00-07-12">La primera vacuna rusa contra la COVID-19 está lista</a></h4>
            </div>
        </li>
            <li class="media">
                    <a class="pull-left" href="/consejos-covid/2020-10-14/latinoamerica-sera-la-primera-en-recibir-el-farmaco-ruso-avifavir">
                <img class="img-responsive" src="/file/img/2020/07/thumb/f0173725.jpg" alt="coronavirus">
            </a>
                    <div class="media-body">
                <h4 class="media-heading"><a href="/consejos-covid/2020-10-14/latinoamerica-sera-la-primera-en-recibir-el-farmaco-ruso-avifavir">Latinoamérica será la primera en recibir el fármaco ruso Avifavir</a></h4>
            </div>
        </li>
            <li class="media">
                    <a class="pull-left" href="/consejos-covid/2020-10-14/no-sobran-las-medidas-de-proteccion">
                <img class="img-responsive" src="/file/img/2020/07/thumb/f0173725.jpg" alt="coronavirus">
            </a>
                    <div class="media-body">
                <h4 class="media-heading"><a href="/consejos-covid/2020-10-14/no-sobran-las-medidas-de-proteccion">No sobran las medidas de protección</a></h4>
            </div>
        </li>
            <li class="media">
                    <a class="pull-left" href="/consejos-covid/2020-10-14/nuevos-estudios-sobre-la-covid-19-en-cuba-02-07-2020-00-07-12">
                <img class="img-responsive" src="/file/img/2020/04/thumb/f0167165.jpg" alt="coronavirus">
            </a>
                    <div class="media-body">
                <h4 class="media-heading"><a href="/consejos-covid/2020-10-14/nuevos-estudios-sobre-la-covid-19-en-cuba-02-07-2020-00-07-12">Nuevos estudios sobre la COVID-19  en Cuba</a></h4>
            </div>
        </li>
        </ul>
</div>        </div>
            <div class="col-md-12 col-sm-6 sidebar-widget">
            <div id="fb-root"></div>
<script>(function(d, s, id) {
  var js, fjs = d.getElementsByTagName(s)[0];
  if (d.getElementById(id)) return;
  js = d.createElement(s); js.id = id;
  js.src = "//connect.facebook.net/es_LA/all.js#xfbml=1";
  fjs.parentNode.insertBefore(js, fjs);
}(document, 'script', 'facebook-jssdk'));</script>

<div class="fb-like-box" data-href="https://www.facebook.com/granmadigital" data-colorscheme="light"
 data-show-faces="true" data-header="true" data-stream="false" data-show-border="true"></div>        </div>
            <div class="col-md-12 col-sm-6 sidebar-widget">
            <a class="twitter-timeline" data-height="300" data-theme="light" href="https://twitter.com/Granma_Digital?ref_src=twsrc%5Etfw">Tweets by Granma_Digital</a> <script async src="https://platform.twitter.com/widgets.js" charset="utf-8"></script>
<a href="https://twitter.com/Granma_Digital?ref_src=twsrc%5Etfw" class="twitter-follow-button" data-show-count="true">Follow @Granma_Digital</a><script async src="https://platform.twitter.com/widgets.js" charset="utf-8"></script>        </div>
        </div><!--/sidebar-->

</div><!--/container-->



        <div id="footer">

            <nav class="navbar navbar-default g-menus g-main-menu g-main-menu-footer hidden-xs" role="navigation">

    <div class="collapse navbar-collapse navbar-main-collapse">

        <ul class="nav navbar-nav">
          <li ><a href="/">Portada</a></li>
                  <li ><a href="/cuba">Cuba</a></li>
                  <li ><a href="/mundo">Mundo</a></li>
                  <li ><a href="/deportes">Deportes</a></li>
                  <li ><a href="/cultura">Cultura</a></li>
                  <li ><a href="/opinion">Opinión</a></li>
                  <li ><a href="/ciencia">Ciencia</a></li>
                  <li ><a href="/salud">Salud</a></li>
                </ul>

        <ul class="nav navbar-nav navbar-right">
          <li><a href="/quienes-somos">¿Quiénes somos?</a></li>
        </ul>

    </div>
</nav>
            <div class="row g-footer-directory">
                <h3>DIRECTORIO DE PRENSA:</h3>
                <div class="col-lg-4 g-footer-directories">
                    <h4>PUBLICACIONES EN INTERNET</h4>
                    <p>
                        <a target="_blank" href=" http://www.juventudrebelde.cu">Juventud Rebelde</a> |
                        <a target="_blank" href="http://www.trabajadores.cu">Trabajadores.</a> |
                        <a target="_blank" href="http://www.cubadebate.cu/">Cubadebate</a> |
                        <a target="_blank" href="http://www.acn.cu">ACN</a> |
                        <a target="_blank" href="http://prensa-latina.cu/">Prensa Latina</a> |
                        <a target="_blank" href="http://www.opciones.cu/">Opciones</a> |
                        <a target="_blank" href="http://cubasi.cu/">CubaSi</a> |
                        <a target="_blank" href="http://www.cubahora.cu/">Cubahora</a> |
                        <a target="_blank" href="http://www.cubaperiodistas.cu/">Cubaperiodistas</a> |
                        <a target="_blank" href="http://www.jit.cu/">JIT</a>
                    </p>
                </div>
                <div class="col-lg-4 g-footer-directories">
                    <h4>REVISTAS</h4>
                    <p>
                        <a target="_blank" href="http://bohemia.cu/">Bohemia</a> |
                        <!--a target="_blank" href="http://www.cubasocialista.cu/">Cuba Socialista</a--> 
                        <a target="_blank" href="http://www.mujeres.co.cu/">Mujeres</a> |
                        <!--a target="_blank" href="http://www.tricontinental.cu/">Tricontinental</a-->
                        <a target="_blank" href="http://www.lajiribilla.cu/">La Jiribilla</a> |
                        <a target="_blank" href="http://www.almamater.cu/">Alma Mater</a> |
                        <a target="_blank" href="http://www.caimanbarbudo.cu/">Caimán Barbudo</a> |
                        <a target="_blank" href="http://www.juventudtecnica.cu/">Juventud Técnica</a> |
                        <a target="_blank" href="http://www.pionero.cu/">Pionero</a>
                    </p>
                </div>
                <div class="col-lg-4 g-footer-directories">
                    <h4>TV Y RADIO</h4>
                    <p>
                        <a target="_blank" href="http://www.tvcubana.icrt.cu/">TV Cubana</a> |
                        <a target="_blank" href="http://www.radiocubana.cu/">Radio Cubana</a> |
                        <a target="_blank" href="http://www.radiohc.cu/">Radio Habana Cuba</a> |
                        <a target="_blank" href="http://www.radioprogreso.cu/">Radio Progreso</a> |
                        <a target="_blank" href="http://www.radiorebelde.cu/">Radio Rebelde</a> |
                        <a target="_blank" href="http://www.radioreloj.cu/ ">Radio Reloj</a> |
                        <a target="_blank" href="http://www.radiotaino.cu/">Radio Taino</a> |
                        <a target="_blank" href="http://www.cmbfradio.cu/">Radio Musical Nacional</a>
                    </p>
                </div>
                 <div class="col-lg-4 g-footer-directories">
                    <h4>ENLACES DE INTER&Eacute;S</h4>
                    <p>
                        <a target="_blank" href="http://www.pcc.cu/">Partido Comunista de Cuba</a> |
                        <a target="_blank" href="http://www.parlamentocubano.cu/">Sitio oficial del Gobierno de la Rep&uacute;blica de Cuba</a> |
                        <a target="_blank" href="http://www.minrex.gob.cu/">Ministerio de Relaciones Exteriores</a> |
                        <a target="_blank" href="http://www.aduana.gob.cu/">Aduana</a>
                    </p>
                </div>
            </div>
            <div class="g-footer-staff">
                <div class="g-footer-logo"><img src="/static/img/granma-logo.png" class="img-responsive" alt="Responsive image" width="100"></div>
                <h5 class="g-footer-subtitle">&Oacute;RGANO OFICIAL DEL COMIT&Eacute; CENTRAL DEL PARTIDO COMUNISTA DE CUBA</h5>
                <h3>GRANMA © 2018 | Fundado el 3 de octubre de 1965 | ISNN 0864-0424 | General Su&aacute;rez y Territorial, Plaza de la Revoluci&oacute;n, La Habana. Cuba | Tel&eacute;fono: 881-3333</h3>
            </div>

        </div>

    </div> <!-- /container -->
	<div class="back-top btn btn-danger">
		<span class="glyphicon glyphicon-chevron-up"></span>
	</div>
    <!--script src="/static/js/jquery.js"></script-->
	<script src="/static/js/linea/jquery2.min.js"></script>

    <script src="/static/js/bootstrap.min.js"></script>
    <script src="/static/js/encuestas.votar.js"></script>
	<script src="/static/js/infografia.js"></script>
	<script async src="https://platform.twitter.com/widgets.js" charset="utf-8"></script>
	<script async src="//www.instagram.com/embed.js"></script>
	
    <script  src="/static/js/linea/index.js"></script>
	<script  src="/static/js/underscore-1.8.3.min.js"></script>
    <script  src="/static/js/quizzer.js"></script>
    
<script src="/static/js/comment.js"></script>

<!-- Google Analytic -->
<script type="text/javascript">

  var _gaq = _gaq || [];
  _gaq.push(['_setAccount', 'UA-112047104-1']);
  _gaq.push(['_trackPageview']);

  (function() {
    var ga = document.createElement('script'); 
	ga.type = 'text/javascript'; 
	ga.async = true;
    ga.src = ('https:' == document.location.protocol ? 'https://ssl' : 'http://www') + '.google-analytics.com/ga.js';
    var s = document.getElementsByTagName('script')[0]; 
	s.parentNode.insertBefore(ga, s);
  })();

</script>
<!-- End Google Analytic -->

<!-- Piwik -->
<script type="text/javascript">
   var _paq = _paq || [];
   _paq.push(["setDocumentTitle", document.domain + "/" + document.title]);
   _paq.push(["setCookieDomain", "*.www.granma.cu"]);
   _paq.push(["setDomains", ["*.www.granma.cu"]]);
   _paq.push(["trackPageView"]);
   _paq.push(["enableLinkTracking"]);

   (function() {
     var u=(("https:" == document.location.protocol) ? "https" : "http") 
+ "://piwikstats.cip.cu/";
     _paq.push(["setTrackerUrl", u+"piwik.php"]);
     _paq.push(["setSiteId", "101"]);
     var d=document, g=d.createElement("script"),
s=d.getElementsByTagName("script")[0]; g.type="text/javascript";
     g.defer=true; g.async=true; g.src=u+"piwik.js";
s.parentNode.insertBefore(g,s);
   })();
</script>
<!-- End Piwik Code -->




</body>
</html>